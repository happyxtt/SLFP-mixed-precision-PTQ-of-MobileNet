#include "hip/hip_runtime.h"
#include "./common/MobileNetv1.h"
//输入图像归一化
  __global__ void image(uint8_t *image, sfp *out, int inputRow, float Ka)  
  {
    const int idx = blockDim.x*blockIdx.x + threadIdx.x;
    const int i = blockIdx.y;
    float temp[224*224*3];
    temp[i*inputRow*inputRow + idx] = float(image[i*inputRow*inputRow + idx])/255;
    temp[i*inputRow*inputRow + idx] = 2*(temp[i*inputRow*inputRow + idx] - 0.5);
    temp[i*inputRow*inputRow + idx] = temp[i*inputRow*inputRow + idx]/Ka;
    out[i*inputRow*inputRow + idx] = float2sfp(temp[i*inputRow*inputRow + idx], 5);
  }
//3*3普通卷积，步长2
    __global__ void conv2(sfp *active, sfp *weight, float *bias, sfp *out, float *outint24,  int inputRow, int outputRow, int inputChannel, float Ka, float Kw, float Kr, int type_in, int type_out) //modified
    {
      const int idx = blockDim.x*blockIdx.x + threadIdx.x;
      int row = blockIdx.x;
      int col = threadIdx.x;
      int kernelSize = 3;
      int i = blockIdx.y;
      outint24[i*outputRow*outputRow + idx] = 0;
      for(int l = 0; l < inputChannel; ++l)
      {
        for(int j = 0; j < kernelSize; ++j)
        {
          for(int k = 0; k < kernelSize; ++k)
          {
              sfp imgValue;
              int curRow = row*2 - kernelSize / 2 + j;
              int curCol = col*2 - kernelSize / 2 + k;
              if(curRow < 0 || curCol < 0 || curRow >= inputRow || curCol >= inputRow)
              {
                imgValue.sign = 0;
                imgValue.exp = 0;
                imgValue.mnt = 0;
              }
              else
              {
                imgValue = active[l*inputRow*inputRow + curRow*inputRow  + curCol];
              }
              outint24[i*outputRow*outputRow + idx] += sfp2fixed(imgValue, weight[i*inputChannel*kernelSize*kernelSize + l*kernelSize*kernelSize + j*kernelSize + k], type_in);
          }
        }
      }
      outint24[i*outputRow*outputRow + idx] = outint24[i*outputRow*outputRow + idx] * (Kw*Ka);
      outint24[i*outputRow*outputRow + idx] = (outint24[i*outputRow*outputRow + idx] + bias[i])/Kr;
      outint24[i*outputRow*outputRow + idx] = (outint24[i*outputRow*outputRow + idx] > 0) ? outint24[i*outputRow*outputRow + idx] : 0;   
      outint24[i*outputRow*outputRow + idx] = (outint24[i*outputRow*outputRow + idx] < 252) ? outint24[i*outputRow*outputRow + idx] : 252;
      out[i*outputRow*outputRow + idx] = float2sfp(outint24[i*outputRow*outputRow + idx], type_out);
    }

//1*1普通卷积，步长1
    __global__ void conv1(sfp *active, sfp *weight, float *bias, sfp *out, float *outint24 , int inputRow, int outputRow, int inputChannel, float Ka, float Kw, float Kr, int type_in, int type_out)
    {
      const int idx = blockDim.x*blockIdx.x + threadIdx.x;
      int i = blockIdx.y;
      outint24[i*outputRow*outputRow + idx] = 0;
      for(int j = 0; j < inputChannel; ++j)//遍历输入的每个通道
      {
        outint24[i*outputRow*outputRow + idx] += sfp2fixed(active[j*inputRow*inputRow + idx ] , weight[i*inputChannel + j], type_in);
      }
      outint24[i*outputRow*outputRow + idx] = outint24[i*outputRow*outputRow + idx] * (Kw*Ka);
      outint24[i*outputRow*outputRow + idx] = (outint24[i*outputRow*outputRow + idx] + bias[i])/Kr;
      outint24[i*outputRow*outputRow + idx] = (outint24[i*outputRow*outputRow + idx] > 0) ? outint24[i*outputRow*outputRow + idx] : 0;   
      outint24[i*outputRow*outputRow + idx] = (outint24[i*outputRow*outputRow + idx] < 252) ? outint24[i*outputRow*outputRow + idx] : 252;
      out[i*outputRow*outputRow + idx] = float2sfp(outint24[i*outputRow*outputRow + idx], type_out);
    } 
//3*3DW卷积，步长1或2
	__global__ void conv_DW(sfp *active, sfp *weight, float *bias, sfp *out, float *outint24 , int inputRow, int outputRow, int stride, float Ka, float Kw, float Kr, int type_in, int type_out)
	{
    const int idx = blockDim.x*blockIdx.x + threadIdx.x;
    int row = blockIdx.x;
    int col = threadIdx.x;
    int kernelSize = 3;
    int i = blockIdx.y;
    outint24[i*outputRow*outputRow + idx] = 0; 
    for(int j = 0; j < kernelSize; ++j)
    {
      for(int k = 0; k < kernelSize; ++k)
      {
        sfp imgValue;
        int curRow;
        int curCol;
        if(stride == 1)
        {
          curRow = stride*row - kernelSize / 2 + j;
          curCol = stride*col - kernelSize / 2 + k;
        }
        else
        {
          curRow = stride*row - kernelSize / 2 + j + 1;
          curCol = stride*col - kernelSize / 2 + k + 1;
        }
        if(curRow < 0 || curCol < 0 || curRow >= inputRow || curCol >= inputRow)
        {
          imgValue.sign = 0;
          imgValue.exp = 0;
          imgValue.mnt = 0;
        }
        else
        {
          imgValue = active[i*inputRow*inputRow + curRow * inputRow + curCol];
        }
        outint24[i*outputRow*outputRow + idx] += sfp2fixed(weight[i*kernelSize*kernelSize + j * kernelSize + k] , imgValue, type_in);
      }
    }
    outint24[i*outputRow*outputRow + idx] = outint24[i*outputRow*outputRow + idx] * (Kw*Ka);
    outint24[i*outputRow*outputRow + idx] = (outint24[i*outputRow*outputRow + idx] + bias[i])/Kr;
    outint24[i*outputRow*outputRow + idx] = (outint24[i*outputRow*outputRow + idx] > 0) ? outint24[i*outputRow*outputRow + idx] : 0; 
    outint24[i*outputRow*outputRow + idx] = (outint24[i*outputRow*outputRow + idx] < 252) ? outint24[i*outputRow*outputRow + idx] : 252;
    out[i*outputRow*outputRow + idx] = float2sfp(outint24[i*outputRow*outputRow + idx], type_out);
	}
//Avgpool
  __global__ void avgpool(sfp *active, sfp *out, int inputRow)
  {
    const int idx = blockDim.x*blockIdx.x + threadIdx.x;
    float sumadd[1024];
    sumadd[idx] = 0;
    for(int j = 0; j < inputRow*inputRow; ++j)//遍历输入的每个像素点
    {
      sumadd[idx] += sfp2float(active[idx*inputRow*inputRow + j])/(inputRow*inputRow);
    }
    out[idx] = float2sfp(sumadd[idx], 5);
  }

//fullconnection
  __global__ void fullconnection(sfp *active , sfp *weight , float *bias , sfp *out ,  float *outint24, float Ka, float Kw, float Kr)
  {
    int idx = threadIdx.x;
    outint24[idx] = 0;
    if(idx < 1000)
    {
      for(int i = 0; i < 1024 ; i++)
      {
        outint24[idx] += sfp2fixed(active[i],weight[idx*1024 + i], 5);
      }
      outint24[idx] = outint24[idx] * (Kw*Ka);
      outint24[idx] = (outint24[idx] + bias[idx])/Kr;
      out[idx] = float2sfp(outint24[idx], 5);
    }
  }

//排序
  __global__ void BubbleSort(sfp* data , int *rank , int *TOP1 , int *TOP5 , int *pic_idx , int idx , int maxnum,float Ka)
  {
      int i = 0;
      int j = 0;
      int position[1000];
      float a[1000];
      float tmp = 0;
      for(i = 0; i<1000 ; ++i)
      {
        position[i] = i;
        a[i] = sfp2float(data[i]);
      }
      for (i = 0; i < 1000 - 1; ++i)
      {
          for (j = 1; j < 1000 - i; ++j)
          {
              if (a[j] > a[j-1])    /* 从大到小排序，把较小的交换到后面来 */
              {
                  tmp = a[j-1];
                  a[j-1] = a[j];
                  a[j] = tmp;
                  /* 记录位置 */
                  tmp = position[j-1];
                  position[j-1] = position[j];
                  position[j] = tmp;
              }
          }
      }
      for(int j = 0; j < 5; ++j)
      {
        rank[j] = position[j];
        //  printf("rank[%d]:%d\n",j,rank[j]);
        //  printf("a[%d]:%f\n",j,a[j]*Ka);
      }
      //  printf("%d\n",pic_idx[idx]);
      if(rank[0] == pic_idx[idx])
      {
        TOP1[0]++;
        printf("Top1:%d\n",TOP1[0]);
      }
      if(rank[0] == pic_idx[idx] || rank[1] == pic_idx[idx] || rank[2] == pic_idx[idx] || rank[3] == pic_idx[idx] || rank[4] == pic_idx[idx])
      {
        TOP5[0]++;
        printf("Top5:%d\n",TOP5[0]);
      }
      if(idx == maxnum-1)
      {
        // printf("d_top1 = %f , d_top5 = %f\n" , float(TOP1[0]) , float(TOP5[0]));
        //printf("d_top1 = %f%% , d_top5 = %f%%\n" , float(TOP1[0]/maxnum*100) , float(TOP5[0]/maxnum*100));
      }
  }