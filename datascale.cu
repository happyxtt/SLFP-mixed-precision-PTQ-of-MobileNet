#include "hip/hip_runtime.h"
#include "./common/MobileNetv1.h"
//浮点数转定sfp<4,5> or sfp<4,3>
sfp floattosfp(float a , int type) // modified
  {
    myfixed reg;
    myfixed regFixed;
    sfp out;
    float Mfraction;
    if(type == 5)
    {
      reg.sign = (a >= 0)?0:1;//符号位
      reg.integer = (int)abs(a);//整数部分
      Mfraction = abs(a) - reg.integer;//取小数部分
      reg.fraction = 0;
      for(int i = 0; i < 13; ++i)
      {
        reg.fraction  = reg.fraction << 1; 
        if( Mfraction*2 >= 1)
        {
          reg.fraction = reg.fraction + 1;
          Mfraction = Mfraction*2 - 1;
        }
        else
        {
          reg.fraction = reg.fraction + 0;
          Mfraction = Mfraction*2;
        }
      }
      regFixed.fix  = ((reg.fraction&0b1) == 1)?((reg.integer << 12) + (reg.fraction>>1) + 1) : ((reg.integer << 12) + (reg.fraction>>1));
      //首先判断符号位
      if(reg.sign == 1)
      {
        out.sign = 1;
      }
      else
      {
        out.sign = 0;
      }

      //判断是否超出SFP表示范围
      if(regFixed.fix >= 0xFE000)//格式为1111 11_1X.XXXX XXXX XXXX
      {
        out.exp = 15;
        out.mnt = 31;
      }
      else if(regFixed.fix <= 0x10)//格式为0.0000 0001 0000 
      {
        out.exp = 0;
        out.mnt = 0;
      }
      else
      {
        if(regFixed.fix >= 0x80000)//格式为1XXX XX_XX.XXXX XXXX XXXX
          {
            if((regFixed.fix&0x3FFF) == 0x2000)//余位为1000...的情况
            {
              if((regFixed.fix&0x4000) == 0x4000)//尾数为奇数+1,偶数不进位
              {
                out.exp = 15;
                out.mnt = ((regFixed.fix&0x7C000)>>14) + 1;
              }
              else
              {
                out.exp = 15;
                out.mnt = ((regFixed.fix&0x7C000)>>14);
              }
            }
            else if((regFixed.fix&0x3FFF) > 0x2000)//不是1000...的情况。大于进位，小于舍去
            {
              out.exp = 15;
              out.mnt = ((regFixed.fix&0x7C000)>>14) + 1;
            }
            else
            {
              out.exp = 15;
              out.mnt = ((regFixed.fix&0x7C000)>>14);
            }
          }
        else if((regFixed.fix >= 0x40000) && (regFixed.fix < 0x80000))//格式为1XX XXX_X.XXXX XXXX XXXX
          {
            if((regFixed.fix&0x1FFF) == 0x1000)//余位为1000...的情况
            {
              if((regFixed.fix&0x2000) == 0x2000)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x3E000)>>13) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 14 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 14;
                  out.mnt = ((regFixed.fix&0x3E000)>>13) + 1;
                }
              }
              else
              {
                out.exp = 14;
                out.mnt = ((regFixed.fix&0x3E000)>>13);
              }
            }
            else if((regFixed.fix&0x1FFF) > 0x1000)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0x3E000)>>13) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 14 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 14;
                  out.mnt = ((regFixed.fix&0x3E000)>>13) + 1;
                }
              }
              else
              {
                out.exp = 14;
                out.mnt = ((regFixed.fix&0x3E000)>>13);
              }
          }
        else if((regFixed.fix >= 0x20000) && (regFixed.fix < 0x40000))//格式为1X XXXX._XXXX XXXX XXXX
          {
            if((regFixed.fix&0xFFF) == 0x800)//余位为1000...的情况
            {
              if((regFixed.fix&0x1000) == 0x1000)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x1F000)>>12) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 13 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 13;
                  out.mnt = ((regFixed.fix&0x1F000)>>12) + 1;
                }
              }
              else
              {
                out.exp = 13;
                out.mnt = ((regFixed.fix&0x1F000)>>12);
              }
            }
            else if((regFixed.fix&0xFFF) > 0x800)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0x1F000)>>12) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 13 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 13;
                  out.mnt = ((regFixed.fix&0x1F000)>>12) + 1;
                }
              }
              else
              {
                out.exp = 13;
                out.mnt = ((regFixed.fix&0x1F000)>>12);
              }
          }
        else if((regFixed.fix >= 0x10000) && (regFixed.fix < 0x20000))//格式为1 XXXX.X_XXX XXXX XXXX
          {
            if((regFixed.fix&0x7FF) == 0x400)//余位为1000...的情况
            {
              if((regFixed.fix&0x800) == 0x800)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0xF800)>>11) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 12 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 12;
                  out.mnt = ((regFixed.fix&0xF800)>>11) + 1;
                }
              }
              else
              {
                out.exp = 12;
                out.mnt = ((regFixed.fix&0xF800)>>11);
              }
            }
            else if((regFixed.fix&0x7FF) > 0x400)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0xF800)>>11) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 12 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 12;
                  out.mnt = ((regFixed.fix&0xF800)>>11) + 1;
                }
              }
              else
              {
                out.exp = 12;
                out.mnt = ((regFixed.fix&0xF800)>>11);
              }
          }
        else if((regFixed.fix >= 0x8000) && (regFixed.fix < 0x10000))//格式为1XXX.XX_XX XXXX XXXX
          {
            if((regFixed.fix&0x3FF) == 0x200)//余位为1000...的情况
            {
              if((regFixed.fix&0x400) == 0x400)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x7C00)>>10) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 11 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 11;
                  out.mnt = ((regFixed.fix&0x7C00)>>10) + 1;
                }
              }
              else
              {
                out.exp = 11;
                out.mnt = ((regFixed.fix&0x7C00)>>10);
              }
            }
            else if((regFixed.fix&0x3FF) > 0x200)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0x7C00)>>10) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 11 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 11;
                  out.mnt = ((regFixed.fix&0x7C00)>>10) + 1;
                }
              }
              else
              {
                out.exp = 11;
                out.mnt = ((regFixed.fix&0x7C00)>>10);
              }
          }
        else if((regFixed.fix >= 0x4000) && (regFixed.fix < 0x8000))//格式为1XX.XXX_X XXXX XXXX
          {
            if((regFixed.fix&0x1FF) == 0x100)//余位为1000...的情况
            {
              if((regFixed.fix&0x200) == 0x200)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x3E00)>>9) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 10 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 10;
                  out.mnt = ((regFixed.fix&0x3E00)>>9) + 1;
                }
              }
              else
              {
                out.exp = 10;
                out.mnt = ((regFixed.fix&0x3E00)>>9);
              }
            }
            else if((regFixed.fix&0x1FF) > 0x100)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0x3E00)>>9) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 10 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 10;
                  out.mnt = ((regFixed.fix&0x3E00)>>9) + 1;
                }
              }
              else
              {
                out.exp = 10;
                out.mnt = ((regFixed.fix&0x3E00)>>9);
              }
          }
        else if((regFixed.fix >= 0x2000) && (regFixed.fix < 0x4000))//格式为1X.XXXX _XXXX XXXX
          {
            if((regFixed.fix&0xFF) == 0x80)//余位为1000...的情况
            {
              if((regFixed.fix&0x100) == 0x100)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x1F00)>>8) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 9 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 9;
                  out.mnt = ((regFixed.fix&0x1F00)>>8) + 1;
                }
              }
              else
              {
                out.exp = 9;
                out.mnt = ((regFixed.fix&0x1F00)>>8);
              }
            }
            else if((regFixed.fix&0xFF) > 0x80)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0x1F00)>>8) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 9 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 9;
                  out.mnt = ((regFixed.fix&0x1F00)>>8) + 1;
                }
              }
              else
              {
                out.exp = 9;
                out.mnt = ((regFixed.fix&0x1F00)>>8);
              }
          }
        else if((regFixed.fix >= 0x1000) && (regFixed.fix < 0x2000))//格式为1.XXXX X_XXX XXXX
          {
            if((regFixed.fix&0x7F) == 0x40)//余位为1000...的情况
            {
              if((regFixed.fix&0x80) == 0x80)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0xF80)>>7) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 8 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 8;
                  out.mnt = ((regFixed.fix&0xF80)>>7) + 1;
                }
              }
              else
              {
                out.exp = 8;
                out.mnt = ((regFixed.fix&0xF80)>>7);
              }
            }
            else if((regFixed.fix&0x7F) > 0x40)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0xF80)>>7) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 8 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 8;
                  out.mnt = ((regFixed.fix&0xF80)>>7) + 1;
                }
              }
              else
              {
                out.exp = 8;
                out.mnt = ((regFixed.fix&0xF80)>>7);
              }
          }
        else if((regFixed.fix >= 0x800) && (regFixed.fix < 0x1000))//格式为0.1XXX XX_XX XXXX
          {
            if((regFixed.fix&0x3F) == 0x20)//余位为1000...的情况
            {
              if((regFixed.fix&0x40) == 0x40)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x7C0)>>6) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 7 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 7;
                  out.mnt = ((regFixed.fix&0x7C0)>>6) + 1;
                }
              }
              else
              {
                out.exp = 7;
                out.mnt = ((regFixed.fix&0x7C0)>>6);
              }
            }
            else if((regFixed.fix&0x3F) > 0x20)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0x7C0)>>6) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 7 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 7;
                  out.mnt = ((regFixed.fix&0x7C0)>>6) + 1;
                }
              }
              else
              {
                out.exp = 7;
                out.mnt = ((regFixed.fix&0x7C0)>>6);
              }
          }
        else if((regFixed.fix >= 0x400) && (regFixed.fix < 0x800))//格式为0.01XX XXX_X XXXX
          {
            if((regFixed.fix&0x1F) == 0x10)//余位为1000...的情况
            {
              if((regFixed.fix&0x20) == 0x20)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x3E0)>>5) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 6 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 6;
                  out.mnt = ((regFixed.fix&0x3E0)>>5) + 1;
                }
              }
              else
              {
                out.exp = 6;
                out.mnt = ((regFixed.fix&0x3E0)>>5);
              }
            }
            else if((regFixed.fix&0x1F) > 0x10)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0x3E0)>>5) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 6 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 6;
                  out.mnt = ((regFixed.fix&0x3E0)>>5) + 1;
                }
              }
              else
              {
                out.exp = 6;
                out.mnt = ((regFixed.fix&0x3E0)>>5);
              }
          }
        else if((regFixed.fix >= 0x200) && (regFixed.fix < 0x400))//格式为0.001X XXXX _XXXX
          {
            if((regFixed.fix&0xF) == 0x8)//余位为1000...的情况
            {
              if((regFixed.fix&0x10) == 0x10)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x1F0)>>4) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 5 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 5;
                  out.mnt = ((regFixed.fix&0x1F0)>>4) + 1;
                }
              }
              else
              {
                out.exp = 5;
                out.mnt = ((regFixed.fix&0x1F0)>>4);
              }
            }
            else if((regFixed.fix&0xF) > 0x8)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0x1F0)>>4) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 5 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 5;
                  out.mnt = ((regFixed.fix&0x1F0)>>4) + 1;
                }
              }
              else
              {
                out.exp = 5;
                out.mnt = ((regFixed.fix&0x1F0)>>4);
              }
          }
        else if((regFixed.fix >= 0x100) && (regFixed.fix < 0x200))//格式为0.0001 XXXX X_XXX
          {
            if((regFixed.fix&0x7) == 0x4)//余位为1000...的情况
            {
              if((regFixed.fix&0x8) == 0x8)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0xF8)>>3) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 4 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 4;
                  out.mnt = ((regFixed.fix&0xF8)>>3) + 1;
                }
              }
              else
              {
                out.exp = 4;
                out.mnt = ((regFixed.fix&0xF8)>>3);
              }
            }
            else if((regFixed.fix&0x7) > 0x4)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0xF8)>>3) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 4 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 4;
                  out.mnt = ((regFixed.fix&0xF8)>>3) + 1;
                }
              }
              else
              {
                out.exp = 4;
                out.mnt = ((regFixed.fix&0xF8)>>3);
              }
          }
        else if((regFixed.fix >= 0x80) && (regFixed.fix < 0x100))//格式为0.0000 1XXX XX_XX
          {
            if((regFixed.fix&0x3) == 0x2)//余位为1000...的情况
            {
              if((regFixed.fix&0x4) == 0x4)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x7C)>>2) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 3 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 3;
                  out.mnt = ((regFixed.fix&0x7C)>>2) + 1;
                }
              }
              else
              {
                out.exp = 3;
                out.mnt = ((regFixed.fix&0x7C)>>2);
              }
            }
            else if((regFixed.fix&0x3) > 0x2)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0x7C)>>2) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 3 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 3;
                  out.mnt = ((regFixed.fix&0x7C)>>2) + 1;
                }
              }
              else
              {
                out.exp = 3;
                out.mnt = ((regFixed.fix&0x7C)>>2);
              }
          }
        else if((regFixed.fix >= 0x40) && (regFixed.fix < 0x80))//格式为0.0000 01XX XXX_X
          {
            if((regFixed.fix&0x1) == 0x1)//余位为1000...的情况
            {
              if((regFixed.fix&0x2) == 0x2)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x3E)>>1) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 2 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 2;
                  out.mnt = ((regFixed.fix&0x3E)>>1) + 1;
                }
              }
              else
              {
                out.exp = 2;
                out.mnt = ((regFixed.fix&0x3E)>>1);
              }
            }
            else
              {
                out.exp = 2;
                out.mnt = ((regFixed.fix&0x3E)>>1);
              }
          }
        else if((regFixed.fix >= 0x20) && (regFixed.fix < 0x40))//格式为0.0000 001X XXXX_
          {
            out.exp = 1;
            out.mnt = (regFixed.fix&0x1F);
          }
        else//格式为0.0000 0001 XXXX
          {
            out.exp = 1;
            out.mnt = 0;
          }
      }
      }

    else if(type == 3)//type == 3
    {
      reg.sign = (a >= 0)?0:1;//符号位
      reg.integer = (int)abs(a);//整数部分
      Mfraction = abs(a) - reg.integer;//取小数部分
      reg.fraction = 0;
      for(int i = 0; i < 11; ++i)
      {
        reg.fraction  = reg.fraction << 1; 
        if( Mfraction*2 >= 1)
        {
          reg.fraction = reg.fraction + 1;
          Mfraction = Mfraction*2 - 1;
        }
        else
        {
          reg.fraction = reg.fraction + 0;
          Mfraction = Mfraction*2;
        }
      }
      regFixed.fix  = ((reg.fraction&0b1) == 1)?((reg.integer << 10) + (reg.fraction>>1) + 1) : ((reg.integer << 10) + (reg.fraction>>1));
      //首先判断符号位
      if( reg.sign == 1 )
      {
        out.sign = 1;
      }
      else
      {
        out.sign = 0;
      }

      //判断是否超出SFP表示范围
      if(regFixed.fix >= 0x3E000)//格式为11 111X XX.XX XXXX XXXX
      {
        out.exp = 15;
        out.mnt = 7;
      }
      else if(regFixed.fix <= 0x4)//格式为0.00 0000 0100 
      {
        out.exp = 0;
        out.mnt = 0;
      }
      else
      {
        if(regFixed.fix >= 0x20000)//格式为1X XX_XX XX.XX XXXX XXXX
          {
            if((regFixed.fix&0x3FFF) == 0x2000)//余位为1000...的情况
            {
              if((regFixed.fix&0x4000) == 0x4000)//尾数为奇数+1,偶数不进位
              {
                out.exp = 15;
                out.mnt = ((regFixed.fix&0x1C000)>>14) + 1;
              }
              else
              {
                out.exp = 15;
                out.mnt = ((regFixed.fix&0x1C000)>>14);
              }
            }
            else if((regFixed.fix&0x3FFF) > 0x2000)//不是1000...的情况。大于进位，小于舍去
            {
              out.exp = 15;
              out.mnt = ((regFixed.fix&0x1C000)>>14) + 1;
            }
            else
            {
              out.exp = 15;
              out.mnt = ((regFixed.fix&0x1C000)>>14);
            }
          }
        else if((regFixed.fix >= 0x10000) && (regFixed.fix < 0x20000))//格式为1 XXX_X XX.XX XXXX XXXX
          {
            if((regFixed.fix&0x1FFF) == 0x1000)//余位为1000...的情况
            {
              if((regFixed.fix&0x2000) == 0x2000)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0xE000)>>13) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 14 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 14;
                  out.mnt = ((regFixed.fix&0xE000)>>13) + 1;
                }
              }
              else
              {
                out.exp = 14;
                out.mnt = ((regFixed.fix&0xE000)>>13);
              }
            }
            else if((regFixed.fix&0x1FFF) > 0x1000)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0xE000)>>13) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 14 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 14;
                  out.mnt = ((regFixed.fix&0xE000)>>13) + 1;
                }
            }
            else
            {
              out.exp = 14;
              out.mnt = ((regFixed.fix&0xE000)>>13);
            }
          }
        else if((regFixed.fix >= 0x8000) && (regFixed.fix < 0x10000))//格式为1XXX _XX.XX XXXX XXXX
          {
            if((regFixed.fix&0xFFF) == 0x800)//余位为1000...的情况
            {
              if((regFixed.fix&0x1000) == 0x1000)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x7000)>>12) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 13 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 13;
                  out.mnt = ((regFixed.fix&0x7000)>>12) + 1;
                }
              }
              else
              {
                out.exp = 13;
                out.mnt = ((regFixed.fix&0x7000)>>12);
              }
            }
            else if((regFixed.fix&0xFFF) > 0x800)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0x7000)>>12) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 13 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 13;
                  out.mnt = ((regFixed.fix&0x7000)>>12) + 1;
                }
              }
            else
            {
              out.exp = 13;
              out.mnt = ((regFixed.fix&0x7000)>>12);
            }
          }
        else if((regFixed.fix >= 0x4000) && (regFixed.fix < 0x8000))//格式为1XX X_X.XX XXXX XXXX
          {
            if((regFixed.fix&0x7FF) == 0x400)//余位为1000...的情况
            {
              if((regFixed.fix&0x800) == 0x800)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x3800)>>11) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 12 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 12;
                  out.mnt = ((regFixed.fix&0x3800)>>11) + 1;
                }
              }
              else
              {
                out.exp = 12;
                out.mnt = ((regFixed.fix&0x3800)>>11);
              }
            }
            else if((regFixed.fix&0x7FF) > 0x400)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0x3800)>>11) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 12 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 12;
                  out.mnt = ((regFixed.fix&0x3800)>>11) + 1;
                }
            }
            else
            {
              out.exp = 12;
              out.mnt = ((regFixed.fix&0x3800)>>11);
            }
          }
        else if((regFixed.fix >= 0x2000) && (regFixed.fix < 0x4000))//格式为1X XX._XX XXXX XXXX
          {
            if((regFixed.fix&0x3FF) == 0x200)//余位为1000...的情况
            {
              if((regFixed.fix&0x400) == 0x400)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x1C00)>>10) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 11 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 11;
                  out.mnt = ((regFixed.fix&0x1C00)>>10) + 1;
                }
              }
              else
              {
                out.exp = 11;
                out.mnt = ((regFixed.fix&0x1C00)>>10);
              }
            }
            else if((regFixed.fix&0x3FF) > 0x200)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0x1C00)>>10) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 11 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 11;
                  out.mnt = ((regFixed.fix&0x1C00)>>10) + 1;
                }
              }
            else
            {
              out.exp = 11;
              out.mnt = ((regFixed.fix&0x1C00)>>10);
            }
          }
        else if((regFixed.fix >= 0x1000) && (regFixed.fix < 0x2000))//格式为1 XX.X_X XXXX XXXX
          {
            if((regFixed.fix&0x1FF) == 0x100)//余位为1000...的情况
            {
              if((regFixed.fix&0x200) == 0x200)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0xE00)>>9) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 10 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 10;
                  out.mnt = ((regFixed.fix&0xE00)>>9) + 1;
                }
              }
              else
              {
                out.exp = 10;
                out.mnt = ((regFixed.fix&0xE00)>>9);
              }
            }
            else if((regFixed.fix&0x1FF) > 0x100)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0xE00)>>9) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 10 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 10;
                  out.mnt = ((regFixed.fix&0xE00)>>9) + 1;
                }
              }
            else
            {
              out.exp = 10;
              out.mnt = ((regFixed.fix&0xE00)>>9);
            }
          }
        else if((regFixed.fix >= 0x800) && (regFixed.fix < 0x1000))//格式为1X.XX _XXXX XXXX
          {
            if((regFixed.fix&0xFF) == 0x80)//余位为1000...的情况
            {
              if((regFixed.fix&0x100) == 0x100)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x700)>>8) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 9 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 9;
                  out.mnt = ((regFixed.fix&0x700)>>8) + 1;
                }
              }
              else
              {
                out.exp = 9;
                out.mnt = ((regFixed.fix&0x700)>>8);
              }
            }
            else if((regFixed.fix&0xFF) > 0x80)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0x700)>>8) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 9 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 9;
                  out.mnt = ((regFixed.fix&0x700)>>8) + 1;
                }
              }
            else
            {
              out.exp = 9;
              out.mnt = ((regFixed.fix&0x700)>>8);
            }
          }
        else if((regFixed.fix >= 0x400) && (regFixed.fix < 0x800))//格式为1.XX X_XXX XXXX
          {
            if((regFixed.fix&0x7F) == 0x40)//余位为1000...的情况
            {
              if((regFixed.fix&0x80) == 0x80)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x380)>>7) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 8 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 8;
                  out.mnt = ((regFixed.fix&0x380)>>7) + 1;
                }
              }
              else
              {
                out.exp = 8;
                out.mnt = ((regFixed.fix&0x380)>>7);
              }
            }
            else if((regFixed.fix&0x7F) > 0x40)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0x380)>>7) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 8 + 1;
                  out.mnt = 0;
                }
              else
                {
                  out.exp = 8;
                  out.mnt = ((regFixed.fix&0x380)>>7) + 1;
                }
            }
            else
            {
              out.exp = 8;
              out.mnt = ((regFixed.fix&0x380)>>7);
            }
          }
        else if((regFixed.fix >= 0x200) && (regFixed.fix < 0x400))//格式为0.1X XX_XX XXXX
          {
            if((regFixed.fix&0x3F) == 0x20)//余位为1000...的情况
            {
              if((regFixed.fix&0x40) == 0x40)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x1C0)>>6) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 7 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 7;
                  out.mnt = ((regFixed.fix&0x1C0)>>6) + 1;
                }
              }
              else
              {
                out.exp = 7;
                out.mnt = ((regFixed.fix&0x1C0)>>6);
              }
            }
            else if((regFixed.fix&0x3F) > 0x20)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0x1C0)>>6) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 7 + 1;
                  out.mnt = 0;
                }
              else
                {
                  out.exp = 7;
                  out.mnt = ((regFixed.fix&0x1C0)>>6) + 1;
                }
            }
            else
            {
              out.exp = 7;
              out.mnt = ((regFixed.fix&0x1C0)>>6);
            }
          }
        else if((regFixed.fix >= 0x100) && (regFixed.fix < 0x200))//格式为0.01 XXX_X XXXX
          {
            if((regFixed.fix&0x1F) == 0x10)//余位为1000...的情况
            {
              if((regFixed.fix&0x20) == 0x20)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0xE0)>>5) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 6 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 6;
                  out.mnt = ((regFixed.fix&0xE0)>>5) + 1;
                }
              }
              else
              {
                out.exp = 6;
                out.mnt = ((regFixed.fix&0xE0)>>5);
              }
            }
            else if((regFixed.fix&0x1F) > 0x10)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0xE0)>>5) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 6 + 1;
                  out.mnt = 0;
                }
              else
                {
                  out.exp = 6;
                  out.mnt = ((regFixed.fix&0xE0)>>5) + 1;
                }
              }
            else
            {
              out.exp = 6;
              out.mnt = ((regFixed.fix&0xE0)>>5);
            }
          }
        else if((regFixed.fix >= 0x80) && (regFixed.fix < 0x100))//格式为0.00 1XXX _XXXX
          {
            if((regFixed.fix&0xF) == 0x8)//余位为1000...的情况
            {
              if((regFixed.fix&0x10) == 0x10)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x70)>>4) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 5 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 5;
                  out.mnt = ((regFixed.fix&0x70)>>4) + 1;
                }
              }
              else
              {
                out.exp = 5;
                out.mnt = ((regFixed.fix&0x70)>>4);
              }
            }
            else if((regFixed.fix&0xF) > 0x8)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0x70)>>4) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 5 + 1;
                  out.mnt = 0;
                }
              else
                {
                  out.exp = 5;
                  out.mnt = ((regFixed.fix&0x70)>>4) + 1;
                }
              }
            else
            {
              out.exp = 5;
              out.mnt = ((regFixed.fix&0x70)>>4);
            }
          }
        else if((regFixed.fix >= 0x40) && (regFixed.fix < 0x80))//格式为0.00 01XX X_XXX
          {
            if((regFixed.fix&0x7) == 0x4)//余位为1000...的情况
            {
              if((regFixed.fix&0x8) == 0x8)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x38)>>3) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 4 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 4;
                  out.mnt = ((regFixed.fix&0x38)>>3) + 1;
                }
              }
              else
              {
                out.exp = 4;
                out.mnt = ((regFixed.fix&0x38)>>3);
              }
            }
            else if((regFixed.fix&0x7) > 0x4)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0x38)>>3) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 4 + 1;
                  out.mnt = 0;
                }
              else
                {
                  out.exp = 4;
                  out.mnt = ((regFixed.fix&0x38)>>3) + 1;
                }
              }
            else
            {
              out.exp = 4;
              out.mnt = ((regFixed.fix&0x38)>>3);
            }
          }
        else if((regFixed.fix >= 0x20) && (regFixed.fix < 0x40))//格式为0.00 001X XX_XX
          {
            if((regFixed.fix&0x3) == 0x2)//余位为1000...的情况
            {
              if((regFixed.fix&0x4) == 0x4)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x1C)>>2) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 3 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 3;
                  out.mnt = ((regFixed.fix&0x1C)>>2) + 1;
                }
              }
              else
              {
                out.exp = 3;
                out.mnt = ((regFixed.fix&0x1C)>>2);
              }
            }
            else if((regFixed.fix&0x3) > 0x2)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0x1C)>>2) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 3 + 1;
                  out.mnt = 0;
                }
              else
                {
                  out.exp = 3;
                  out.mnt = ((regFixed.fix&0x1C)>>2) + 1;
                }
              }
            else
            {
              out.exp = 3;
              out.mnt = ((regFixed.fix&0x1C)>>2);
            }
          }
        else if((regFixed.fix >= 0x10) && (regFixed.fix < 0x20))//格式为0.00 0001 XXX_X
          {
            if((regFixed.fix&0x1) == 0x1)//余位为1000...的情况
            {
              if((regFixed.fix&0x2) == 0x2)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0xE)>>1) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 2 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 2;
                  out.mnt = ((regFixed.fix&0xE)>>1) + 1;
                }
              }
              else
              {
                out.exp = 2;
                out.mnt = ((regFixed.fix&0xE)>>1);
              }
            }
            else if((regFixed.fix&0x1) > 0x1)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0xE)>>1) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 2 + 1;
                  out.mnt = 0;
                }
              else
                {
                  out.exp = 2;
                  out.mnt = ((regFixed.fix&0xE)>>1) + 1;
                }
              }
            else
            {
              out.exp = 2;
              out.mnt = ((regFixed.fix&0xE)>>1);
            }
          }
        else if((regFixed.fix >= 0x8) && (regFixed.fix < 0x10))//格式为0.00 0000 1XXX
          {
            out.exp = 1;
            out.mnt = (regFixed.fix&0x7);
          }
        else//格式为0.00 0000 01XX
          {
            out.exp = 1;
            out.mnt = 0;
          }
      }
    
    }
    else {
     printf("Wrong Type of float2sfp") ;
    }
    return out;
  }

int24 floattofixed(float a)
  {
    bfixed reg;
    int24 out;
    float Mfraction;

    reg.sign = (a >= 0)?0:1;//符号位
    reg.integer = (int)abs(a);//整数部分
    Mfraction = abs(a) - reg.integer;//取小数部分
    // printf("integer:%d\n",reg.integer);
    // printf("Mf:%f\n",Mfraction);
    reg.fraction = 0;
    for(int i = 0; i < 9; ++i)
    {
      reg.fraction  = reg.fraction << 1; 
      if( Mfraction*2 >= 1)
      {
        reg.fraction = reg.fraction + 1;
        Mfraction = Mfraction*2 - 1;
      }
      else
      {
        reg.fraction = reg.fraction + 0;
        Mfraction = Mfraction*2;
      }
    }
    out.out  = ((reg.fraction&0b1) == 1)?((reg.integer << 8) + (reg.fraction >> 1) + 1):((reg.integer << 8) + (reg.fraction >> 1));
    out.out  = reg.sign?(0xFFFFFF- out.out + 1):out.out;
    return out;
  }

void get_weightSfp(float* weight , sfp *out, float Kw, int size, int type) //modified
  {
      float *weightSfp = (float *)malloc(size*sizeof(float));
      for(int i = 0; i < size; ++i)
      {
        weightSfp[i] = weight[i]/Kw;
        out[i] = floattosfp(weightSfp[i], type);
      }
      free(weightSfp);
  }

void get_biasint24(float* bias , int24 *out, float Kr, int size)  //no change
  {
    float *biasint24 = (float *)malloc(size*sizeof(float));
      for(int i = 0; i < size; ++i)
      {
        biasint24[i] = bias[i]/Kr;
        out[i] = floattofixed(biasint24[i]);
      }
      free(biasint24);
  }