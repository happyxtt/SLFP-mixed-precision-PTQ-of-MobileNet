#include "./common/MobileNetv1.h"

int main(void)
{
  //数据读取
    const char *input_pic = "../output_image/";         //存放二进制文件目录
    const char *weight_filename = "./inputdata/mb_new_weight.txt";            //存放权重
    const char *bias_filename = "./inputdata/mb_new_bias.txt";                //存放权重
    const char *Kw_filename = "./inputdata/Kw_e4m3.txt";
    const char *Kr_filename = "./inputdata/Kr_e4m3.txt";
    char filename[] = "./inputdata/val.txt";        //存放标签
    int picnumb = 50000;                                                //读取数量

  //图像信息
    int inputRow = 224;
    int inputCol = 224;
    int inputChannel = 3;
    int weight_size = 3*3*3*32 + 3*3*32 + 32*64 + 3*3*64 + 64*128 + 3*3*128 + \
    128*128 + 3*3*128 + 128*256 + 3*3*256 + 256*256 + 3*3*256 + 256*512 + \
    (3*3*512 + 512*512)*5 + 3*3*512 + 512*1024 + 3*3*1024 + 1024*1024 + 1024*1000;
    int bias_size = 32*2 + 64*2 + 128*4 + 256*4 + 512*12 + 1024*3 + 1000;
    int *pic_idx = (int *)malloc(picnumb * sizeof(int));
    fileidx_pointer *jpg_data = get_file_idx_pointer(filename ,picnumb , pic_idx);
    const char *bin_file_name = ".bin";
    char filelocate[60];

  //开辟主机端内存
    float *h_weight = get_weight(weight_filename , weight_size);
    float *h_bias = get_bias(bias_filename , bias_size);
    float *Kw = get_Kw(Kw_filename , 28);
    float *Kr = get_Kr(Kr_filename , 28);
    int *h_rank = (int *)malloc(5*sizeof(int));
    uint8_t* h_active = (uint8_t*)malloc(224*224*3);
    sfp   *h_weightSfp = (sfp *)malloc(weight_size*sizeof(sfp));
    int24   *h_biasint24 = (int24 *)malloc(bias_size*sizeof(int24));

  //定义type_in数组      1  2  3  4  5  6  7  8  9 10 11 12 13 14 15 16 17 18 19 20 21 22 23 24 25 26 27 28
    int Type_in[28] =  {5, 5, 5, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5};
    int Type_out[28] = {5, 5, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 5};

  /////test-5- 全低精度，单独把第二层精度提高---///  602/1000
  //  int Type_in[28] =  {5, 5, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 5};
  //  int Type_out[28] = {5, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 5, 5};

  /////test-4- 全低精度，对照测试---///  395/1000
  //  int Type_in[28] =  {5, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 5};
  //  int Type_out[28] = {3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 5, 5};

  /////test-3- 全高精度，对照测试---///  679/1000
  //  int Type_in[28] =  {5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5};
  //  int Type_out[28] = {5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5};

  /////test-2- Dw（偶数层）用高精度，普通卷积用低精度---/// 659/1000
  //  int Type_in[28] =  {5, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5};
  //  int Type_out[28] = {5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 5};

  /////test-1- 按权重平均值，平均值小的用sfp43-------////  643/1000
  //  int Type_in[28] =  {5, 5, 5, 5, 5, 3, 5, 3, 5, 3, 5, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5};
  //  int Type_out[28] = {5, 5, 5, 5, 3, 5, 3, 5, 3, 5, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 3, 5, 5};

  //weight,bias量化
    float *h_weight_conv1 = h_weight;
    float *h_bias_conv1 = h_bias;                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                ;
    sfp *h_weightSfp_conv1 = h_weightSfp;
    int24 *h_biasint24_conv1 = h_biasint24;
    get_weightSfp(h_weight_conv1, h_weightSfp_conv1, Kw[0], 3*3*3*32, Type_in[0]);
    get_biasint24(h_bias_conv1, h_biasint24_conv1, Kr[0], 32);
    float *h_weight_conv2 = h_weight_conv1 + 3*3*3*32;
    float *h_bias_conv2 = h_bias_conv1 + 32;
    sfp *h_weightSfp_conv2 = h_weightSfp_conv1 + 3*3*3*32;
    int24 *h_biasint24_conv2 = h_biasint24_conv1 + 32;
    get_weightSfp(h_weight_conv2, h_weightSfp_conv2, Kw[1], 3*3*32, Type_in[1]);
    get_biasint24(h_bias_conv2, h_biasint24_conv2, Kr[1], 32);
    float *h_weight_conv3 = h_weight_conv2 + 3*3*32;
    float *h_bias_conv3 = h_bias_conv2 + 32;
    sfp *h_weightSfp_conv3 = h_weightSfp_conv2 + 3*3*32;
    int24 *h_biasint24_conv3 = h_biasint24_conv2 + 32;
    get_weightSfp(h_weight_conv3, h_weightSfp_conv3, Kw[2], 32*64, Type_in[2]);
    get_biasint24(h_bias_conv3, h_biasint24_conv3, Kr[2], 64);
    float *h_weight_conv4 = h_weight_conv3 + 32*64;
    float *h_bias_conv4 = h_bias_conv3 + 64;
    sfp *h_weightSfp_conv4 = h_weightSfp_conv3 + 32*64;
    int24 *h_biasint24_conv4 = h_biasint24_conv3 + 64;
    get_weightSfp(h_weight_conv4, h_weightSfp_conv4, Kw[3], 3*3*64, Type_in[3]);
    get_biasint24(h_bias_conv4, h_biasint24_conv4, Kr[3], 64);
    float *h_weight_conv5 = h_weight_conv4 + 3*3*64;
    float *h_bias_conv5 = h_bias_conv4 + 64;
    sfp *h_weightSfp_conv5 = h_weightSfp_conv4 + 3*3*64;
    int24 *h_biasint24_conv5 = h_biasint24_conv4 + 64;
    get_weightSfp(h_weight_conv5, h_weightSfp_conv5, Kw[4], 64*128, Type_in[4]);
    get_biasint24(h_bias_conv5, h_biasint24_conv5, Kr[4], 128);
    float *h_weight_conv6 = h_weight_conv5 + 64*128;
    float *h_bias_conv6 = h_bias_conv5 + 128;
    sfp *h_weightSfp_conv6 = h_weightSfp_conv5 + 64*128;
    int24 *h_biasint24_conv6 = h_biasint24_conv5 + 128;
    get_weightSfp(h_weight_conv6, h_weightSfp_conv6, Kw[5], 3*3*128, Type_in[5]);
    get_biasint24(h_bias_conv6, h_biasint24_conv6, Kr[5], 128);
    float *h_weight_conv7 = h_weight_conv6 + 3*3*128;
    float *h_bias_conv7 = h_bias_conv6 + 128;
    sfp *h_weightSfp_conv7 = h_weightSfp_conv6 + 3*3*128;
    int24 *h_biasint24_conv7 = h_biasint24_conv6 + 128;
    get_weightSfp(h_weight_conv7, h_weightSfp_conv7, Kw[6], 128*128, Type_in[6]);
    get_biasint24(h_bias_conv7, h_biasint24_conv7, Kr[6], 128);
    float *h_weight_conv8 = h_weight_conv7 + 128*128;
    float *h_bias_conv8 = h_bias_conv7 + 128;
    sfp *h_weightSfp_conv8 = h_weightSfp_conv7 + 128*128;
    int24 *h_biasint24_conv8 = h_biasint24_conv7 + 128;
    get_weightSfp(h_weight_conv8, h_weightSfp_conv8, Kw[7], 3*3*128, Type_in[7]);
    get_biasint24(h_bias_conv8, h_biasint24_conv8, Kr[7], 128);
    float *h_weight_conv9 = h_weight_conv8 + 3*3*128;
    float *h_bias_conv9 = h_bias_conv8 + 128;
    sfp *h_weightSfp_conv9 = h_weightSfp_conv8 + 3*3*128;
    int24 *h_biasint24_conv9 = h_biasint24_conv8 + 128;
    get_weightSfp(h_weight_conv9, h_weightSfp_conv9, Kw[8], 128*256, Type_in[8]);
    get_biasint24(h_bias_conv9, h_biasint24_conv9, Kr[8], 256);
    float *h_weight_conv10 = h_weight_conv9 + 128*256;
    float *h_bias_conv10 = h_bias_conv9 + 256;
    sfp *h_weightSfp_conv10 = h_weightSfp_conv9 + 128*256;
    int24 *h_biasint24_conv10 = h_biasint24_conv9 + 256;
    get_weightSfp(h_weight_conv10, h_weightSfp_conv10, Kw[9], 3*3*256, Type_in[9]);
    get_biasint24(h_bias_conv10, h_biasint24_conv10, Kr[9], 256);
    float *h_weight_conv11 = h_weight_conv10 + 3*3*256;
    float *h_bias_conv11 = h_bias_conv10 + 256;
    sfp *h_weightSfp_conv11 = h_weightSfp_conv10 + 3*3*256;
    int24 *h_biasint24_conv11 = h_biasint24_conv10+ 256;
    get_weightSfp(h_weight_conv11, h_weightSfp_conv11, Kw[10], 256*256, Type_in[10]);
    get_biasint24(h_bias_conv11, h_biasint24_conv11, Kr[10], 256);
    float *h_weight_conv12 = h_weight_conv11 + 256*256;
    float *h_bias_conv12 = h_bias_conv11 + 256;
    sfp *h_weightSfp_conv12 = h_weightSfp_conv11 + 256*256;
    int24 *h_biasint24_conv12 = h_biasint24_conv11 + 256;
    get_weightSfp(h_weight_conv12, h_weightSfp_conv12, Kw[11], 3*3*256, Type_in[11]);
    get_biasint24(h_bias_conv12, h_biasint24_conv12, Kr[11], 256);
    float *h_weight_conv13 = h_weight_conv12 + 3*3*256;
    float *h_bias_conv13 = h_bias_conv12 + 256;
    sfp *h_weightSfp_conv13 = h_weightSfp_conv12 + 3*3*256;
    int24 *h_biasint24_conv13 = h_biasint24_conv12 + 256;
    get_weightSfp(h_weight_conv13, h_weightSfp_conv13, Kw[12], 256*512, Type_in[12]);
    get_biasint24(h_bias_conv13, h_biasint24_conv13, Kr[12], 512);
    float *h_weight_conv14 = h_weight_conv13 + 256*512;
    float *h_bias_conv14 = h_bias_conv13 + 512;
    sfp *h_weightSfp_conv14 = h_weightSfp_conv13 + 256*512;
    int24 *h_biasint24_conv14 = h_biasint24_conv13 + 512;
    get_weightSfp(h_weight_conv14, h_weightSfp_conv14, Kw[13], 3*3*512, Type_in[13]);
    get_biasint24(h_bias_conv14, h_biasint24_conv14, Kr[13], 512);
    float *h_weight_conv15 = h_weight_conv14 + 3*3*512;
    float *h_bias_conv15 = h_bias_conv14 + 512;
    sfp *h_weightSfp_conv15 = h_weightSfp_conv14 + 3*3*512;
    int24 *h_biasint24_conv15 = h_biasint24_conv14 + 512;
    get_weightSfp(h_weight_conv15, h_weightSfp_conv15, Kw[14], 512*512, Type_in[14]);
    get_biasint24(h_bias_conv15, h_biasint24_conv15, Kr[14], 512);
    float *h_weight_conv16 = h_weight_conv15 + 512*512;
    float *h_bias_conv16 = h_bias_conv15 + 512;
    sfp *h_weightSfp_conv16 = h_weightSfp_conv15 + 512*512;
    int24 *h_biasint24_conv16 = h_biasint24_conv15 + 512;
    get_weightSfp(h_weight_conv16, h_weightSfp_conv16, Kw[15], 3*3*512, Type_in[15]);
    get_biasint24(h_bias_conv16, h_biasint24_conv16, Kr[15], 512);
    float *h_weight_conv17 = h_weight_conv16 + 3*3*512;
    float *h_bias_conv17 = h_bias_conv16 + 512;
    sfp *h_weightSfp_conv17 = h_weightSfp_conv16 + 3*3*512;
    int24 *h_biasint24_conv17 = h_biasint24_conv16 + 512;
    get_weightSfp(h_weight_conv17, h_weightSfp_conv17, Kw[16], 512*512, Type_in[16]);
    get_biasint24(h_bias_conv17, h_biasint24_conv17, Kr[16], 512);
    float *h_weight_conv18 = h_weight_conv17 + 512*512;
    float *h_bias_conv18 = h_bias_conv17 + 512;
    sfp *h_weightSfp_conv18 = h_weightSfp_conv17 + 512*512;
    int24 *h_biasint24_conv18 = h_biasint24_conv17 + 512;
    get_weightSfp(h_weight_conv18, h_weightSfp_conv18, Kw[17], 3*3*512, Type_in[17]);
    get_biasint24(h_bias_conv18, h_biasint24_conv18, Kr[17], 512);
    float *h_weight_conv19 = h_weight_conv18 + 3*3*512;
    float *h_bias_conv19 = h_bias_conv18 + 512;
    sfp *h_weightSfp_conv19 = h_weightSfp_conv18 + 3*3*512;
    int24 *h_biasint24_conv19 = h_biasint24_conv18 + 512;
    get_weightSfp(h_weight_conv19, h_weightSfp_conv19, Kw[18], 512*512, Type_in[18]);
    get_biasint24(h_bias_conv19, h_biasint24_conv19, Kr[18], 512);
    float *h_weight_conv20 = h_weight_conv19 + 512*512;
    float *h_bias_conv20 = h_bias_conv19 + 512;
    sfp *h_weightSfp_conv20 = h_weightSfp_conv19 + 512*512;
    int24 *h_biasint24_conv20 = h_biasint24_conv19 + 512;
    get_weightSfp(h_weight_conv20, h_weightSfp_conv20, Kw[19], 3*3*512, Type_in[19]);
    get_biasint24(h_bias_conv20, h_biasint24_conv20, Kr[19], 512);
    float *h_weight_conv21 = h_weight_conv20 + 3*3*512;
    float *h_bias_conv21 = h_bias_conv20 + 512;
    sfp *h_weightSfp_conv21 = h_weightSfp_conv20 + 3*3*512;
    int24 *h_biasint24_conv21 = h_biasint24_conv20 + 512;
    get_weightSfp(h_weight_conv21, h_weightSfp_conv21, Kw[20], 512*512, Type_in[20]);
    get_biasint24(h_bias_conv21, h_biasint24_conv21, Kr[20], 512);
    float *h_weight_conv22 = h_weight_conv21 + 512*512;
    float *h_bias_conv22 = h_bias_conv21 + 512;
    sfp *h_weightSfp_conv22 = h_weightSfp_conv21 + 512*512;
    int24 *h_biasint24_conv22 = h_biasint24_conv21 + 512;
    get_weightSfp(h_weight_conv22, h_weightSfp_conv22, Kw[21], 3*3*512, Type_in[21]);
    get_biasint24(h_bias_conv22, h_biasint24_conv22, Kr[21], 512);
    float *h_weight_conv23 = h_weight_conv22 + 3*3*512;
    float *h_bias_conv23 = h_bias_conv22 + 512;
    sfp *h_weightSfp_conv23 = h_weightSfp_conv22 + 3*3*512;
    int24 *h_biasint24_conv23 = h_biasint24_conv22 + 512;
    get_weightSfp(h_weight_conv23, h_weightSfp_conv23, Kw[22], 512*512, Type_in[22]);
    get_biasint24(h_bias_conv23, h_biasint24_conv23, Kr[22], 512);
    float *h_weight_conv24 = h_weight_conv23 + 512*512;
    float *h_bias_conv24 = h_bias_conv23 + 512;
    sfp *h_weightSfp_conv24 = h_weightSfp_conv23 + 512*512;
    int24 *h_biasint24_conv24 = h_biasint24_conv23 + 512;
    get_weightSfp(h_weight_conv24, h_weightSfp_conv24, Kw[23], 3*3*512, Type_in[23]);
    get_biasint24(h_bias_conv24, h_biasint24_conv24, Kr[23], 512);
    float *h_weight_conv25 = h_weight_conv24 + 3*3*512;
    float *h_bias_conv25 = h_bias_conv24 + 512;
    sfp *h_weightSfp_conv25 = h_weightSfp_conv24 + 3*3*512;
    int24 *h_biasint24_conv25 = h_biasint24_conv24 + 512;
    get_weightSfp(h_weight_conv25, h_weightSfp_conv25, Kw[24], 512*1024, Type_in[24]);
    get_biasint24(h_bias_conv25, h_biasint24_conv25, Kr[24], 1024);
    float *h_weight_conv26 = h_weight_conv25 + 512*1024;
    float *h_bias_conv26 = h_bias_conv25 + 1024;
    sfp *h_weightSfp_conv26 = h_weightSfp_conv25 + 512*1024;
    int24 *h_biasint24_conv26 = h_biasint24_conv25 + 1024;
    get_weightSfp(h_weight_conv26, h_weightSfp_conv26, Kw[25], 3*3*1024, Type_in[25]);
    get_biasint24(h_bias_conv26, h_biasint24_conv26, Kr[25], 1024);
    float *h_weight_conv27 = h_weight_conv26 + 3*3*1024;
    float *h_bias_conv27 = h_bias_conv26 + 1024;
    sfp *h_weightSfp_conv27 = h_weightSfp_conv26 + 3*3*1024;
    int24 *h_biasint24_conv27 = h_biasint24_conv26 + 1024;
    get_weightSfp(h_weight_conv27, h_weightSfp_conv27, Kw[26], 1024*1024, Type_in[26]);
    get_biasint24(h_bias_conv27, h_biasint24_conv27, Kr[26], 1024);
    float *h_weight_conv28 = h_weight_conv27 + 1024*1024;
    float *h_bias_conv28 = h_bias_conv27 + 1024;
    sfp *h_weightSfp_conv28 = h_weightSfp_conv27 + 1024*1024;
    int24 *h_biasint24_conv28 = h_biasint24_conv27 + 1024;
    get_weightSfp(h_weight_conv28, h_weightSfp_conv28, Kw[27], 1024*1000, Type_in[27]);
    get_biasint24(h_bias_conv28, h_biasint24_conv28, Kr[27], 1000);
  //开辟设备端内存 
    uint8_t *d_active = NULL;
    (hipMalloc((void**) &d_active, inputRow*inputCol*inputChannel));
    sfp *d_weight = NULL;
    (hipMalloc((void**) &d_weight, weight_size*sizeof(sfp)));
    float *d_bias = NULL;
    (hipMalloc((void**) &d_bias, bias_size*sizeof(int24)));
    int *d_pic_idx = NULL;
    (hipMalloc((void**) &d_pic_idx, picnumb*sizeof(int)));
    sfp *d_image = NULL;
    (hipMalloc((void**) &d_image, 224*224*3*sizeof(sfp)));



    (hipMemcpy(d_weight, h_weightSfp, weight_size*sizeof(sfp), hipMemcpyHostToDevice));
    (hipMemcpy(d_bias, h_bias, bias_size*sizeof(float), hipMemcpyHostToDevice));
    (hipMemcpy(d_pic_idx , pic_idx , picnumb*sizeof(int) , hipMemcpyHostToDevice));


    sfp *d_out_net1 = NULL;
    sfp *d_weight_net1 = d_weight;
    float *d_bias_net1 = d_bias;
    (hipMalloc((void**) &d_out_net1, 112*112*32*sizeof(sfp)));
    sfp *d_out_net2 = NULL;
    sfp *d_weight_net2 = d_weight_net1 + 3*3*3*32;
    float *d_bias_net2 = d_bias_net1 + 32;
    (hipMalloc((void**) &d_out_net2, 112*112*32*sizeof(sfp)));
    sfp *d_out_net3 = NULL;
    sfp *d_weight_net3 = d_weight_net2 + 3*3*32;
    float *d_bias_net3 = d_bias_net2 + 32;
    (hipMalloc((void**) &d_out_net3, 112*112*64*sizeof(sfp)));
    sfp *d_out_net4 = NULL;
    sfp *d_weight_net4 = d_weight_net3 + 32*64;
    float *d_bias_net4 = d_bias_net3 + 64;
    (hipMalloc((void**) &d_out_net4, 56*56*64*sizeof(sfp)));
    sfp *d_out_net5 = NULL;
    sfp *d_weight_net5 = d_weight_net4 + 3*3*64;
    float *d_bias_net5 = d_bias_net4 + 64;
    (hipMalloc((void**) &d_out_net5, 56*56*128*sizeof(sfp)));
    sfp *d_out_net6 = NULL;
    sfp *d_weight_net6 = d_weight_net5 + 64*128;
    float *d_bias_net6 = d_bias_net5 + 128;
    (hipMalloc((void**) &d_out_net6, 56*56*128*sizeof(sfp)));
    sfp *d_out_net7 = NULL;
    sfp *d_weight_net7 = d_weight_net6 + 3*3*128;
    float *d_bias_net7 = d_bias_net6 + 128;
    (hipMalloc((void**) &d_out_net7, 56*56*128*sizeof(sfp)));
    sfp *d_out_net8 = NULL;
    sfp *d_weight_net8 = d_weight_net7 + 128*128;
    float *d_bias_net8 = d_bias_net7 + 128;
    (hipMalloc((void**) &d_out_net8, 28*28*128*sizeof(sfp)));
    sfp *d_out_net9 = NULL;
    sfp *d_weight_net9 = d_weight_net8 + 3*3*128;
    float *d_bias_net9 = d_bias_net8 + 128;
    (hipMalloc((void**) &d_out_net9, 28*28*256*sizeof(sfp)));
    sfp *d_out_net10 = NULL;
    sfp *d_weight_net10 = d_weight_net9 + 128*256;
    float *d_bias_net10 = d_bias_net9 + 256;
    (hipMalloc((void**) &d_out_net10, 28*28*256*sizeof(sfp)));
    sfp *d_out_net11 = NULL;
    sfp *d_weight_net11 = d_weight_net10 + 3*3*256;
    float *d_bias_net11 = d_bias_net10 + 256;
    (hipMalloc((void**) &d_out_net11, 28*28*256*sizeof(sfp)));
    sfp *d_out_net12 = NULL;
    sfp *d_weight_net12 = d_weight_net11 + 256*256;
    float *d_bias_net12 = d_bias_net11 + 256;
    (hipMalloc((void**) &d_out_net12, 14*14*256*sizeof(sfp)));
    sfp *d_out_net13 = NULL;
    sfp *d_weight_net13 = d_weight_net12 + 3*3*256;
    float *d_bias_net13 = d_bias_net12 + 256;
    (hipMalloc((void**) &d_out_net13, 14*14*512*sizeof(sfp)));
    sfp *d_out_net14 = NULL;
    sfp *d_weight_net14 = d_weight_net13 + 256*512;
    float *d_bias_net14 = d_bias_net13 + 512;
    (hipMalloc((void**) &d_out_net14, 14*14*512*sizeof(sfp)));
    sfp *d_out_net15 = NULL;
    sfp *d_weight_net15 = d_weight_net14 + 3*3*512;
    float *d_bias_net15 = d_bias_net14 + 512;
    (hipMalloc((void**) &d_out_net15, 14*14*512*sizeof(sfp)));
    sfp *d_out_net16 = NULL;
    sfp *d_weight_net16 = d_weight_net15 + 512*512;
    float *d_bias_net16 = d_bias_net15 + 512;
    (hipMalloc((void**) &d_out_net16, 14*14*512*sizeof(sfp)));
    sfp *d_out_net17 = NULL;
    sfp *d_weight_net17 = d_weight_net16 + 3*3*512;
    float *d_bias_net17 = d_bias_net16 + 512;
    (hipMalloc((void**) &d_out_net17, 14*14*512*sizeof(sfp)));
    sfp *d_out_net18 = NULL;
    sfp *d_weight_net18 = d_weight_net17 + 512*512;
    float *d_bias_net18 = d_bias_net17 + 512;
    (hipMalloc((void**) &d_out_net18, 14*14*512*sizeof(sfp)));
    sfp *d_out_net19 = NULL;
    sfp *d_weight_net19 = d_weight_net18 + 3*3*512;
    float *d_bias_net19 = d_bias_net18 + 512;
    (hipMalloc((void**) &d_out_net19, 14*14*512*sizeof(sfp)));
    sfp *d_out_net20 = NULL;
    sfp *d_weight_net20 = d_weight_net19 + 512*512;
    float *d_bias_net20 = d_bias_net19 + 512;
    (hipMalloc((void**) &d_out_net20, 14*14*512*sizeof(sfp)));
    sfp *d_out_net21 = NULL;
    sfp *d_weight_net21 = d_weight_net20 + 3*3*512;
    float *d_bias_net21 = d_bias_net20 + 512;
    (hipMalloc((void**) &d_out_net21, 14*14*512*sizeof(sfp)));
    sfp *d_out_net22 = NULL;
    sfp *d_weight_net22 = d_weight_net21 + 512*512;
    float *d_bias_net22 = d_bias_net21 + 512;
    (hipMalloc((void**) &d_out_net22, 14*14*512*sizeof(sfp)));
    sfp *d_out_net23 = NULL;
    sfp *d_weight_net23 = d_weight_net22 + 3*3*512;
    float *d_bias_net23 = d_bias_net22 + 512;
    (hipMalloc((void**) &d_out_net23, 14*14*512*sizeof(sfp)));
    sfp *d_out_net24 = NULL;
    sfp *d_weight_net24 = d_weight_net23 + 512*512;
    float *d_bias_net24 = d_bias_net23 + 512;
    (hipMalloc((void**) &d_out_net24, 7*7*512*sizeof(sfp)));
    sfp *d_out_net25 = NULL;
    sfp *d_weight_net25 = d_weight_net24 + 3*3*512;
    float *d_bias_net25 = d_bias_net24 + 512;
    (hipMalloc((void**) &d_out_net25, 7*7*1024*sizeof(sfp)));
    sfp *d_out_net26 = NULL;
    sfp *d_weight_net26 = d_weight_net25 + 512*1024;
    float *d_bias_net26 = d_bias_net25 + 1024;
    (hipMalloc((void**) &d_out_net26, 7*7*1024*sizeof(sfp)));
    sfp *d_out_net27 = NULL;
    sfp *d_weight_net27 = d_weight_net26 + 3*3*1024;
    float *d_bias_net27 = d_bias_net26 + 1024;
    (hipMalloc((void**) &d_out_net27, 7*7*1024*sizeof(sfp)));
    sfp *d_out_avg = NULL;
    (hipMalloc((void**) &d_out_avg, 1024*sizeof(sfp)));
    sfp *d_out_fc = NULL;
    sfp *d_weight_fc = d_weight_net27 + 1024*1024;
    float *d_bias_fc = d_bias_net27 + 1024;
    (hipMalloc((void**) &d_out_fc, 1000*sizeof(sfp)));
    int *d_rank = NULL;
    (hipMalloc((void**) &d_rank , 5*sizeof(int)));
    int *d_TOP1 = NULL;
    (hipMalloc((void**) &d_TOP1 , sizeof(int)));
    int *d_TOP5 = NULL;
    (hipMalloc((void**) &d_TOP5 , sizeof(int)));
    float *outint24 = NULL;
    (hipMalloc((void**) &outint24 , 112*112*64*sizeof(float)));


    sfp *test = (sfp *)malloc(112*112*64*sizeof(sfp));
    float Ka = 0.003960;
  //计算卷积，卷积开始
//  for(int l = 0 ; l < 27; ++l)
//   {
//     printf("[%d]:%d\n", l , Type_in[l]);
//   }

    for (int i = 0 ; i < 1000 ; ++i)
    {
      strcpy(filelocate , input_pic);
      strcat(filelocate , jpg_data[i].jpgname);
      strcat(filelocate , bin_file_name);
      get_parameter(filelocate , h_active);
      (hipMemcpy(d_active, h_active, inputRow*inputCol*inputChannel*sizeof(uint8_t) , hipMemcpyHostToDevice));
      image   <<<dim3(224 , 3),dim3(224)>>>(d_active , d_image , 224, Ka);  //sfp45
      conv2   <<<dim3(112 , 32),dim3(112)>>>(d_image , d_weight_net1 , d_bias_net1 , d_out_net1 , outint24 , 224 , 112 , 3, Ka, Kw[0], Kr[0], Type_in[0], Type_out[0]);  //layer-1
      conv_DW <<<dim3(112 , 32),dim3(112)>>>(d_out_net1 , d_weight_net2 , d_bias_net2 , d_out_net2 , outint24 , 112 , 112 , 1, Kr[0], Kw[1], Kr[1], Type_in[1], Type_out[1]);  //2
      conv1   <<<dim3(112 , 64),dim3(112)>>>(d_out_net2 , d_weight_net3 , d_bias_net3 , d_out_net3 , outint24 , 112 , 112 , 32, Kr[1], Kw[2], Kr[2],Type_in[2], Type_out[2]);  //3
      conv_DW <<<dim3(56 , 64),dim3(56)>>>(d_out_net3 , d_weight_net4 , d_bias_net4 , d_out_net4 , outint24 , 112 , 56 , 2, Kr[2], Kw[3], Kr[3],Type_in[3], Type_out[3]);      //4
      conv1   <<<dim3(56 , 128),dim3(56)>>>(d_out_net4 , d_weight_net5 , d_bias_net5 , d_out_net5 , outint24 , 56 , 56 , 64, Kr[3], Kw[4], Kr[4],Type_in[4], Type_out[4]);     //5
      conv_DW <<<dim3(56 , 128),dim3(56)>>>(d_out_net5 , d_weight_net6 , d_bias_net6 , d_out_net6 , outint24 , 56 , 56 , 1, Kr[4], Kw[5], Kr[5],Type_in[5], Type_out[5]);      //6
      conv1   <<<dim3(56 , 128),dim3(56)>>>(d_out_net6 , d_weight_net7 , d_bias_net7 , d_out_net7 , outint24 , 56 , 56 , 128, Kr[5], Kw[6], Kr[6],Type_in[6], Type_out[6]);    //7
      conv_DW <<<dim3(28 , 128),dim3(28)>>>(d_out_net7 , d_weight_net8 , d_bias_net8 , d_out_net8 , outint24 , 56 , 28 , 2, Kr[6], Kw[7], Kr[7],Type_in[7], Type_out[7]);      //8
      conv1   <<<dim3(28 , 256),dim3(28)>>>(d_out_net8 , d_weight_net9 , d_bias_net9 , d_out_net9 , outint24 , 28 , 28 , 128, Kr[7], Kw[8], Kr[8],Type_in[8], Type_out[8]);    //9
      conv_DW <<<dim3(28 , 256),dim3(28)>>>(d_out_net9 , d_weight_net10 , d_bias_net10 , d_out_net10 , outint24 , 28 , 28 , 1, Kr[8], Kw[9], Kr[9],Type_in[9], Type_out[9]);   //10
      conv1   <<<dim3(28 , 256),dim3(28)>>>(d_out_net10 , d_weight_net11 , d_bias_net11 , d_out_net11 , outint24 , 28 , 28 , 256, Kr[9], Kw[10], Kr[10], Type_in[10], Type_out[10]); //11
      conv_DW <<<dim3(14 , 256),dim3(14)>>>(d_out_net11 , d_weight_net12 , d_bias_net12 , d_out_net12 , outint24 , 28 , 14 , 2, Kr[10], Kw[11], Kr[11], Type_in[11], Type_out[11]);  //12
      conv1   <<<dim3(14 , 512),dim3(14)>>>(d_out_net12 , d_weight_net13 , d_bias_net13 , d_out_net13 , outint24 , 14 , 14 , 256, Kr[11], Kw[12], Kr[12], Type_in[12], Type_out[12]);//13
      conv_DW <<<dim3(14 , 512),dim3(14)>>>(d_out_net13 , d_weight_net14 , d_bias_net14 , d_out_net14 , outint24 , 14 , 14 , 1, Kr[12], Kw[13], Kr[13], Type_in[13], Type_out[13]);  //14
      conv1   <<<dim3(14 , 512),dim3(14)>>>(d_out_net14 , d_weight_net15 , d_bias_net15 , d_out_net15 , outint24 , 14 , 14 , 512, Kr[13], Kw[14], Kr[14], Type_in[14], Type_out[14]);//15
      conv_DW <<<dim3(14 , 512),dim3(14)>>>(d_out_net15 , d_weight_net16 , d_bias_net16 , d_out_net16 , outint24 , 14 , 14 , 1, Kr[14], Kw[15], Kr[15], Type_in[15], Type_out[15]);  //16
      conv1   <<<dim3(14 , 512),dim3(14)>>>(d_out_net16 , d_weight_net17 , d_bias_net17 , d_out_net17 , outint24 , 14 , 14 , 512, Kr[15], Kw[16], Kr[16], Type_in[16], Type_out[16]);//17
      conv_DW <<<dim3(14 , 512),dim3(14)>>>(d_out_net17 , d_weight_net18 , d_bias_net18 , d_out_net18 , outint24 , 14 , 14 , 1, Kr[16], Kw[17], Kr[17], Type_in[17], Type_out[17]);  //18
      conv1   <<<dim3(14 , 512),dim3(14)>>>(d_out_net18 , d_weight_net19 , d_bias_net19 , d_out_net19 , outint24 , 14 , 14 , 512, Kr[17], Kw[18], Kr[18], Type_in[18], Type_out[18]);//19
      conv_DW <<<dim3(14 , 512),dim3(14)>>>(d_out_net19 , d_weight_net20 , d_bias_net20 , d_out_net20 , outint24 , 14 , 14 , 1, Kr[18], Kw[19], Kr[19], Type_in[19], Type_out[19]);  //20
      conv1   <<<dim3(14 , 512),dim3(14)>>>(d_out_net20 , d_weight_net21 , d_bias_net21 , d_out_net21 , outint24 , 14 , 14 , 512, Kr[19], Kw[20], Kr[20], Type_in[20], Type_out[20]);//21
      conv_DW <<<dim3(14 , 512),dim3(14)>>>(d_out_net21 , d_weight_net22 , d_bias_net22 , d_out_net22 , outint24 , 14 , 14 , 1, Kr[20], Kw[21], Kr[21], Type_in[21], Type_out[21]);  //22
      conv1   <<<dim3(14 , 512),dim3(14)>>>(d_out_net22 , d_weight_net23 , d_bias_net23 , d_out_net23 , outint24 , 14 , 14 , 512, Kr[21], Kw[22], Kr[22], Type_in[22], Type_out[22]);//23
      conv_DW <<<dim3(7 , 512),dim3(7)>>>(d_out_net23 , d_weight_net24 , d_bias_net24 , d_out_net24 , outint24 , 14 , 7 , 2, Kr[22], Kw[23], Kr[23], Type_in[23], Type_out[23]);     //24
      conv1   <<<dim3(7 , 1024),dim3(7)>>>(d_out_net24 , d_weight_net25 , d_bias_net25 , d_out_net25 , outint24 , 7 , 7 , 512, Kr[23], Kw[24], Kr[24], Type_in[24], Type_out[24]);   //25
      conv_DW <<<dim3(7 , 1024),dim3(7)>>>(d_out_net25 , d_weight_net26 , d_bias_net26 , d_out_net26 , outint24 , 7 , 7 , 1, Kr[24], Kw[25], Kr[25], Type_in[25], Type_out[25]);     //26
      conv1   <<<dim3(7 , 1024),dim3(7)>>>(d_out_net26 , d_weight_net27 , d_bias_net27 , d_out_net27 , outint24 , 7 , 7 , 1024, Kr[25], Kw[26], Kr[26], Type_in[26], Type_out[26]);  //27
      avgpool <<<dim3(1) , dim3(1024)>>>(d_out_net27 , d_out_avg , 7);
      fullconnection <<<dim3(1) , dim3(1024)>>>(d_out_avg , d_weight_fc , d_bias_fc , d_out_fc, outint24 , Kr[26], Kw[27], Kr[27]);              // 第28层
      BubbleSort <<<dim3(1) , dim3(1)>>>(d_out_fc , d_rank , d_TOP1 , d_TOP5 , d_pic_idx , i , picnumb, Kr[27]);
    }
    // hipMemcpy(test, d_out_net2,112*112*32*sizeof(sfp), hipMemcpyDeviceToHost);
    // for(int l = 0 ; l < 112*112*32; ++l)
    // {
    //   // printf("[%d]:%f\n",l,Kr[1]*sfpfloat(test[l]));
    // }
    
  // Free device global memory
    (hipFree(d_out_net1));
    (hipFree(d_out_net2));
    (hipFree(d_out_net3));
    (hipFree(d_out_net4));
    (hipFree(d_out_net5));
    (hipFree(d_out_net6));
    (hipFree(d_out_net7));
    (hipFree(d_out_net8));
    (hipFree(d_out_net9));
    (hipFree(d_out_net10));
    (hipFree(d_out_net11));
    (hipFree(d_out_net12));
    (hipFree(d_out_net13));
    (hipFree(d_out_net14));
    (hipFree(d_out_net15));
    (hipFree(d_out_net16));
    (hipFree(d_out_net17));
    (hipFree(d_out_net18));
    (hipFree(d_out_net19));
    (hipFree(d_out_net20));
    (hipFree(d_out_net21));
    (hipFree(d_out_net22));
    (hipFree(d_out_net23));
    (hipFree(d_out_net24));
    (hipFree(d_out_net25));
    (hipFree(d_out_net26));
    (hipFree(d_out_net27));
    (hipFree(d_out_avg));
    (hipFree(d_rank));
    (hipFree(d_active));
    (hipFree(d_weight));
    (hipFree(d_pic_idx));
    (hipFree(d_TOP1));
    (hipFree(d_TOP5));

    // Free host memory
    free(h_active);
    free(h_weight);
    free(h_rank);
    free(jpg_data);
    free(pic_idx);
    hipDeviceReset();  
    return 0;
}

