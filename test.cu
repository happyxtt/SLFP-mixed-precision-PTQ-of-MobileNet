#include "hip/hip_runtime.h"
#include "./common/MobileNetv1.h"
#include <iostream>

using namespace std;
using std::cout;

sfp floatsfp(float a, int type)  //finished
  {
    myfixed reg;
    myfixed regFixed;
    sfp out;
    float Mfraction;
    if(type == 5)
    {
      reg.sign = (a >= 0)?0:1;//符号位
      reg.integer = (int)abs(a);//整数部分
      Mfraction = abs(a) - reg.integer;//取小数部分
      reg.fraction = 0;
      for(int i = 0; i < 13; ++i)
      {
        reg.fraction  = reg.fraction << 1; 
        if( Mfraction*2 >= 1)
        {
          reg.fraction = reg.fraction + 1;
          Mfraction = Mfraction*2 - 1;
        }
        else
        {
          reg.fraction = reg.fraction + 0;
          Mfraction = Mfraction*2;
        }
      }
      regFixed.fix  = ((reg.fraction&0b1) == 1)?((reg.integer << 12) + (reg.fraction>>1) + 1) : ((reg.integer << 12) + (reg.fraction>>1));
      //首先判断符号位
      if(reg.sign == 1)
      {
        out.sign = 1;
      }
      else
      {
        out.sign = 0;
      }

      //判断是否超出SFP表示范围
      if(regFixed.fix >= 0xFE000)//格式为1111 11_1X.XXXX XXXX XXXX
      {
        out.exp = 15;
        out.mnt = 31;
      }
      else if(regFixed.fix <= 0x10)//格式为0.0000 0001 0000 
      {
        out.exp = 0;
        out.mnt = 0;
      }
      else
      {
        if(regFixed.fix >= 0x80000)//格式为1XXX XX_XX.XXXX XXXX XXXX
          {
            if((regFixed.fix&0x3FFF) == 0x2000)//余位为1000...的情况
            {
              if((regFixed.fix&0x4000) == 0x4000)//尾数为奇数+1,偶数不进位
              {
                out.exp = 15;
                out.mnt = ((regFixed.fix&0x7C000)>>14) + 1;
              }
              else
              {
                out.exp = 15;
                out.mnt = ((regFixed.fix&0x7C000)>>14);
              }
            }
            else if((regFixed.fix&0x3FFF) > 0x2000)//不是1000...的情况。大于进位，小于舍去
            {
              out.exp = 15;
              out.mnt = ((regFixed.fix&0x7C000)>>14) + 1;
            }
            else
            {
              out.exp = 15;
              out.mnt = ((regFixed.fix&0x7C000)>>14);
            }
          }
        else if((regFixed.fix >= 0x40000) && (regFixed.fix < 0x80000))//格式为1XX XXX_X.XXXX XXXX XXXX
          {
            if((regFixed.fix&0x1FFF) == 0x1000)//余位为1000...的情况
            {
              if((regFixed.fix&0x2000) == 0x2000)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x3E000)>>13) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 14 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 14;
                  out.mnt = ((regFixed.fix&0x3E000)>>13) + 1;
                }
              }
              else
              {
                out.exp = 14;
                out.mnt = ((regFixed.fix&0x3E000)>>13);
              }
            }
            else if((regFixed.fix&0x1FFF) > 0x1000)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0x3E000)>>13) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 14 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 14;
                  out.mnt = ((regFixed.fix&0x3E000)>>13) + 1;
                }
              }
              else
              {
                out.exp = 14;
                out.mnt = ((regFixed.fix&0x3E000)>>13);
              }
          }
        else if((regFixed.fix >= 0x20000) && (regFixed.fix < 0x40000))//格式为1X XXXX._XXXX XXXX XXXX
          {
            if((regFixed.fix&0xFFF) == 0x800)//余位为1000...的情况
            {
              if((regFixed.fix&0x1000) == 0x1000)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x1F000)>>12) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 13 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 13;
                  out.mnt = ((regFixed.fix&0x1F000)>>12) + 1;
                }
              }
              else
              {
                out.exp = 13;
                out.mnt = ((regFixed.fix&0x1F000)>>12);
              }
            }
            else if((regFixed.fix&0xFFF) > 0x800)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0x1F000)>>12) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 13 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 13;
                  out.mnt = ((regFixed.fix&0x1F000)>>12) + 1;
                }
              }
              else
              {
                out.exp = 13;
                out.mnt = ((regFixed.fix&0x1F000)>>12);
              }
          }
        else if((regFixed.fix >= 0x10000) && (regFixed.fix < 0x20000))//格式为1 XXXX.X_XXX XXXX XXXX
          {
            if((regFixed.fix&0x7FF) == 0x400)//余位为1000...的情况
            {
              if((regFixed.fix&0x800) == 0x800)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0xF800)>>11) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 12 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 12;
                  out.mnt = ((regFixed.fix&0xF800)>>11) + 1;
                }
              }
              else
              {
                out.exp = 12;
                out.mnt = ((regFixed.fix&0xF800)>>11);
              }
            }
            else if((regFixed.fix&0x7FF) > 0x400)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0xF800)>>11) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 12 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 12;
                  out.mnt = ((regFixed.fix&0xF800)>>11) + 1;
                }
              }
              else
              {
                out.exp = 12;
                out.mnt = ((regFixed.fix&0xF800)>>11);
              }
          }
        else if((regFixed.fix >= 0x8000) && (regFixed.fix < 0x10000))//格式为1XXX.XX_XX XXXX XXXX
          {
            if((regFixed.fix&0x3FF) == 0x200)//余位为1000...的情况
            {
              if((regFixed.fix&0x400) == 0x400)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x7C00)>>10) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 11 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 11;
                  out.mnt = ((regFixed.fix&0x7C00)>>10) + 1;
                }
              }
              else
              {
                out.exp = 11;
                out.mnt = ((regFixed.fix&0x7C00)>>10);
              }
            }
            else if((regFixed.fix&0x3FF) > 0x200)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0x7C00)>>10) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 11 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 11;
                  out.mnt = ((regFixed.fix&0x7C00)>>10) + 1;
                }
              }
              else
              {
                out.exp = 11;
                out.mnt = ((regFixed.fix&0x7C00)>>10);
              }
          }
        else if((regFixed.fix >= 0x4000) && (regFixed.fix < 0x8000))//格式为1XX.XXX_X XXXX XXXX
          {
            if((regFixed.fix&0x1FF) == 0x100)//余位为1000...的情况
            {
              if((regFixed.fix&0x200) == 0x200)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x3E00)>>9) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 10 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 10;
                  out.mnt = ((regFixed.fix&0x3E00)>>9) + 1;
                }
              }
              else
              {
                out.exp = 10;
                out.mnt = ((regFixed.fix&0x3E00)>>9);
              }
            }
            else if((regFixed.fix&0x1FF) > 0x100)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0x3E00)>>9) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 10 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 10;
                  out.mnt = ((regFixed.fix&0x3E00)>>9) + 1;
                }
              }
              else
              {
                out.exp = 10;
                out.mnt = ((regFixed.fix&0x3E00)>>9);
              }
          }
        else if((regFixed.fix >= 0x2000) && (regFixed.fix < 0x4000))//格式为1X.XXXX _XXXX XXXX
          {
            if((regFixed.fix&0xFF) == 0x80)//余位为1000...的情况
            {
              if((regFixed.fix&0x100) == 0x100)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x1F00)>>8) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 9 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 9;
                  out.mnt = ((regFixed.fix&0x1F00)>>8) + 1;
                }
              }
              else
              {
                out.exp = 9;
                out.mnt = ((regFixed.fix&0x1F00)>>8);
              }
            }
            else if((regFixed.fix&0xFF) > 0x80)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0x1F00)>>8) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 9 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 9;
                  out.mnt = ((regFixed.fix&0x1F00)>>8) + 1;
                }
              }
              else
              {
                out.exp = 9;
                out.mnt = ((regFixed.fix&0x1F00)>>8);
              }
          }
        else if((regFixed.fix >= 0x1000) && (regFixed.fix < 0x2000))//格式为1.XXXX X_XXX XXXX
          {
            if((regFixed.fix&0x7F) == 0x40)//余位为1000...的情况
            {
              if((regFixed.fix&0x80) == 0x80)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0xF80)>>7) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 8 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 8;
                  out.mnt = ((regFixed.fix&0xF80)>>7) + 1;
                }
              }
              else
              {
                out.exp = 8;
                out.mnt = ((regFixed.fix&0xF80)>>7);
              }
            }
            else if((regFixed.fix&0x7F) > 0x40)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0xF80)>>7) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 8 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 8;
                  out.mnt = ((regFixed.fix&0xF80)>>7) + 1;
                }
              }
              else
              {
                out.exp = 8;
                out.mnt = ((regFixed.fix&0xF80)>>7);
              }
          }
        else if((regFixed.fix >= 0x800) && (regFixed.fix < 0x1000))//格式为0.1XXX XX_XX XXXX
          {
            if((regFixed.fix&0x3F) == 0x20)//余位为1000...的情况
            {
              if((regFixed.fix&0x40) == 0x40)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x7C0)>>6) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 7 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 7;
                  out.mnt = ((regFixed.fix&0x7C0)>>6) + 1;
                }
              }
              else
              {
                out.exp = 7;
                out.mnt = ((regFixed.fix&0x7C0)>>6);
              }
            }
            else if((regFixed.fix&0x3F) > 0x20)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0x7C0)>>6) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 7 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 7;
                  out.mnt = ((regFixed.fix&0x7C0)>>6) + 1;
                }
              }
              else
              {
                out.exp = 7;
                out.mnt = ((regFixed.fix&0x7C0)>>6);
              }
          }
        else if((regFixed.fix >= 0x400) && (regFixed.fix < 0x800))//格式为0.01XX XXX_X XXXX
          {
            if((regFixed.fix&0x1F) == 0x10)//余位为1000...的情况
            {
              if((regFixed.fix&0x20) == 0x20)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x3E0)>>5) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 6 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 6;
                  out.mnt = ((regFixed.fix&0x3E0)>>5) + 1;
                }
              }
              else
              {
                out.exp = 6;
                out.mnt = ((regFixed.fix&0x3E0)>>5);
              }
            }
            else if((regFixed.fix&0x1F) > 0x10)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0x3E0)>>5) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 6 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 6;
                  out.mnt = ((regFixed.fix&0x3E0)>>5) + 1;
                }
              }
              else
              {
                out.exp = 6;
                out.mnt = ((regFixed.fix&0x3E0)>>5);
              }
          }
        else if((regFixed.fix >= 0x200) && (regFixed.fix < 0x400))//格式为0.001X XXXX _XXXX
          {
            if((regFixed.fix&0xF) == 0x8)//余位为1000...的情况
            {
              if((regFixed.fix&0x10) == 0x10)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x1F0)>>4) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 5 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 5;
                  out.mnt = ((regFixed.fix&0x1F0)>>4) + 1;
                }
              }
              else
              {
                out.exp = 5;
                out.mnt = ((regFixed.fix&0x1F0)>>4);
              }
            }
            else if((regFixed.fix&0xF) > 0x8)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0x1F0)>>4) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 5 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 5;
                  out.mnt = ((regFixed.fix&0x1F0)>>4) + 1;
                }
              }
              else
              {
                out.exp = 5;
                out.mnt = ((regFixed.fix&0x1F0)>>4);
              }
          }
        else if((regFixed.fix >= 0x100) && (regFixed.fix < 0x200))//格式为0.0001 XXXX X_XXX
          {
            if((regFixed.fix&0x7) == 0x4)//余位为1000...的情况
            {
              if((regFixed.fix&0x8) == 0x8)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0xF8)>>3) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 4 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 4;
                  out.mnt = ((regFixed.fix&0xF8)>>3) + 1;
                }
              }
              else
              {
                out.exp = 4;
                out.mnt = ((regFixed.fix&0xF8)>>3);
              }
            }
            else if((regFixed.fix&0x7) > 0x4)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0xF8)>>3) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 4 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 4;
                  out.mnt = ((regFixed.fix&0xF8)>>3) + 1;
                }
              }
              else
              {
                out.exp = 4;
                out.mnt = ((regFixed.fix&0xF8)>>3);
              }
          }
        else if((regFixed.fix >= 0x80) && (regFixed.fix < 0x100))//格式为0.0000 1XXX XX_XX
          {
            if((regFixed.fix&0x3) == 0x2)//余位为1000...的情况
            {
              if((regFixed.fix&0x4) == 0x4)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x7C)>>2) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 3 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 3;
                  out.mnt = ((regFixed.fix&0x7C)>>2) + 1;
                }
              }
              else
              {
                out.exp = 3;
                out.mnt = ((regFixed.fix&0x7C)>>2);
              }
            }
            else if((regFixed.fix&0x3) > 0x2)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0x7C)>>2) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 3 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 3;
                  out.mnt = ((regFixed.fix&0x7C)>>2) + 1;
                }
              }
              else
              {
                out.exp = 3;
                out.mnt = ((regFixed.fix&0x7C)>>2);
              }
          }
        else if((regFixed.fix >= 0x40) && (regFixed.fix < 0x80))//格式为0.0000 01XX XXX_X
          {
            if((regFixed.fix&0x1) == 0x1)//余位为1000...的情况
            {
              if((regFixed.fix&0x2) == 0x2)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x3E)>>1) == 31)//处理 mnt = 111的情况
                {
                  out.exp = 2 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 2;
                  out.mnt = ((regFixed.fix&0x3E)>>1) + 1;
                }
              }
              else
              {
                out.exp = 2;
                out.mnt = ((regFixed.fix&0x3E)>>1);
              }
            }
            else
              {
                out.exp = 2;
                out.mnt = ((regFixed.fix&0x3E)>>1);
              }
          }
        else if((regFixed.fix >= 0x20) && (regFixed.fix < 0x40))//格式为0.0000 001X XXXX_
          {
            out.exp = 1;
            out.mnt = (regFixed.fix&0x1F);
          }
        else//格式为0.0000 0001 XXXX
          {
            out.exp = 1;
            out.mnt = 0;
          }
      }
      }

    else if(type == 3)//type == 3
    {
      reg.sign = (a >= 0)?0:1;//符号位
      reg.integer = (int)abs(a);//整数部分
      Mfraction = abs(a) - reg.integer;//取小数部分
      reg.fraction = 0;
      for(int i = 0; i < 11; ++i)
      {
        reg.fraction  = reg.fraction << 1; 
        if( Mfraction*2 >= 1)
        {
          reg.fraction = reg.fraction + 1;
          Mfraction = Mfraction*2 - 1;
        }
        else
        {
          reg.fraction = reg.fraction + 0;
          Mfraction = Mfraction*2;
        }
      }
      regFixed.fix  = ((reg.fraction&0b1) == 1)?((reg.integer << 10) + (reg.fraction>>1) + 1) : ((reg.integer << 10) + (reg.fraction>>1));
      //首先判断符号位
      if( reg.sign == 1 )
      {
        out.sign = 1;
      }
      else
      {
        out.sign = 0;
      }

      //判断是否超出SFP表示范围
      if(regFixed.fix >= 0x3E000)//格式为11 111X XX.XX XXXX XXXX
      {
        out.exp = 15;
        out.mnt = 7;
      }
      else if(regFixed.fix <= 0x4)//格式为0.00 0000 0100 
      {
        out.exp = 0;
        out.mnt = 0;
      }
      else
      {
        if(regFixed.fix >= 0x20000)//格式为1X XX_XX XX.XX XXXX XXXX
          {
            if((regFixed.fix&0x3FFF) == 0x2000)//余位为1000...的情况
            {
              if((regFixed.fix&0x4000) == 0x4000)//尾数为奇数+1,偶数不进位
              {
                out.exp = 15;
                out.mnt = ((regFixed.fix&0x1C000)>>14) + 1;
              }
              else
              {
                out.exp = 15;
                out.mnt = ((regFixed.fix&0x1C000)>>14);
              }
            }
            else if((regFixed.fix&0x3FFF) > 0x2000)//不是1000...的情况。大于进位，小于舍去
            {
              out.exp = 15;
              out.mnt = ((regFixed.fix&0x1C000)>>14) + 1;
            }
            else
            {
              out.exp = 15;
              out.mnt = ((regFixed.fix&0x1C000)>>14);
            }
          }
        else if((regFixed.fix >= 0x10000) && (regFixed.fix < 0x20000))//格式为1 XXX_X XX.XX XXXX XXXX
          {
            if((regFixed.fix&0x1FFF) == 0x1000)//余位为1000...的情况
            {
              if((regFixed.fix&0x2000) == 0x2000)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0xE000)>>13) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 14 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 14;
                  out.mnt = ((regFixed.fix&0xE000)>>13) + 1;
                }
              }
              else
              {
                out.exp = 14;
                out.mnt = ((regFixed.fix&0xE000)>>13);
              }
            }
            else if((regFixed.fix&0x1FFF) > 0x1000)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0xE000)>>13) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 14 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 14;
                  out.mnt = ((regFixed.fix&0xE000)>>13) + 1;
                }
            }
            else
            {
              out.exp = 14;
              out.mnt = ((regFixed.fix&0xE000)>>13);
            }
          }
        else if((regFixed.fix >= 0x8000) && (regFixed.fix < 0x10000))//格式为1XXX _XX.XX XXXX XXXX
          {
            if((regFixed.fix&0xFFF) == 0x800)//余位为1000...的情况
            {
              if((regFixed.fix&0x1000) == 0x1000)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x7000)>>12) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 13 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 13;
                  out.mnt = ((regFixed.fix&0x7000)>>12) + 1;
                }
              }
              else
              {
                out.exp = 13;
                out.mnt = ((regFixed.fix&0x7000)>>12);
              }
            }
            else if((regFixed.fix&0xFFF) > 0x800)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0x7000)>>12) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 13 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 13;
                  out.mnt = ((regFixed.fix&0x7000)>>12) + 1;
                }
              }
            else
            {
              out.exp = 13;
              out.mnt = ((regFixed.fix&0x7000)>>12);
            }
          }
        else if((regFixed.fix >= 0x4000) && (regFixed.fix < 0x8000))//格式为1XX X_X.XX XXXX XXXX
          {
            if((regFixed.fix&0x7FF) == 0x400)//余位为1000...的情况
            {
              if((regFixed.fix&0x800) == 0x800)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x3800)>>11) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 12 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 12;
                  out.mnt = ((regFixed.fix&0x3800)>>11) + 1;
                }
              }
              else
              {
                out.exp = 12;
                out.mnt = ((regFixed.fix&0x3800)>>11);
              }
            }
            else if((regFixed.fix&0x7FF) > 0x400)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0x3800)>>11) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 12 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 12;
                  out.mnt = ((regFixed.fix&0x3800)>>11) + 1;
                }
            }
            else
            {
              out.exp = 12;
              out.mnt = ((regFixed.fix&0x3800)>>11);
            }
          }
        else if((regFixed.fix >= 0x2000) && (regFixed.fix < 0x4000))//格式为1X XX._XX XXXX XXXX
          {
            if((regFixed.fix&0x3FF) == 0x200)//余位为1000...的情况
            {
              if((regFixed.fix&0x400) == 0x400)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x1C00)>>10) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 11 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 11;
                  out.mnt = ((regFixed.fix&0x1C00)>>10) + 1;
                }
              }
              else
              {
                out.exp = 11;
                out.mnt = ((regFixed.fix&0x1C00)>>10);
              }
            }
            else if((regFixed.fix&0x3FF) > 0x200)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0x1C00)>>10) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 11 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 11;
                  out.mnt = ((regFixed.fix&0x1C00)>>10) + 1;
                }
              }
            else
            {
              out.exp = 11;
              out.mnt = ((regFixed.fix&0x1C00)>>10);
            }
          }
        else if((regFixed.fix >= 0x1000) && (regFixed.fix < 0x2000))//格式为1 XX.X_X XXXX XXXX
          {
            if((regFixed.fix&0x1FF) == 0x100)//余位为1000...的情况
            {
              if((regFixed.fix&0x200) == 0x200)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0xE00)>>9) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 10 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 10;
                  out.mnt = ((regFixed.fix&0xE00)>>9) + 1;
                }
              }
              else
              {
                out.exp = 10;
                out.mnt = ((regFixed.fix&0xE00)>>9);
              }
            }
            else if((regFixed.fix&0x1FF) > 0x100)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0xE00)>>9) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 10 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 10;
                  out.mnt = ((regFixed.fix&0xE00)>>9) + 1;
                }
              }
            else
            {
              out.exp = 10;
              out.mnt = ((regFixed.fix&0xE00)>>9);
            }
          }
        else if((regFixed.fix >= 0x800) && (regFixed.fix < 0x1000))//格式为1X.XX _XXXX XXXX
          {
            if((regFixed.fix&0xFF) == 0x80)//余位为1000...的情况
            {
              if((regFixed.fix&0x100) == 0x100)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x700)>>8) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 9 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 9;
                  out.mnt = ((regFixed.fix&0x700)>>8) + 1;
                }
              }
              else
              {
                out.exp = 9;
                out.mnt = ((regFixed.fix&0x700)>>8);
              }
            }
            else if((regFixed.fix&0xFF) > 0x80)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0x700)>>8) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 9 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 9;
                  out.mnt = ((regFixed.fix&0x700)>>8) + 1;
                }
              }
            else
            {
              out.exp = 9;
              out.mnt = ((regFixed.fix&0x700)>>8);
            }
          }
        else if((regFixed.fix >= 0x400) && (regFixed.fix < 0x800))//格式为1.XX X_XXX XXXX
          {
            if((regFixed.fix&0x7F) == 0x40)//余位为1000...的情况
            {
              if((regFixed.fix&0x80) == 0x80)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x380)>>7) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 8 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 8;
                  out.mnt = ((regFixed.fix&0x380)>>7) + 1;
                }
              }
              else
              {
                out.exp = 8;
                out.mnt = ((regFixed.fix&0x380)>>7);
              }
            }
            else if((regFixed.fix&0x7F) > 0x40)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0x380)>>7) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 8 + 1;
                  out.mnt = 0;
                }
              else
                {
                  out.exp = 8;
                  out.mnt = ((regFixed.fix&0x380)>>7) + 1;
                }
            }
            else
            {
              out.exp = 8;
              out.mnt = ((regFixed.fix&0x380)>>7);
            }
          }
        else if((regFixed.fix >= 0x200) && (regFixed.fix < 0x400))//格式为0.1X XX_XX XXXX
          {
            if((regFixed.fix&0x3F) == 0x20)//余位为1000...的情况
            {
              if((regFixed.fix&0x40) == 0x40)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x1C0)>>6) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 7 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 7;
                  out.mnt = ((regFixed.fix&0x1C0)>>6) + 1;
                }
              }
              else
              {
                out.exp = 7;
                out.mnt = ((regFixed.fix&0x1C0)>>6);
              }
            }
            else if((regFixed.fix&0x3F) > 0x20)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0x1C0)>>6) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 7 + 1;
                  out.mnt = 0;
                }
              else
                {
                  out.exp = 7;
                  out.mnt = ((regFixed.fix&0x1C0)>>6) + 1;
                }
            }
            else
            {
              out.exp = 7;
              out.mnt = ((regFixed.fix&0x1C0)>>6);
            }
          }
        else if((regFixed.fix >= 0x100) && (regFixed.fix < 0x200))//格式为0.01 XXX_X XXXX
          {
            if((regFixed.fix&0x1F) == 0x10)//余位为1000...的情况
            {
              if((regFixed.fix&0x20) == 0x20)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0xE0)>>5) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 6 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 6;
                  out.mnt = ((regFixed.fix&0xE0)>>5) + 1;
                }
              }
              else
              {
                out.exp = 6;
                out.mnt = ((regFixed.fix&0xE0)>>5);
              }
            }
            else if((regFixed.fix&0x1F) > 0x10)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0xE0)>>5) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 6 + 1;
                  out.mnt = 0;
                }
              else
                {
                  out.exp = 6;
                  out.mnt = ((regFixed.fix&0xE0)>>5) + 1;
                }
              }
            else
            {
              out.exp = 6;
              out.mnt = ((regFixed.fix&0xE0)>>5);
            }
          }
        else if((regFixed.fix >= 0x80) && (regFixed.fix < 0x100))//格式为0.00 1XXX _XXXX
          {
            if((regFixed.fix&0xF) == 0x8)//余位为1000...的情况
            {
              if((regFixed.fix&0x10) == 0x10)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x70)>>4) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 5 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 5;
                  out.mnt = ((regFixed.fix&0x70)>>4) + 1;
                }
              }
              else
              {
                out.exp = 5;
                out.mnt = ((regFixed.fix&0x70)>>4);
              }
            }
            else if((regFixed.fix&0xF) > 0x8)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0x70)>>4) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 5 + 1;
                  out.mnt = 0;
                }
              else
                {
                  out.exp = 5;
                  out.mnt = ((regFixed.fix&0x70)>>4) + 1;
                }
              }
            else
            {
              out.exp = 5;
              out.mnt = ((regFixed.fix&0x70)>>4);
            }
          }
        else if((regFixed.fix >= 0x40) && (regFixed.fix < 0x80))//格式为0.00 01XX X_XXX
          {
            if((regFixed.fix&0x7) == 0x4)//余位为1000...的情况
            {
              if((regFixed.fix&0x8) == 0x8)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x38)>>3) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 4 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 4;
                  out.mnt = ((regFixed.fix&0x38)>>3) + 1;
                }
              }
              else
              {
                out.exp = 4;
                out.mnt = ((regFixed.fix&0x38)>>3);
              }
            }
            else if((regFixed.fix&0x7) > 0x4)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0x38)>>3) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 4 + 1;
                  out.mnt = 0;
                }
              else
                {
                  out.exp = 4;
                  out.mnt = ((regFixed.fix&0x38)>>3) + 1;
                }
              }
            else
            {
              out.exp = 4;
              out.mnt = ((regFixed.fix&0x38)>>3);
            }
          }
        else if((regFixed.fix >= 0x20) && (regFixed.fix < 0x40))//格式为0.00 001X XX_XX
          {
            if((regFixed.fix&0x3) == 0x2)//余位为1000...的情况
            {
              if((regFixed.fix&0x4) == 0x4)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0x1C)>>2) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 3 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 3;
                  out.mnt = ((regFixed.fix&0x1C)>>2) + 1;
                }
              }
              else
              {
                out.exp = 3;
                out.mnt = ((regFixed.fix&0x1C)>>2);
              }
            }
            else if((regFixed.fix&0x3) > 0x2)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0x1C)>>2) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 3 + 1;
                  out.mnt = 0;
                }
              else
                {
                  out.exp = 3;
                  out.mnt = ((regFixed.fix&0x1C)>>2) + 1;
                }
              }
            else
            {
              out.exp = 3;
              out.mnt = ((regFixed.fix&0x1C)>>2);
            }
          }
        else if((regFixed.fix >= 0x10) && (regFixed.fix < 0x20))//格式为0.00 0001 XXX_X
          {
            if((regFixed.fix&0x1) == 0x1)//余位为1000...的情况
            {
              if((regFixed.fix&0x2) == 0x2)//尾数为奇数+1,偶数不进位
              {
                if(((regFixed.fix&0xE)>>1) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 2 + 1;
                  out.mnt = 0;
                }
                else
                {
                  out.exp = 2;
                  out.mnt = ((regFixed.fix&0xE)>>1) + 1;
                }
              }
              else
              {
                out.exp = 2;
                out.mnt = ((regFixed.fix&0xE)>>1);
              }
            }
            else if((regFixed.fix&0x1) > 0x1)//不是1000...的情况。大于进位，小于舍去
            {
              if(((regFixed.fix&0xE)>>1) == 7)//处理 mnt = 111的情况
                {
                  out.exp = 2 + 1;
                  out.mnt = 0;
                }
              else
                {
                  out.exp = 2;
                  out.mnt = ((regFixed.fix&0xE)>>1) + 1;
                }
              }
            else
            {
              out.exp = 2;
              out.mnt = ((regFixed.fix&0xE)>>1);
            }
          }
        else if((regFixed.fix >= 0x8) && (regFixed.fix < 0x10))//格式为0.00 0000 1XXX
          {
            out.exp = 1;
            out.mnt = (regFixed.fix&0x7);
          }
        else//格式为0.00 0000 01XX
          {
            out.exp = 1;
            out.mnt = 0;
          }
      }
    
    }
    else {
     printf("Wrong Type of float2sfp") ;
    }
    return out;
  }

float sfpfixed(sfp active, sfp weight, int type)
  {
    lut regMult;
    float out;   //output float
    if( ((active.exp == 0) || (weight.exp == 0) ) && (type == 3 || type == 5))
    {
      out = 0;
    } 
    else if(type == 5)
    {
      regMult.cbMnt = (active.mnt << 5) + weight.mnt;  
      switch (regMult.cbMnt)
      {
        case 0:    {regMult.outMnt = 0b000000; regMult.carry = 0;}; break;
        case 1:    {regMult.outMnt = 0b000010; regMult.carry = 0;}; break;
        case 2:    {regMult.outMnt = 0b000100; regMult.carry = 0;}; break;
        case 3:    {regMult.outMnt = 0b000110; regMult.carry = 0;}; break;
        case 4:    {regMult.outMnt = 0b001000; regMult.carry = 0;}; break;
        case 5:    {regMult.outMnt = 0b001010; regMult.carry = 0;}; break;
        case 6:    {regMult.outMnt = 0b001100; regMult.carry = 0;}; break;
        case 7:    {regMult.outMnt = 0b001110; regMult.carry = 0;}; break;
        case 8:    {regMult.outMnt = 0b010000; regMult.carry = 0;}; break;
        case 9:    {regMult.outMnt = 0b010010; regMult.carry = 0;}; break;
        case 10:    {regMult.outMnt = 0b010100; regMult.carry = 0;}; break;
        case 11:    {regMult.outMnt = 0b010110; regMult.carry = 0;}; break;
        case 12:    {regMult.outMnt = 0b011000; regMult.carry = 0;}; break;
        case 13:    {regMult.outMnt = 0b011010; regMult.carry = 0;}; break;
        case 14:    {regMult.outMnt = 0b011100; regMult.carry = 0;}; break;
        case 15:    {regMult.outMnt = 0b011110; regMult.carry = 0;}; break;
        case 16:    {regMult.outMnt = 0b100000; regMult.carry = 0;}; break;
        case 17:    {regMult.outMnt = 0b100010; regMult.carry = 0;}; break;
        case 18:    {regMult.outMnt = 0b100100; regMult.carry = 0;}; break;
        case 19:    {regMult.outMnt = 0b100110; regMult.carry = 0;}; break;
        case 20:    {regMult.outMnt = 0b101000; regMult.carry = 0;}; break;
        case 21:    {regMult.outMnt = 0b101010; regMult.carry = 0;}; break;
        case 22:    {regMult.outMnt = 0b101100; regMult.carry = 0;}; break;
        case 23:    {regMult.outMnt = 0b101110; regMult.carry = 0;}; break;
        case 24:    {regMult.outMnt = 0b110000; regMult.carry = 0;}; break;
        case 25:    {regMult.outMnt = 0b110010; regMult.carry = 0;}; break;
        case 26:    {regMult.outMnt = 0b110100; regMult.carry = 0;}; break;
        case 27:    {regMult.outMnt = 0b110110; regMult.carry = 0;}; break;
        case 28:    {regMult.outMnt = 0b111000; regMult.carry = 0;}; break;
        case 29:    {regMult.outMnt = 0b111010; regMult.carry = 0;}; break;
        case 30:    {regMult.outMnt = 0b111100; regMult.carry = 0;}; break;
        case 31:    {regMult.outMnt = 0b111110; regMult.carry = 0;}; break;
        case 32:    {regMult.outMnt = 0b000010; regMult.carry = 0;}; break;
        case 33:    {regMult.outMnt = 0b000100; regMult.carry = 0;}; break;
        case 34:    {regMult.outMnt = 0b000110; regMult.carry = 0;}; break;
        case 35:    {regMult.outMnt = 0b001000; regMult.carry = 0;}; break;
        case 36:    {regMult.outMnt = 0b001010; regMult.carry = 0;}; break;
        case 37:    {regMult.outMnt = 0b001100; regMult.carry = 0;}; break;
        case 38:    {regMult.outMnt = 0b001110; regMult.carry = 0;}; break;
        case 39:    {regMult.outMnt = 0b010000; regMult.carry = 0;}; break;
        case 40:    {regMult.outMnt = 0b010010; regMult.carry = 0;}; break;
        case 41:    {regMult.outMnt = 0b010101; regMult.carry = 0;}; break;
        case 42:    {regMult.outMnt = 0b010111; regMult.carry = 0;}; break;
        case 43:    {regMult.outMnt = 0b011001; regMult.carry = 0;}; break;
        case 44:    {regMult.outMnt = 0b011011; regMult.carry = 0;}; break;
        case 45:    {regMult.outMnt = 0b011101; regMult.carry = 0;}; break;
        case 46:    {regMult.outMnt = 0b011111; regMult.carry = 0;}; break;
        case 47:    {regMult.outMnt = 0b100001; regMult.carry = 0;}; break;
        case 48:    {regMult.outMnt = 0b100011; regMult.carry = 0;}; break;
        case 49:    {regMult.outMnt = 0b100101; regMult.carry = 0;}; break;
        case 50:    {regMult.outMnt = 0b100111; regMult.carry = 0;}; break;
        case 51:    {regMult.outMnt = 0b101001; regMult.carry = 0;}; break;
        case 52:    {regMult.outMnt = 0b101011; regMult.carry = 0;}; break;
        case 53:    {regMult.outMnt = 0b101101; regMult.carry = 0;}; break;
        case 54:    {regMult.outMnt = 0b101111; regMult.carry = 0;}; break;
        case 55:    {regMult.outMnt = 0b110001; regMult.carry = 0;}; break;
        case 56:    {regMult.outMnt = 0b110100; regMult.carry = 0;}; break;
        case 57:    {regMult.outMnt = 0b110110; regMult.carry = 0;}; break;
        case 58:    {regMult.outMnt = 0b111000; regMult.carry = 0;}; break;
        case 59:    {regMult.outMnt = 0b111010; regMult.carry = 0;}; break;
        case 60:    {regMult.outMnt = 0b111100; regMult.carry = 0;}; break;
        case 61:    {regMult.outMnt = 0b111110; regMult.carry = 0;}; break;
        case 62:    {regMult.outMnt = 0b000000; regMult.carry = 1;}; break;
        case 63:    {regMult.outMnt = 0b000001; regMult.carry = 1;}; break;
        case 64:    {regMult.outMnt = 0b000100; regMult.carry = 0;}; break;
        case 65:    {regMult.outMnt = 0b000110; regMult.carry = 0;}; break;
        case 66:    {regMult.outMnt = 0b001000; regMult.carry = 0;}; break;
        case 67:    {regMult.outMnt = 0b001010; regMult.carry = 0;}; break;
        case 68:    {regMult.outMnt = 0b001100; regMult.carry = 0;}; break;
        case 69:    {regMult.outMnt = 0b001111; regMult.carry = 0;}; break;
        case 70:    {regMult.outMnt = 0b010001; regMult.carry = 0;}; break;
        case 71:    {regMult.outMnt = 0b010011; regMult.carry = 0;}; break;
        case 72:    {regMult.outMnt = 0b010101; regMult.carry = 0;}; break;
        case 73:    {regMult.outMnt = 0b010111; regMult.carry = 0;}; break;
        case 74:    {regMult.outMnt = 0b011001; regMult.carry = 0;}; break;
        case 75:    {regMult.outMnt = 0b011011; regMult.carry = 0;}; break;
        case 76:    {regMult.outMnt = 0b011110; regMult.carry = 0;}; break;
        case 77:    {regMult.outMnt = 0b100000; regMult.carry = 0;}; break;
        case 78:    {regMult.outMnt = 0b100010; regMult.carry = 0;}; break;
        case 79:    {regMult.outMnt = 0b100100; regMult.carry = 0;}; break;
        case 80:    {regMult.outMnt = 0b100110; regMult.carry = 0;}; break;
        case 81:    {regMult.outMnt = 0b101000; regMult.carry = 0;}; break;
        case 82:    {regMult.outMnt = 0b101010; regMult.carry = 0;}; break;
        case 83:    {regMult.outMnt = 0b101100; regMult.carry = 0;}; break;
        case 84:    {regMult.outMnt = 0b101110; regMult.carry = 0;}; break;
        case 85:    {regMult.outMnt = 0b110001; regMult.carry = 0;}; break;
        case 86:    {regMult.outMnt = 0b110011; regMult.carry = 0;}; break;
        case 87:    {regMult.outMnt = 0b110101; regMult.carry = 0;}; break;
        case 88:    {regMult.outMnt = 0b110111; regMult.carry = 0;}; break;
        case 89:    {regMult.outMnt = 0b111001; regMult.carry = 0;}; break;
        case 90:    {regMult.outMnt = 0b111011; regMult.carry = 0;}; break;
        case 91:    {regMult.outMnt = 0b111101; regMult.carry = 0;}; break;
        case 92:    {regMult.outMnt = 0b000000; regMult.carry = 1;}; break;
        case 93:    {regMult.outMnt = 0b000001; regMult.carry = 1;}; break;
        case 94:    {regMult.outMnt = 0b000010; regMult.carry = 1;}; break;
        case 95:    {regMult.outMnt = 0b000011; regMult.carry = 1;}; break;
        case 96:    {regMult.outMnt = 0b000110; regMult.carry = 0;}; break;
        case 97:    {regMult.outMnt = 0b001000; regMult.carry = 0;}; break;
        case 98:    {regMult.outMnt = 0b001010; regMult.carry = 0;}; break;
        case 99:    {regMult.outMnt = 0b001101; regMult.carry = 0;}; break;
        case 100:    {regMult.outMnt = 0b001111; regMult.carry = 0;}; break;
        case 101:    {regMult.outMnt = 0b010001; regMult.carry = 0;}; break;
        case 102:    {regMult.outMnt = 0b010011; regMult.carry = 0;}; break;
        case 103:    {regMult.outMnt = 0b010101; regMult.carry = 0;}; break;
        case 104:    {regMult.outMnt = 0b011000; regMult.carry = 0;}; break;
        case 105:    {regMult.outMnt = 0b011010; regMult.carry = 0;}; break;
        case 106:    {regMult.outMnt = 0b011100; regMult.carry = 0;}; break;
        case 107:    {regMult.outMnt = 0b011110; regMult.carry = 0;}; break;
        case 108:    {regMult.outMnt = 0b100000; regMult.carry = 0;}; break;
        case 109:    {regMult.outMnt = 0b100010; regMult.carry = 0;}; break;
        case 110:    {regMult.outMnt = 0b100101; regMult.carry = 0;}; break;
        case 111:    {regMult.outMnt = 0b100111; regMult.carry = 0;}; break;
        case 112:    {regMult.outMnt = 0b101001; regMult.carry = 0;}; break;
        case 113:    {regMult.outMnt = 0b101011; regMult.carry = 0;}; break;
        case 114:    {regMult.outMnt = 0b101101; regMult.carry = 0;}; break;
        case 115:    {regMult.outMnt = 0b110000; regMult.carry = 0;}; break;
        case 116:    {regMult.outMnt = 0b110010; regMult.carry = 0;}; break;
        case 117:    {regMult.outMnt = 0b110100; regMult.carry = 0;}; break;
        case 118:    {regMult.outMnt = 0b110110; regMult.carry = 0;}; break;
        case 119:    {regMult.outMnt = 0b111000; regMult.carry = 0;}; break;
        case 120:    {regMult.outMnt = 0b111010; regMult.carry = 0;}; break;
        case 121:    {regMult.outMnt = 0b111101; regMult.carry = 0;}; break;
        case 122:    {regMult.outMnt = 0b111111; regMult.carry = 0;}; break;
        case 123:    {regMult.outMnt = 0b000001; regMult.carry = 1;}; break;
        case 124:    {regMult.outMnt = 0b000010; regMult.carry = 1;}; break;
        case 125:    {regMult.outMnt = 0b000011; regMult.carry = 1;}; break;
        case 126:    {regMult.outMnt = 0b000100; regMult.carry = 1;}; break;
        case 127:    {regMult.outMnt = 0b000101; regMult.carry = 1;}; break;
        case 128:    {regMult.outMnt = 0b001000; regMult.carry = 0;}; break;
        case 129:    {regMult.outMnt = 0b001010; regMult.carry = 0;}; break;
        case 130:    {regMult.outMnt = 0b001100; regMult.carry = 0;}; break;
        case 131:    {regMult.outMnt = 0b001111; regMult.carry = 0;}; break;
        case 132:    {regMult.outMnt = 0b010001; regMult.carry = 0;}; break;
        case 133:    {regMult.outMnt = 0b010011; regMult.carry = 0;}; break;
        case 134:    {regMult.outMnt = 0b010110; regMult.carry = 0;}; break;
        case 135:    {regMult.outMnt = 0b011000; regMult.carry = 0;}; break;
        case 136:    {regMult.outMnt = 0b011010; regMult.carry = 0;}; break;
        case 137:    {regMult.outMnt = 0b011100; regMult.carry = 0;}; break;
        case 138:    {regMult.outMnt = 0b011110; regMult.carry = 0;}; break;
        case 139:    {regMult.outMnt = 0b100001; regMult.carry = 0;}; break;
        case 140:    {regMult.outMnt = 0b100011; regMult.carry = 0;}; break;
        case 141:    {regMult.outMnt = 0b100101; regMult.carry = 0;}; break;
        case 142:    {regMult.outMnt = 0b101000; regMult.carry = 0;}; break;
        case 143:    {regMult.outMnt = 0b101010; regMult.carry = 0;}; break;
        case 144:    {regMult.outMnt = 0b101100; regMult.carry = 0;}; break;
        case 145:    {regMult.outMnt = 0b101110; regMult.carry = 0;}; break;
        case 146:    {regMult.outMnt = 0b110000; regMult.carry = 0;}; break;
        case 147:    {regMult.outMnt = 0b110011; regMult.carry = 0;}; break;
        case 148:    {regMult.outMnt = 0b110101; regMult.carry = 0;}; break;
        case 149:    {regMult.outMnt = 0b110111; regMult.carry = 0;}; break;
        case 150:    {regMult.outMnt = 0b111010; regMult.carry = 0;}; break;
        case 151:    {regMult.outMnt = 0b111100; regMult.carry = 0;}; break;
        case 152:    {regMult.outMnt = 0b111110; regMult.carry = 0;}; break;
        case 153:    {regMult.outMnt = 0b000000; regMult.carry = 1;}; break;
        case 154:    {regMult.outMnt = 0b000001; regMult.carry = 1;}; break;
        case 155:    {regMult.outMnt = 0b000010; regMult.carry = 1;}; break;
        case 156:    {regMult.outMnt = 0b000100; regMult.carry = 1;}; break;
        case 157:    {regMult.outMnt = 0b000101; regMult.carry = 1;}; break;
        case 158:    {regMult.outMnt = 0b000110; regMult.carry = 1;}; break;
        case 159:    {regMult.outMnt = 0b000111; regMult.carry = 1;}; break;
        case 160:    {regMult.outMnt = 0b001010; regMult.carry = 0;}; break;
        case 161:    {regMult.outMnt = 0b001100; regMult.carry = 0;}; break;
        case 162:    {regMult.outMnt = 0b001111; regMult.carry = 0;}; break;
        case 163:    {regMult.outMnt = 0b010001; regMult.carry = 0;}; break;
        case 164:    {regMult.outMnt = 0b010011; regMult.carry = 0;}; break;
        case 165:    {regMult.outMnt = 0b010110; regMult.carry = 0;}; break;
        case 166:    {regMult.outMnt = 0b011000; regMult.carry = 0;}; break;
        case 167:    {regMult.outMnt = 0b011010; regMult.carry = 0;}; break;
        case 168:    {regMult.outMnt = 0b011100; regMult.carry = 0;}; break;
        case 169:    {regMult.outMnt = 0b011111; regMult.carry = 0;}; break;
        case 170:    {regMult.outMnt = 0b100001; regMult.carry = 0;}; break;
        case 171:    {regMult.outMnt = 0b100011; regMult.carry = 0;}; break;
        case 172:    {regMult.outMnt = 0b100110; regMult.carry = 0;}; break;
        case 173:    {regMult.outMnt = 0b101000; regMult.carry = 0;}; break;
        case 174:    {regMult.outMnt = 0b101010; regMult.carry = 0;}; break;
        case 175:    {regMult.outMnt = 0b101101; regMult.carry = 0;}; break;
        case 176:    {regMult.outMnt = 0b101111; regMult.carry = 0;}; break;
        case 177:    {regMult.outMnt = 0b110001; regMult.carry = 0;}; break;
        case 178:    {regMult.outMnt = 0b110100; regMult.carry = 0;}; break;
        case 179:    {regMult.outMnt = 0b110110; regMult.carry = 0;}; break;
        case 180:    {regMult.outMnt = 0b111000; regMult.carry = 0;}; break;
        case 181:    {regMult.outMnt = 0b111011; regMult.carry = 0;}; break;
        case 182:    {regMult.outMnt = 0b111101; regMult.carry = 0;}; break;
        case 183:    {regMult.outMnt = 0b111111; regMult.carry = 0;}; break;
        case 184:    {regMult.outMnt = 0b000001; regMult.carry = 1;}; break;
        case 185:    {regMult.outMnt = 0b000010; regMult.carry = 1;}; break;
        case 186:    {regMult.outMnt = 0b000011; regMult.carry = 1;}; break;
        case 187:    {regMult.outMnt = 0b000100; regMult.carry = 1;}; break;
        case 188:    {regMult.outMnt = 0b000101; regMult.carry = 1;}; break;
        case 189:    {regMult.outMnt = 0b000111; regMult.carry = 1;}; break;
        case 190:    {regMult.outMnt = 0b001000; regMult.carry = 1;}; break;
        case 191:    {regMult.outMnt = 0b001001; regMult.carry = 1;}; break;
        case 192:    {regMult.outMnt = 0b001100; regMult.carry = 0;}; break;
        case 193:    {regMult.outMnt = 0b001110; regMult.carry = 0;}; break;
        case 194:    {regMult.outMnt = 0b010001; regMult.carry = 0;}; break;
        case 195:    {regMult.outMnt = 0b010011; regMult.carry = 0;}; break;
        case 196:    {regMult.outMnt = 0b010110; regMult.carry = 0;}; break;
        case 197:    {regMult.outMnt = 0b011000; regMult.carry = 0;}; break;
        case 198:    {regMult.outMnt = 0b011010; regMult.carry = 0;}; break;
        case 199:    {regMult.outMnt = 0b011101; regMult.carry = 0;}; break;
        case 200:    {regMult.outMnt = 0b011111; regMult.carry = 0;}; break;
        case 201:    {regMult.outMnt = 0b100001; regMult.carry = 0;}; break;
        case 202:    {regMult.outMnt = 0b100100; regMult.carry = 0;}; break;
        case 203:    {regMult.outMnt = 0b100110; regMult.carry = 0;}; break;
        case 204:    {regMult.outMnt = 0b101000; regMult.carry = 0;}; break;
        case 205:    {regMult.outMnt = 0b101011; regMult.carry = 0;}; break;
        case 206:    {regMult.outMnt = 0b101101; regMult.carry = 0;}; break;
        case 207:    {regMult.outMnt = 0b110000; regMult.carry = 0;}; break;
        case 208:    {regMult.outMnt = 0b110010; regMult.carry = 0;}; break;
        case 209:    {regMult.outMnt = 0b110100; regMult.carry = 0;}; break;
        case 210:    {regMult.outMnt = 0b110111; regMult.carry = 0;}; break;
        case 211:    {regMult.outMnt = 0b111001; regMult.carry = 0;}; break;
        case 212:    {regMult.outMnt = 0b111100; regMult.carry = 0;}; break;
        case 213:    {regMult.outMnt = 0b111110; regMult.carry = 0;}; break;
        case 214:    {regMult.outMnt = 0b000000; regMult.carry = 1;}; break;
        case 215:    {regMult.outMnt = 0b000001; regMult.carry = 1;}; break;
        case 216:    {regMult.outMnt = 0b000010; regMult.carry = 1;}; break;
        case 217:    {regMult.outMnt = 0b000100; regMult.carry = 1;}; break;
        case 218:    {regMult.outMnt = 0b000101; regMult.carry = 1;}; break;
        case 219:    {regMult.outMnt = 0b000110; regMult.carry = 1;}; break;
        case 220:    {regMult.outMnt = 0b000111; regMult.carry = 1;}; break;
        case 221:    {regMult.outMnt = 0b001000; regMult.carry = 1;}; break;
        case 222:    {regMult.outMnt = 0b001010; regMult.carry = 1;}; break;
        case 223:    {regMult.outMnt = 0b001011; regMult.carry = 1;}; break;
        case 224:    {regMult.outMnt = 0b001110; regMult.carry = 0;}; break;
        case 225:    {regMult.outMnt = 0b010000; regMult.carry = 0;}; break;
        case 226:    {regMult.outMnt = 0b010011; regMult.carry = 0;}; break;
        case 227:    {regMult.outMnt = 0b010101; regMult.carry = 0;}; break;
        case 228:    {regMult.outMnt = 0b011000; regMult.carry = 0;}; break;
        case 229:    {regMult.outMnt = 0b011010; regMult.carry = 0;}; break;
        case 230:    {regMult.outMnt = 0b011101; regMult.carry = 0;}; break;
        case 231:    {regMult.outMnt = 0b011111; regMult.carry = 0;}; break;
        case 232:    {regMult.outMnt = 0b100010; regMult.carry = 0;}; break;
        case 233:    {regMult.outMnt = 0b100100; regMult.carry = 0;}; break;
        case 234:    {regMult.outMnt = 0b100110; regMult.carry = 0;}; break;
        case 235:    {regMult.outMnt = 0b101001; regMult.carry = 0;}; break;
        case 236:    {regMult.outMnt = 0b101011; regMult.carry = 0;}; break;
        case 237:    {regMult.outMnt = 0b101110; regMult.carry = 0;}; break;
        case 238:    {regMult.outMnt = 0b110000; regMult.carry = 0;}; break;
        case 239:    {regMult.outMnt = 0b110011; regMult.carry = 0;}; break;
        case 240:    {regMult.outMnt = 0b110101; regMult.carry = 0;}; break;
        case 241:    {regMult.outMnt = 0b110111; regMult.carry = 0;}; break;
        case 242:    {regMult.outMnt = 0b111010; regMult.carry = 0;}; break;
        case 243:    {regMult.outMnt = 0b111100; regMult.carry = 0;}; break;
        case 244:    {regMult.outMnt = 0b111111; regMult.carry = 0;}; break;
        case 245:    {regMult.outMnt = 0b000001; regMult.carry = 1;}; break;
        case 246:    {regMult.outMnt = 0b000010; regMult.carry = 1;}; break;
        case 247:    {regMult.outMnt = 0b000011; regMult.carry = 1;}; break;
        case 248:    {regMult.outMnt = 0b000100; regMult.carry = 1;}; break;
        case 249:    {regMult.outMnt = 0b000101; regMult.carry = 1;}; break;
        case 250:    {regMult.outMnt = 0b000111; regMult.carry = 1;}; break;
        case 251:    {regMult.outMnt = 0b001000; regMult.carry = 1;}; break;
        case 252:    {regMult.outMnt = 0b001001; regMult.carry = 1;}; break;
        case 253:    {regMult.outMnt = 0b001010; regMult.carry = 1;}; break;
        case 254:    {regMult.outMnt = 0b001100; regMult.carry = 1;}; break;
        case 255:    {regMult.outMnt = 0b001101; regMult.carry = 1;}; break;
        case 256:    {regMult.outMnt = 0b010000; regMult.carry = 0;}; break;
        case 257:    {regMult.outMnt = 0b010010; regMult.carry = 0;}; break;
        case 258:    {regMult.outMnt = 0b010101; regMult.carry = 0;}; break;
        case 259:    {regMult.outMnt = 0b011000; regMult.carry = 0;}; break;
        case 260:    {regMult.outMnt = 0b011010; regMult.carry = 0;}; break;
        case 261:    {regMult.outMnt = 0b011100; regMult.carry = 0;}; break;
        case 262:    {regMult.outMnt = 0b011111; regMult.carry = 0;}; break;
        case 263:    {regMult.outMnt = 0b100010; regMult.carry = 0;}; break;
        case 264:    {regMult.outMnt = 0b100100; regMult.carry = 0;}; break;
        case 265:    {regMult.outMnt = 0b100110; regMult.carry = 0;}; break;
        case 266:    {regMult.outMnt = 0b101001; regMult.carry = 0;}; break;
        case 267:    {regMult.outMnt = 0b101100; regMult.carry = 0;}; break;
        case 268:    {regMult.outMnt = 0b101110; regMult.carry = 0;}; break;
        case 269:    {regMult.outMnt = 0b110000; regMult.carry = 0;}; break;
        case 270:    {regMult.outMnt = 0b110011; regMult.carry = 0;}; break;
        case 271:    {regMult.outMnt = 0b110110; regMult.carry = 0;}; break;
        case 272:    {regMult.outMnt = 0b111000; regMult.carry = 0;}; break;
        case 273:    {regMult.outMnt = 0b111010; regMult.carry = 0;}; break;
        case 274:    {regMult.outMnt = 0b111101; regMult.carry = 0;}; break;
        case 275:    {regMult.outMnt = 0b000000; regMult.carry = 1;}; break;
        case 276:    {regMult.outMnt = 0b000001; regMult.carry = 1;}; break;
        case 277:    {regMult.outMnt = 0b000010; regMult.carry = 1;}; break;
        case 278:    {regMult.outMnt = 0b000100; regMult.carry = 1;}; break;
        case 279:    {regMult.outMnt = 0b000101; regMult.carry = 1;}; break;
        case 280:    {regMult.outMnt = 0b000110; regMult.carry = 1;}; break;
        case 281:    {regMult.outMnt = 0b000111; regMult.carry = 1;}; break;
        case 282:    {regMult.outMnt = 0b001000; regMult.carry = 1;}; break;
        case 283:    {regMult.outMnt = 0b001010; regMult.carry = 1;}; break;
        case 284:    {regMult.outMnt = 0b001011; regMult.carry = 1;}; break;
        case 285:    {regMult.outMnt = 0b001100; regMult.carry = 1;}; break;
        case 286:    {regMult.outMnt = 0b001110; regMult.carry = 1;}; break;
        case 287:    {regMult.outMnt = 0b001111; regMult.carry = 1;}; break;
        case 288:    {regMult.outMnt = 0b010010; regMult.carry = 0;}; break;
        case 289:    {regMult.outMnt = 0b010101; regMult.carry = 0;}; break;
        case 290:    {regMult.outMnt = 0b010111; regMult.carry = 0;}; break;
        case 291:    {regMult.outMnt = 0b011010; regMult.carry = 0;}; break;
        case 292:    {regMult.outMnt = 0b011100; regMult.carry = 0;}; break;
        case 293:    {regMult.outMnt = 0b011111; regMult.carry = 0;}; break;
        case 294:    {regMult.outMnt = 0b100001; regMult.carry = 0;}; break;
        case 295:    {regMult.outMnt = 0b100100; regMult.carry = 0;}; break;
        case 296:    {regMult.outMnt = 0b100110; regMult.carry = 0;}; break;
        case 297:    {regMult.outMnt = 0b101001; regMult.carry = 0;}; break;
        case 298:    {regMult.outMnt = 0b101100; regMult.carry = 0;}; break;
        case 299:    {regMult.outMnt = 0b101110; regMult.carry = 0;}; break;
        case 300:    {regMult.outMnt = 0b110001; regMult.carry = 0;}; break;
        case 301:    {regMult.outMnt = 0b110011; regMult.carry = 0;}; break;
        case 302:    {regMult.outMnt = 0b110110; regMult.carry = 0;}; break;
        case 303:    {regMult.outMnt = 0b111000; regMult.carry = 0;}; break;
        case 304:    {regMult.outMnt = 0b111011; regMult.carry = 0;}; break;
        case 305:    {regMult.outMnt = 0b111110; regMult.carry = 0;}; break;
        case 306:    {regMult.outMnt = 0b000000; regMult.carry = 1;}; break;
        case 307:    {regMult.outMnt = 0b000001; regMult.carry = 1;}; break;
        case 308:    {regMult.outMnt = 0b000011; regMult.carry = 1;}; break;
        case 309:    {regMult.outMnt = 0b000100; regMult.carry = 1;}; break;
        case 310:    {regMult.outMnt = 0b000101; regMult.carry = 1;}; break;
        case 311:    {regMult.outMnt = 0b000110; regMult.carry = 1;}; break;
        case 312:    {regMult.outMnt = 0b001000; regMult.carry = 1;}; break;
        case 313:    {regMult.outMnt = 0b001001; regMult.carry = 1;}; break;
        case 314:    {regMult.outMnt = 0b001010; regMult.carry = 1;}; break;
        case 315:    {regMult.outMnt = 0b001100; regMult.carry = 1;}; break;
        case 316:    {regMult.outMnt = 0b001101; regMult.carry = 1;}; break;
        case 317:    {regMult.outMnt = 0b001110; regMult.carry = 1;}; break;
        case 318:    {regMult.outMnt = 0b001111; regMult.carry = 1;}; break;
        case 319:    {regMult.outMnt = 0b010001; regMult.carry = 1;}; break;
        case 320:    {regMult.outMnt = 0b010100; regMult.carry = 0;}; break;
        case 321:    {regMult.outMnt = 0b010111; regMult.carry = 0;}; break;
        case 322:    {regMult.outMnt = 0b011001; regMult.carry = 0;}; break;
        case 323:    {regMult.outMnt = 0b011100; regMult.carry = 0;}; break;
        case 324:    {regMult.outMnt = 0b011110; regMult.carry = 0;}; break;
        case 325:    {regMult.outMnt = 0b100001; regMult.carry = 0;}; break;
        case 326:    {regMult.outMnt = 0b100100; regMult.carry = 0;}; break;
        case 327:    {regMult.outMnt = 0b100110; regMult.carry = 0;}; break;
        case 328:    {regMult.outMnt = 0b101001; regMult.carry = 0;}; break;
        case 329:    {regMult.outMnt = 0b101100; regMult.carry = 0;}; break;
        case 330:    {regMult.outMnt = 0b101110; regMult.carry = 0;}; break;
        case 331:    {regMult.outMnt = 0b110001; regMult.carry = 0;}; break;
        case 332:    {regMult.outMnt = 0b110100; regMult.carry = 0;}; break;
        case 333:    {regMult.outMnt = 0b110110; regMult.carry = 0;}; break;
        case 334:    {regMult.outMnt = 0b111001; regMult.carry = 0;}; break;
        case 335:    {regMult.outMnt = 0b111011; regMult.carry = 0;}; break;
        case 336:    {regMult.outMnt = 0b111110; regMult.carry = 0;}; break;
        case 337:    {regMult.outMnt = 0b000000; regMult.carry = 1;}; break;
        case 338:    {regMult.outMnt = 0b000010; regMult.carry = 1;}; break;
        case 339:    {regMult.outMnt = 0b000011; regMult.carry = 1;}; break;
        case 340:    {regMult.outMnt = 0b000100; regMult.carry = 1;}; break;
        case 341:    {regMult.outMnt = 0b000110; regMult.carry = 1;}; break;
        case 342:    {regMult.outMnt = 0b000111; regMult.carry = 1;}; break;
        case 343:    {regMult.outMnt = 0b001000; regMult.carry = 1;}; break;
        case 344:    {regMult.outMnt = 0b001010; regMult.carry = 1;}; break;
        case 345:    {regMult.outMnt = 0b001011; regMult.carry = 1;}; break;
        case 346:    {regMult.outMnt = 0b001100; regMult.carry = 1;}; break;
        case 347:    {regMult.outMnt = 0b001101; regMult.carry = 1;}; break;
        case 348:    {regMult.outMnt = 0b001111; regMult.carry = 1;}; break;
        case 349:    {regMult.outMnt = 0b010000; regMult.carry = 1;}; break;
        case 350:    {regMult.outMnt = 0b010001; regMult.carry = 1;}; break;
        case 351:    {regMult.outMnt = 0b010011; regMult.carry = 1;}; break;
        case 352:    {regMult.outMnt = 0b010110; regMult.carry = 0;}; break;
        case 353:    {regMult.outMnt = 0b011001; regMult.carry = 0;}; break;
        case 354:    {regMult.outMnt = 0b011011; regMult.carry = 0;}; break;
        case 355:    {regMult.outMnt = 0b011110; regMult.carry = 0;}; break;
        case 356:    {regMult.outMnt = 0b100001; regMult.carry = 0;}; break;
        case 357:    {regMult.outMnt = 0b100011; regMult.carry = 0;}; break;
        case 358:    {regMult.outMnt = 0b100110; regMult.carry = 0;}; break;
        case 359:    {regMult.outMnt = 0b101001; regMult.carry = 0;}; break;
        case 360:    {regMult.outMnt = 0b101100; regMult.carry = 0;}; break;
        case 361:    {regMult.outMnt = 0b101110; regMult.carry = 0;}; break;
        case 362:    {regMult.outMnt = 0b110001; regMult.carry = 0;}; break;
        case 363:    {regMult.outMnt = 0b110100; regMult.carry = 0;}; break;
        case 364:    {regMult.outMnt = 0b110110; regMult.carry = 0;}; break;
        case 365:    {regMult.outMnt = 0b111001; regMult.carry = 0;}; break;
        case 366:    {regMult.outMnt = 0b111100; regMult.carry = 0;}; break;
        case 367:    {regMult.outMnt = 0b111110; regMult.carry = 0;}; break;
        case 368:    {regMult.outMnt = 0b000000; regMult.carry = 1;}; break;
        case 369:    {regMult.outMnt = 0b000010; regMult.carry = 1;}; break;
        case 370:    {regMult.outMnt = 0b000011; regMult.carry = 1;}; break;
        case 371:    {regMult.outMnt = 0b000101; regMult.carry = 1;}; break;
        case 372:    {regMult.outMnt = 0b000110; regMult.carry = 1;}; break;
        case 373:    {regMult.outMnt = 0b000111; regMult.carry = 1;}; break;
        case 374:    {regMult.outMnt = 0b001001; regMult.carry = 1;}; break;
        case 375:    {regMult.outMnt = 0b001010; regMult.carry = 1;}; break;
        case 376:    {regMult.outMnt = 0b001011; regMult.carry = 1;}; break;
        case 377:    {regMult.outMnt = 0b001101; regMult.carry = 1;}; break;
        case 378:    {regMult.outMnt = 0b001110; regMult.carry = 1;}; break;
        case 379:    {regMult.outMnt = 0b001111; regMult.carry = 1;}; break;
        case 380:    {regMult.outMnt = 0b010001; regMult.carry = 1;}; break;
        case 381:    {regMult.outMnt = 0b010010; regMult.carry = 1;}; break;
        case 382:    {regMult.outMnt = 0b010011; regMult.carry = 1;}; break;
        case 383:    {regMult.outMnt = 0b010101; regMult.carry = 1;}; break;
        case 384:    {regMult.outMnt = 0b011000; regMult.carry = 0;}; break;
        case 385:    {regMult.outMnt = 0b011011; regMult.carry = 0;}; break;
        case 386:    {regMult.outMnt = 0b011110; regMult.carry = 0;}; break;
        case 387:    {regMult.outMnt = 0b100000; regMult.carry = 0;}; break;
        case 388:    {regMult.outMnt = 0b100011; regMult.carry = 0;}; break;
        case 389:    {regMult.outMnt = 0b100110; regMult.carry = 0;}; break;
        case 390:    {regMult.outMnt = 0b101000; regMult.carry = 0;}; break;
        case 391:    {regMult.outMnt = 0b101011; regMult.carry = 0;}; break;
        case 392:    {regMult.outMnt = 0b101110; regMult.carry = 0;}; break;
        case 393:    {regMult.outMnt = 0b110001; regMult.carry = 0;}; break;
        case 394:    {regMult.outMnt = 0b110100; regMult.carry = 0;}; break;
        case 395:    {regMult.outMnt = 0b110110; regMult.carry = 0;}; break;
        case 396:    {regMult.outMnt = 0b111001; regMult.carry = 0;}; break;
        case 397:    {regMult.outMnt = 0b111100; regMult.carry = 0;}; break;
        case 398:    {regMult.outMnt = 0b111110; regMult.carry = 0;}; break;
        case 399:    {regMult.outMnt = 0b000001; regMult.carry = 1;}; break;
        case 400:    {regMult.outMnt = 0b000010; regMult.carry = 1;}; break;
        case 401:    {regMult.outMnt = 0b000011; regMult.carry = 1;}; break;
        case 402:    {regMult.outMnt = 0b000101; regMult.carry = 1;}; break;
        case 403:    {regMult.outMnt = 0b000110; regMult.carry = 1;}; break;
        case 404:    {regMult.outMnt = 0b001000; regMult.carry = 1;}; break;
        case 405:    {regMult.outMnt = 0b001001; regMult.carry = 1;}; break;
        case 406:    {regMult.outMnt = 0b001010; regMult.carry = 1;}; break;
        case 407:    {regMult.outMnt = 0b001100; regMult.carry = 1;}; break;
        case 408:    {regMult.outMnt = 0b001101; regMult.carry = 1;}; break;
        case 409:    {regMult.outMnt = 0b001110; regMult.carry = 1;}; break;
        case 410:    {regMult.outMnt = 0b010000; regMult.carry = 1;}; break;
        case 411:    {regMult.outMnt = 0b010001; regMult.carry = 1;}; break;
        case 412:    {regMult.outMnt = 0b010010; regMult.carry = 1;}; break;
        case 413:    {regMult.outMnt = 0b010100; regMult.carry = 1;}; break;
        case 414:    {regMult.outMnt = 0b010101; regMult.carry = 1;}; break;
        case 415:    {regMult.outMnt = 0b010111; regMult.carry = 1;}; break;
        case 416:    {regMult.outMnt = 0b011010; regMult.carry = 0;}; break;
        case 417:    {regMult.outMnt = 0b011101; regMult.carry = 0;}; break;
        case 418:    {regMult.outMnt = 0b100000; regMult.carry = 0;}; break;
        case 419:    {regMult.outMnt = 0b100010; regMult.carry = 0;}; break;
        case 420:    {regMult.outMnt = 0b100101; regMult.carry = 0;}; break;
        case 421:    {regMult.outMnt = 0b101000; regMult.carry = 0;}; break;
        case 422:    {regMult.outMnt = 0b101011; regMult.carry = 0;}; break;
        case 423:    {regMult.outMnt = 0b101110; regMult.carry = 0;}; break;
        case 424:    {regMult.outMnt = 0b110000; regMult.carry = 0;}; break;
        case 425:    {regMult.outMnt = 0b110011; regMult.carry = 0;}; break;
        case 426:    {regMult.outMnt = 0b110110; regMult.carry = 0;}; break;
        case 427:    {regMult.outMnt = 0b111001; regMult.carry = 0;}; break;
        case 428:    {regMult.outMnt = 0b111100; regMult.carry = 0;}; break;
        case 429:    {regMult.outMnt = 0b111111; regMult.carry = 0;}; break;
        case 430:    {regMult.outMnt = 0b000001; regMult.carry = 1;}; break;
        case 431:    {regMult.outMnt = 0b000010; regMult.carry = 1;}; break;
        case 432:    {regMult.outMnt = 0b000100; regMult.carry = 1;}; break;
        case 433:    {regMult.outMnt = 0b000101; regMult.carry = 1;}; break;
        case 434:    {regMult.outMnt = 0b000110; regMult.carry = 1;}; break;
        case 435:    {regMult.outMnt = 0b001000; regMult.carry = 1;}; break;
        case 436:    {regMult.outMnt = 0b001001; regMult.carry = 1;}; break;
        case 437:    {regMult.outMnt = 0b001011; regMult.carry = 1;}; break;
        case 438:    {regMult.outMnt = 0b001100; regMult.carry = 1;}; break;
        case 439:    {regMult.outMnt = 0b001101; regMult.carry = 1;}; break;
        case 440:    {regMult.outMnt = 0b001111; regMult.carry = 1;}; break;
        case 441:    {regMult.outMnt = 0b010000; regMult.carry = 1;}; break;
        case 442:    {regMult.outMnt = 0b010010; regMult.carry = 1;}; break;
        case 443:    {regMult.outMnt = 0b010011; regMult.carry = 1;}; break;
        case 444:    {regMult.outMnt = 0b010100; regMult.carry = 1;}; break;
        case 445:    {regMult.outMnt = 0b010110; regMult.carry = 1;}; break;
        case 446:    {regMult.outMnt = 0b010111; regMult.carry = 1;}; break;
        case 447:    {regMult.outMnt = 0b011001; regMult.carry = 1;}; break;
        case 448:    {regMult.outMnt = 0b011100; regMult.carry = 0;}; break;
        case 449:    {regMult.outMnt = 0b011111; regMult.carry = 0;}; break;
        case 450:    {regMult.outMnt = 0b100010; regMult.carry = 0;}; break;
        case 451:    {regMult.outMnt = 0b100101; regMult.carry = 0;}; break;
        case 452:    {regMult.outMnt = 0b101000; regMult.carry = 0;}; break;
        case 453:    {regMult.outMnt = 0b101010; regMult.carry = 0;}; break;
        case 454:    {regMult.outMnt = 0b101101; regMult.carry = 0;}; break;
        case 455:    {regMult.outMnt = 0b110000; regMult.carry = 0;}; break;
        case 456:    {regMult.outMnt = 0b110011; regMult.carry = 0;}; break;
        case 457:    {regMult.outMnt = 0b110110; regMult.carry = 0;}; break;
        case 458:    {regMult.outMnt = 0b111001; regMult.carry = 0;}; break;
        case 459:    {regMult.outMnt = 0b111100; regMult.carry = 0;}; break;
        case 460:    {regMult.outMnt = 0b111110; regMult.carry = 0;}; break;
        case 461:    {regMult.outMnt = 0b000001; regMult.carry = 1;}; break;
        case 462:    {regMult.outMnt = 0b000010; regMult.carry = 1;}; break;
        case 463:    {regMult.outMnt = 0b000100; regMult.carry = 1;}; break;
        case 464:    {regMult.outMnt = 0b000101; regMult.carry = 1;}; break;
        case 465:    {regMult.outMnt = 0b000110; regMult.carry = 1;}; break;
        case 466:    {regMult.outMnt = 0b001000; regMult.carry = 1;}; break;
        case 467:    {regMult.outMnt = 0b001001; regMult.carry = 1;}; break;
        case 468:    {regMult.outMnt = 0b001011; regMult.carry = 1;}; break;
        case 469:    {regMult.outMnt = 0b001100; regMult.carry = 1;}; break;
        case 470:    {regMult.outMnt = 0b001110; regMult.carry = 1;}; break;
        case 471:    {regMult.outMnt = 0b001111; regMult.carry = 1;}; break;
        case 472:    {regMult.outMnt = 0b010000; regMult.carry = 1;}; break;
        case 473:    {regMult.outMnt = 0b010010; regMult.carry = 1;}; break;
        case 474:    {regMult.outMnt = 0b010011; regMult.carry = 1;}; break;
        case 475:    {regMult.outMnt = 0b010101; regMult.carry = 1;}; break;
        case 476:    {regMult.outMnt = 0b010110; regMult.carry = 1;}; break;
        case 477:    {regMult.outMnt = 0b011000; regMult.carry = 1;}; break;
        case 478:    {regMult.outMnt = 0b011001; regMult.carry = 1;}; break;
        case 479:    {regMult.outMnt = 0b011011; regMult.carry = 1;}; break;
        case 480:    {regMult.outMnt = 0b011110; regMult.carry = 0;}; break;
        case 481:    {regMult.outMnt = 0b100001; regMult.carry = 0;}; break;
        case 482:    {regMult.outMnt = 0b100100; regMult.carry = 0;}; break;
        case 483:    {regMult.outMnt = 0b100111; regMult.carry = 0;}; break;
        case 484:    {regMult.outMnt = 0b101010; regMult.carry = 0;}; break;
        case 485:    {regMult.outMnt = 0b101101; regMult.carry = 0;}; break;
        case 486:    {regMult.outMnt = 0b110000; regMult.carry = 0;}; break;
        case 487:    {regMult.outMnt = 0b110011; regMult.carry = 0;}; break;
        case 488:    {regMult.outMnt = 0b110110; regMult.carry = 0;}; break;
        case 489:    {regMult.outMnt = 0b111000; regMult.carry = 0;}; break;
        case 490:    {regMult.outMnt = 0b111011; regMult.carry = 0;}; break;
        case 491:    {regMult.outMnt = 0b111110; regMult.carry = 0;}; break;
        case 492:    {regMult.outMnt = 0b000001; regMult.carry = 1;}; break;
        case 493:    {regMult.outMnt = 0b000010; regMult.carry = 1;}; break;
        case 494:    {regMult.outMnt = 0b000100; regMult.carry = 1;}; break;
        case 495:    {regMult.outMnt = 0b000101; regMult.carry = 1;}; break;
        case 496:    {regMult.outMnt = 0b000110; regMult.carry = 1;}; break;
        case 497:    {regMult.outMnt = 0b001000; regMult.carry = 1;}; break;
        case 498:    {regMult.outMnt = 0b001001; regMult.carry = 1;}; break;
        case 499:    {regMult.outMnt = 0b001011; regMult.carry = 1;}; break;
        case 500:    {regMult.outMnt = 0b001100; regMult.carry = 1;}; break;
        case 501:    {regMult.outMnt = 0b001110; regMult.carry = 1;}; break;
        case 502:    {regMult.outMnt = 0b001111; regMult.carry = 1;}; break;
        case 503:    {regMult.outMnt = 0b010001; regMult.carry = 1;}; break;
        case 504:    {regMult.outMnt = 0b010010; regMult.carry = 1;}; break;
        case 505:    {regMult.outMnt = 0b010100; regMult.carry = 1;}; break;
        case 506:    {regMult.outMnt = 0b010101; regMult.carry = 1;}; break;
        case 507:    {regMult.outMnt = 0b010111; regMult.carry = 1;}; break;
        case 508:    {regMult.outMnt = 0b011000; regMult.carry = 1;}; break;
        case 509:    {regMult.outMnt = 0b011010; regMult.carry = 1;}; break;
        case 510:    {regMult.outMnt = 0b011011; regMult.carry = 1;}; break;
        case 511:    {regMult.outMnt = 0b011101; regMult.carry = 1;}; break;
        case 512:    {regMult.outMnt = 0b100000; regMult.carry = 0;}; break;
        case 513:    {regMult.outMnt = 0b100011; regMult.carry = 0;}; break;
        case 514:    {regMult.outMnt = 0b100110; regMult.carry = 0;}; break;
        case 515:    {regMult.outMnt = 0b101001; regMult.carry = 0;}; break;
        case 516:    {regMult.outMnt = 0b101100; regMult.carry = 0;}; break;
        case 517:    {regMult.outMnt = 0b101111; regMult.carry = 0;}; break;
        case 518:    {regMult.outMnt = 0b110010; regMult.carry = 0;}; break;
        case 519:    {regMult.outMnt = 0b110101; regMult.carry = 0;}; break;
        case 520:    {regMult.outMnt = 0b111000; regMult.carry = 0;}; break;
        case 521:    {regMult.outMnt = 0b111011; regMult.carry = 0;}; break;
        case 522:    {regMult.outMnt = 0b111110; regMult.carry = 0;}; break;
        case 523:    {regMult.outMnt = 0b000000; regMult.carry = 1;}; break;
        case 524:    {regMult.outMnt = 0b000010; regMult.carry = 1;}; break;
        case 525:    {regMult.outMnt = 0b000100; regMult.carry = 1;}; break;
        case 526:    {regMult.outMnt = 0b000101; regMult.carry = 1;}; break;
        case 527:    {regMult.outMnt = 0b000110; regMult.carry = 1;}; break;
        case 528:    {regMult.outMnt = 0b001000; regMult.carry = 1;}; break;
        case 529:    {regMult.outMnt = 0b001010; regMult.carry = 1;}; break;
        case 530:    {regMult.outMnt = 0b001011; regMult.carry = 1;}; break;
        case 531:    {regMult.outMnt = 0b001100; regMult.carry = 1;}; break;
        case 532:    {regMult.outMnt = 0b001110; regMult.carry = 1;}; break;
        case 533:    {regMult.outMnt = 0b010000; regMult.carry = 1;}; break;
        case 534:    {regMult.outMnt = 0b010001; regMult.carry = 1;}; break;
        case 535:    {regMult.outMnt = 0b010010; regMult.carry = 1;}; break;
        case 536:    {regMult.outMnt = 0b010100; regMult.carry = 1;}; break;
        case 537:    {regMult.outMnt = 0b010110; regMult.carry = 1;}; break;
        case 538:    {regMult.outMnt = 0b010111; regMult.carry = 1;}; break;
        case 539:    {regMult.outMnt = 0b011000; regMult.carry = 1;}; break;
        case 540:    {regMult.outMnt = 0b011010; regMult.carry = 1;}; break;
        case 541:    {regMult.outMnt = 0b011100; regMult.carry = 1;}; break;
        case 542:    {regMult.outMnt = 0b011101; regMult.carry = 1;}; break;
        case 543:    {regMult.outMnt = 0b011110; regMult.carry = 1;}; break;
        case 544:    {regMult.outMnt = 0b100010; regMult.carry = 0;}; break;
        case 545:    {regMult.outMnt = 0b100101; regMult.carry = 0;}; break;
        case 546:    {regMult.outMnt = 0b101000; regMult.carry = 0;}; break;
        case 547:    {regMult.outMnt = 0b101011; regMult.carry = 0;}; break;
        case 548:    {regMult.outMnt = 0b101110; regMult.carry = 0;}; break;
        case 549:    {regMult.outMnt = 0b110001; regMult.carry = 0;}; break;
        case 550:    {regMult.outMnt = 0b110100; regMult.carry = 0;}; break;
        case 551:    {regMult.outMnt = 0b110111; regMult.carry = 0;}; break;
        case 552:    {regMult.outMnt = 0b111010; regMult.carry = 0;}; break;
        case 553:    {regMult.outMnt = 0b111110; regMult.carry = 0;}; break;
        case 554:    {regMult.outMnt = 0b000000; regMult.carry = 1;}; break;
        case 555:    {regMult.outMnt = 0b000010; regMult.carry = 1;}; break;
        case 556:    {regMult.outMnt = 0b000011; regMult.carry = 1;}; break;
        case 557:    {regMult.outMnt = 0b000101; regMult.carry = 1;}; break;
        case 558:    {regMult.outMnt = 0b000110; regMult.carry = 1;}; break;
        case 559:    {regMult.outMnt = 0b001000; regMult.carry = 1;}; break;
        case 560:    {regMult.outMnt = 0b001010; regMult.carry = 1;}; break;
        case 561:    {regMult.outMnt = 0b001011; regMult.carry = 1;}; break;
        case 562:    {regMult.outMnt = 0b001101; regMult.carry = 1;}; break;
        case 563:    {regMult.outMnt = 0b001110; regMult.carry = 1;}; break;
        case 564:    {regMult.outMnt = 0b010000; regMult.carry = 1;}; break;
        case 565:    {regMult.outMnt = 0b010001; regMult.carry = 1;}; break;
        case 566:    {regMult.outMnt = 0b010011; regMult.carry = 1;}; break;
        case 567:    {regMult.outMnt = 0b010100; regMult.carry = 1;}; break;
        case 568:    {regMult.outMnt = 0b010110; regMult.carry = 1;}; break;
        case 569:    {regMult.outMnt = 0b010111; regMult.carry = 1;}; break;
        case 570:    {regMult.outMnt = 0b011001; regMult.carry = 1;}; break;
        case 571:    {regMult.outMnt = 0b011010; regMult.carry = 1;}; break;
        case 572:    {regMult.outMnt = 0b011100; regMult.carry = 1;}; break;
        case 573:    {regMult.outMnt = 0b011101; regMult.carry = 1;}; break;
        case 574:    {regMult.outMnt = 0b011111; regMult.carry = 1;}; break;
        case 575:    {regMult.outMnt = 0b100000; regMult.carry = 1;}; break;
        case 576:    {regMult.outMnt = 0b100100; regMult.carry = 0;}; break;
        case 577:    {regMult.outMnt = 0b100111; regMult.carry = 0;}; break;
        case 578:    {regMult.outMnt = 0b101010; regMult.carry = 0;}; break;
        case 579:    {regMult.outMnt = 0b101101; regMult.carry = 0;}; break;
        case 580:    {regMult.outMnt = 0b110000; regMult.carry = 0;}; break;
        case 581:    {regMult.outMnt = 0b110100; regMult.carry = 0;}; break;
        case 582:    {regMult.outMnt = 0b110111; regMult.carry = 0;}; break;
        case 583:    {regMult.outMnt = 0b111010; regMult.carry = 0;}; break;
        case 584:    {regMult.outMnt = 0b111101; regMult.carry = 0;}; break;
        case 585:    {regMult.outMnt = 0b000000; regMult.carry = 1;}; break;
        case 586:    {regMult.outMnt = 0b000010; regMult.carry = 1;}; break;
        case 587:    {regMult.outMnt = 0b000011; regMult.carry = 1;}; break;
        case 588:    {regMult.outMnt = 0b000101; regMult.carry = 1;}; break;
        case 589:    {regMult.outMnt = 0b000110; regMult.carry = 1;}; break;
        case 590:    {regMult.outMnt = 0b001000; regMult.carry = 1;}; break;
        case 591:    {regMult.outMnt = 0b001001; regMult.carry = 1;}; break;
        case 592:    {regMult.outMnt = 0b001011; regMult.carry = 1;}; break;
        case 593:    {regMult.outMnt = 0b001101; regMult.carry = 1;}; break;
        case 594:    {regMult.outMnt = 0b001110; regMult.carry = 1;}; break;
        case 595:    {regMult.outMnt = 0b010000; regMult.carry = 1;}; break;
        case 596:    {regMult.outMnt = 0b010001; regMult.carry = 1;}; break;
        case 597:    {regMult.outMnt = 0b010011; regMult.carry = 1;}; break;
        case 598:    {regMult.outMnt = 0b010100; regMult.carry = 1;}; break;
        case 599:    {regMult.outMnt = 0b010110; regMult.carry = 1;}; break;
        case 600:    {regMult.outMnt = 0b011000; regMult.carry = 1;}; break;
        case 601:    {regMult.outMnt = 0b011001; regMult.carry = 1;}; break;
        case 602:    {regMult.outMnt = 0b011011; regMult.carry = 1;}; break;
        case 603:    {regMult.outMnt = 0b011100; regMult.carry = 1;}; break;
        case 604:    {regMult.outMnt = 0b011110; regMult.carry = 1;}; break;
        case 605:    {regMult.outMnt = 0b011111; regMult.carry = 1;}; break;
        case 606:    {regMult.outMnt = 0b100001; regMult.carry = 1;}; break;
        case 607:    {regMult.outMnt = 0b100010; regMult.carry = 1;}; break;
        case 608:    {regMult.outMnt = 0b100110; regMult.carry = 0;}; break;
        case 609:    {regMult.outMnt = 0b101001; regMult.carry = 0;}; break;
        case 610:    {regMult.outMnt = 0b101100; regMult.carry = 0;}; break;
        case 611:    {regMult.outMnt = 0b110000; regMult.carry = 0;}; break;
        case 612:    {regMult.outMnt = 0b110011; regMult.carry = 0;}; break;
        case 613:    {regMult.outMnt = 0b110110; regMult.carry = 0;}; break;
        case 614:    {regMult.outMnt = 0b111001; regMult.carry = 0;}; break;
        case 615:    {regMult.outMnt = 0b111100; regMult.carry = 0;}; break;
        case 616:    {regMult.outMnt = 0b000000; regMult.carry = 1;}; break;
        case 617:    {regMult.outMnt = 0b000001; regMult.carry = 1;}; break;
        case 618:    {regMult.outMnt = 0b000011; regMult.carry = 1;}; break;
        case 619:    {regMult.outMnt = 0b000101; regMult.carry = 1;}; break;
        case 620:    {regMult.outMnt = 0b000110; regMult.carry = 1;}; break;
        case 621:    {regMult.outMnt = 0b001000; regMult.carry = 1;}; break;
        case 622:    {regMult.outMnt = 0b001001; regMult.carry = 1;}; break;
        case 623:    {regMult.outMnt = 0b001011; regMult.carry = 1;}; break;
        case 624:    {regMult.outMnt = 0b001100; regMult.carry = 1;}; break;
        case 625:    {regMult.outMnt = 0b001110; regMult.carry = 1;}; break;
        case 626:    {regMult.outMnt = 0b010000; regMult.carry = 1;}; break;
        case 627:    {regMult.outMnt = 0b010001; regMult.carry = 1;}; break;
        case 628:    {regMult.outMnt = 0b010011; regMult.carry = 1;}; break;
        case 629:    {regMult.outMnt = 0b010100; regMult.carry = 1;}; break;
        case 630:    {regMult.outMnt = 0b010110; regMult.carry = 1;}; break;
        case 631:    {regMult.outMnt = 0b011000; regMult.carry = 1;}; break;
        case 632:    {regMult.outMnt = 0b011001; regMult.carry = 1;}; break;
        case 633:    {regMult.outMnt = 0b011011; regMult.carry = 1;}; break;
        case 634:    {regMult.outMnt = 0b011100; regMult.carry = 1;}; break;
        case 635:    {regMult.outMnt = 0b011110; regMult.carry = 1;}; break;
        case 636:    {regMult.outMnt = 0b100000; regMult.carry = 1;}; break;
        case 637:    {regMult.outMnt = 0b100001; regMult.carry = 1;}; break;
        case 638:    {regMult.outMnt = 0b100011; regMult.carry = 1;}; break;
        case 639:    {regMult.outMnt = 0b100100; regMult.carry = 1;}; break;
        case 640:    {regMult.outMnt = 0b101000; regMult.carry = 0;}; break;
        case 641:    {regMult.outMnt = 0b101011; regMult.carry = 0;}; break;
        case 642:    {regMult.outMnt = 0b101110; regMult.carry = 0;}; break;
        case 643:    {regMult.outMnt = 0b110010; regMult.carry = 0;}; break;
        case 644:    {regMult.outMnt = 0b110101; regMult.carry = 0;}; break;
        case 645:    {regMult.outMnt = 0b111000; regMult.carry = 0;}; break;
        case 646:    {regMult.outMnt = 0b111100; regMult.carry = 0;}; break;
        case 647:    {regMult.outMnt = 0b111111; regMult.carry = 0;}; break;
        case 648:    {regMult.outMnt = 0b000001; regMult.carry = 1;}; break;
        case 649:    {regMult.outMnt = 0b000011; regMult.carry = 1;}; break;
        case 650:    {regMult.outMnt = 0b000100; regMult.carry = 1;}; break;
        case 651:    {regMult.outMnt = 0b000110; regMult.carry = 1;}; break;
        case 652:    {regMult.outMnt = 0b001000; regMult.carry = 1;}; break;
        case 653:    {regMult.outMnt = 0b001001; regMult.carry = 1;}; break;
        case 654:    {regMult.outMnt = 0b001011; regMult.carry = 1;}; break;
        case 655:    {regMult.outMnt = 0b001100; regMult.carry = 1;}; break;
        case 656:    {regMult.outMnt = 0b001110; regMult.carry = 1;}; break;
        case 657:    {regMult.outMnt = 0b010000; regMult.carry = 1;}; break;
        case 658:    {regMult.outMnt = 0b010001; regMult.carry = 1;}; break;
        case 659:    {regMult.outMnt = 0b010011; regMult.carry = 1;}; break;
        case 660:    {regMult.outMnt = 0b010100; regMult.carry = 1;}; break;
        case 661:    {regMult.outMnt = 0b010110; regMult.carry = 1;}; break;
        case 662:    {regMult.outMnt = 0b011000; regMult.carry = 1;}; break;
        case 663:    {regMult.outMnt = 0b011001; regMult.carry = 1;}; break;
        case 664:    {regMult.outMnt = 0b011011; regMult.carry = 1;}; break;
        case 665:    {regMult.outMnt = 0b011101; regMult.carry = 1;}; break;
        case 666:    {regMult.outMnt = 0b011110; regMult.carry = 1;}; break;
        case 667:    {regMult.outMnt = 0b100000; regMult.carry = 1;}; break;
        case 668:    {regMult.outMnt = 0b100010; regMult.carry = 1;}; break;
        case 669:    {regMult.outMnt = 0b100011; regMult.carry = 1;}; break;
        case 670:    {regMult.outMnt = 0b100101; regMult.carry = 1;}; break;
        case 671:    {regMult.outMnt = 0b100110; regMult.carry = 1;}; break;
        case 672:    {regMult.outMnt = 0b101010; regMult.carry = 0;}; break;
        case 673:    {regMult.outMnt = 0b101101; regMult.carry = 0;}; break;
        case 674:    {regMult.outMnt = 0b110001; regMult.carry = 0;}; break;
        case 675:    {regMult.outMnt = 0b110100; regMult.carry = 0;}; break;
        case 676:    {regMult.outMnt = 0b110111; regMult.carry = 0;}; break;
        case 677:    {regMult.outMnt = 0b111011; regMult.carry = 0;}; break;
        case 678:    {regMult.outMnt = 0b111110; regMult.carry = 0;}; break;
        case 679:    {regMult.outMnt = 0b000001; regMult.carry = 1;}; break;
        case 680:    {regMult.outMnt = 0b000010; regMult.carry = 1;}; break;
        case 681:    {regMult.outMnt = 0b000100; regMult.carry = 1;}; break;
        case 682:    {regMult.outMnt = 0b000110; regMult.carry = 1;}; break;
        case 683:    {regMult.outMnt = 0b000111; regMult.carry = 1;}; break;
        case 684:    {regMult.outMnt = 0b001001; regMult.carry = 1;}; break;
        case 685:    {regMult.outMnt = 0b001011; regMult.carry = 1;}; break;
        case 686:    {regMult.outMnt = 0b001100; regMult.carry = 1;}; break;
        case 687:    {regMult.outMnt = 0b001110; regMult.carry = 1;}; break;
        case 688:    {regMult.outMnt = 0b010000; regMult.carry = 1;}; break;
        case 689:    {regMult.outMnt = 0b010001; regMult.carry = 1;}; break;
        case 690:    {regMult.outMnt = 0b010011; regMult.carry = 1;}; break;
        case 691:    {regMult.outMnt = 0b010100; regMult.carry = 1;}; break;
        case 692:    {regMult.outMnt = 0b010110; regMult.carry = 1;}; break;
        case 693:    {regMult.outMnt = 0b011000; regMult.carry = 1;}; break;
        case 694:    {regMult.outMnt = 0b011001; regMult.carry = 1;}; break;
        case 695:    {regMult.outMnt = 0b011011; regMult.carry = 1;}; break;
        case 696:    {regMult.outMnt = 0b011101; regMult.carry = 1;}; break;
        case 697:    {regMult.outMnt = 0b011110; regMult.carry = 1;}; break;
        case 698:    {regMult.outMnt = 0b100000; regMult.carry = 1;}; break;
        case 699:    {regMult.outMnt = 0b100010; regMult.carry = 1;}; break;
        case 700:    {regMult.outMnt = 0b100011; regMult.carry = 1;}; break;
        case 701:    {regMult.outMnt = 0b100101; regMult.carry = 1;}; break;
        case 702:    {regMult.outMnt = 0b100111; regMult.carry = 1;}; break;
        case 703:    {regMult.outMnt = 0b101000; regMult.carry = 1;}; break;
        case 704:    {regMult.outMnt = 0b101100; regMult.carry = 0;}; break;
        case 705:    {regMult.outMnt = 0b101111; regMult.carry = 0;}; break;
        case 706:    {regMult.outMnt = 0b110011; regMult.carry = 0;}; break;
        case 707:    {regMult.outMnt = 0b110110; regMult.carry = 0;}; break;
        case 708:    {regMult.outMnt = 0b111010; regMult.carry = 0;}; break;
        case 709:    {regMult.outMnt = 0b111101; regMult.carry = 0;}; break;
        case 710:    {regMult.outMnt = 0b000000; regMult.carry = 1;}; break;
        case 711:    {regMult.outMnt = 0b000010; regMult.carry = 1;}; break;
        case 712:    {regMult.outMnt = 0b000100; regMult.carry = 1;}; break;
        case 713:    {regMult.outMnt = 0b000101; regMult.carry = 1;}; break;
        case 714:    {regMult.outMnt = 0b000111; regMult.carry = 1;}; break;
        case 715:    {regMult.outMnt = 0b001001; regMult.carry = 1;}; break;
        case 716:    {regMult.outMnt = 0b001010; regMult.carry = 1;}; break;
        case 717:    {regMult.outMnt = 0b001100; regMult.carry = 1;}; break;
        case 718:    {regMult.outMnt = 0b001110; regMult.carry = 1;}; break;
        case 719:    {regMult.outMnt = 0b001111; regMult.carry = 1;}; break;
        case 720:    {regMult.outMnt = 0b010001; regMult.carry = 1;}; break;
        case 721:    {regMult.outMnt = 0b010011; regMult.carry = 1;}; break;
        case 722:    {regMult.outMnt = 0b010100; regMult.carry = 1;}; break;
        case 723:    {regMult.outMnt = 0b010110; regMult.carry = 1;}; break;
        case 724:    {regMult.outMnt = 0b011000; regMult.carry = 1;}; break;
        case 725:    {regMult.outMnt = 0b011001; regMult.carry = 1;}; break;
        case 726:    {regMult.outMnt = 0b011011; regMult.carry = 1;}; break;
        case 727:    {regMult.outMnt = 0b011101; regMult.carry = 1;}; break;
        case 728:    {regMult.outMnt = 0b011110; regMult.carry = 1;}; break;
        case 729:    {regMult.outMnt = 0b100000; regMult.carry = 1;}; break;
        case 730:    {regMult.outMnt = 0b100010; regMult.carry = 1;}; break;
        case 731:    {regMult.outMnt = 0b100100; regMult.carry = 1;}; break;
        case 732:    {regMult.outMnt = 0b100101; regMult.carry = 1;}; break;
        case 733:    {regMult.outMnt = 0b100111; regMult.carry = 1;}; break;
        case 734:    {regMult.outMnt = 0b101001; regMult.carry = 1;}; break;
        case 735:    {regMult.outMnt = 0b101010; regMult.carry = 1;}; break;
        case 736:    {regMult.outMnt = 0b101110; regMult.carry = 0;}; break;
        case 737:    {regMult.outMnt = 0b110001; regMult.carry = 0;}; break;
        case 738:    {regMult.outMnt = 0b110101; regMult.carry = 0;}; break;
        case 739:    {regMult.outMnt = 0b111000; regMult.carry = 0;}; break;
        case 740:    {regMult.outMnt = 0b111100; regMult.carry = 0;}; break;
        case 741:    {regMult.outMnt = 0b111111; regMult.carry = 0;}; break;
        case 742:    {regMult.outMnt = 0b000001; regMult.carry = 1;}; break;
        case 743:    {regMult.outMnt = 0b000011; regMult.carry = 1;}; break;
        case 744:    {regMult.outMnt = 0b000101; regMult.carry = 1;}; break;
        case 745:    {regMult.outMnt = 0b000110; regMult.carry = 1;}; break;
        case 746:    {regMult.outMnt = 0b001000; regMult.carry = 1;}; break;
        case 747:    {regMult.outMnt = 0b001010; regMult.carry = 1;}; break;
        case 748:    {regMult.outMnt = 0b001100; regMult.carry = 1;}; break;
        case 749:    {regMult.outMnt = 0b001101; regMult.carry = 1;}; break;
        case 750:    {regMult.outMnt = 0b001111; regMult.carry = 1;}; break;
        case 751:    {regMult.outMnt = 0b010001; regMult.carry = 1;}; break;
        case 752:    {regMult.outMnt = 0b010010; regMult.carry = 1;}; break;
        case 753:    {regMult.outMnt = 0b010100; regMult.carry = 1;}; break;
        case 754:    {regMult.outMnt = 0b010110; regMult.carry = 1;}; break;
        case 755:    {regMult.outMnt = 0b011000; regMult.carry = 1;}; break;
        case 756:    {regMult.outMnt = 0b011001; regMult.carry = 1;}; break;
        case 757:    {regMult.outMnt = 0b011011; regMult.carry = 1;}; break;
        case 758:    {regMult.outMnt = 0b011101; regMult.carry = 1;}; break;
        case 759:    {regMult.outMnt = 0b011111; regMult.carry = 1;}; break;
        case 760:    {regMult.outMnt = 0b100000; regMult.carry = 1;}; break;
        case 761:    {regMult.outMnt = 0b100010; regMult.carry = 1;}; break;
        case 762:    {regMult.outMnt = 0b100100; regMult.carry = 1;}; break;
        case 763:    {regMult.outMnt = 0b100101; regMult.carry = 1;}; break;
        case 764:    {regMult.outMnt = 0b100111; regMult.carry = 1;}; break;
        case 765:    {regMult.outMnt = 0b101001; regMult.carry = 1;}; break;
        case 766:    {regMult.outMnt = 0b101011; regMult.carry = 1;}; break;
        case 767:    {regMult.outMnt = 0b101100; regMult.carry = 1;}; break;
        case 768:    {regMult.outMnt = 0b110000; regMult.carry = 0;}; break;
        case 769:    {regMult.outMnt = 0b110100; regMult.carry = 0;}; break;
        case 770:    {regMult.outMnt = 0b110111; regMult.carry = 0;}; break;
        case 771:    {regMult.outMnt = 0b111010; regMult.carry = 0;}; break;
        case 772:    {regMult.outMnt = 0b111110; regMult.carry = 0;}; break;
        case 773:    {regMult.outMnt = 0b000001; regMult.carry = 1;}; break;
        case 774:    {regMult.outMnt = 0b000010; regMult.carry = 1;}; break;
        case 775:    {regMult.outMnt = 0b000100; regMult.carry = 1;}; break;
        case 776:    {regMult.outMnt = 0b000110; regMult.carry = 1;}; break;
        case 777:    {regMult.outMnt = 0b001000; regMult.carry = 1;}; break;
        case 778:    {regMult.outMnt = 0b001010; regMult.carry = 1;}; break;
        case 779:    {regMult.outMnt = 0b001011; regMult.carry = 1;}; break;
        case 780:    {regMult.outMnt = 0b001101; regMult.carry = 1;}; break;
        case 781:    {regMult.outMnt = 0b001111; regMult.carry = 1;}; break;
        case 782:    {regMult.outMnt = 0b010000; regMult.carry = 1;}; break;
        case 783:    {regMult.outMnt = 0b010010; regMult.carry = 1;}; break;
        case 784:    {regMult.outMnt = 0b010100; regMult.carry = 1;}; break;
        case 785:    {regMult.outMnt = 0b010110; regMult.carry = 1;}; break;
        case 786:    {regMult.outMnt = 0b011000; regMult.carry = 1;}; break;
        case 787:    {regMult.outMnt = 0b011001; regMult.carry = 1;}; break;
        case 788:    {regMult.outMnt = 0b011011; regMult.carry = 1;}; break;
        case 789:    {regMult.outMnt = 0b011101; regMult.carry = 1;}; break;
        case 790:    {regMult.outMnt = 0b011110; regMult.carry = 1;}; break;
        case 791:    {regMult.outMnt = 0b100000; regMult.carry = 1;}; break;
        case 792:    {regMult.outMnt = 0b100010; regMult.carry = 1;}; break;
        case 793:    {regMult.outMnt = 0b100100; regMult.carry = 1;}; break;
        case 794:    {regMult.outMnt = 0b100110; regMult.carry = 1;}; break;
        case 795:    {regMult.outMnt = 0b100111; regMult.carry = 1;}; break;
        case 796:    {regMult.outMnt = 0b101001; regMult.carry = 1;}; break;
        case 797:    {regMult.outMnt = 0b101011; regMult.carry = 1;}; break;
        case 798:    {regMult.outMnt = 0b101100; regMult.carry = 1;}; break;
        case 799:    {regMult.outMnt = 0b101110; regMult.carry = 1;}; break;
        case 800:    {regMult.outMnt = 0b110010; regMult.carry = 0;}; break;
        case 801:    {regMult.outMnt = 0b110110; regMult.carry = 0;}; break;
        case 802:    {regMult.outMnt = 0b111001; regMult.carry = 0;}; break;
        case 803:    {regMult.outMnt = 0b111101; regMult.carry = 0;}; break;
        case 804:    {regMult.outMnt = 0b000000; regMult.carry = 1;}; break;
        case 805:    {regMult.outMnt = 0b000010; regMult.carry = 1;}; break;
        case 806:    {regMult.outMnt = 0b000100; regMult.carry = 1;}; break;
        case 807:    {regMult.outMnt = 0b000101; regMult.carry = 1;}; break;
        case 808:    {regMult.outMnt = 0b000111; regMult.carry = 1;}; break;
        case 809:    {regMult.outMnt = 0b001001; regMult.carry = 1;}; break;
        case 810:    {regMult.outMnt = 0b001011; regMult.carry = 1;}; break;
        case 811:    {regMult.outMnt = 0b001101; regMult.carry = 1;}; break;
        case 812:    {regMult.outMnt = 0b001110; regMult.carry = 1;}; break;
        case 813:    {regMult.outMnt = 0b010000; regMult.carry = 1;}; break;
        case 814:    {regMult.outMnt = 0b010010; regMult.carry = 1;}; break;
        case 815:    {regMult.outMnt = 0b010100; regMult.carry = 1;}; break;
        case 816:    {regMult.outMnt = 0b010110; regMult.carry = 1;}; break;
        case 817:    {regMult.outMnt = 0b010111; regMult.carry = 1;}; break;
        case 818:    {regMult.outMnt = 0b011001; regMult.carry = 1;}; break;
        case 819:    {regMult.outMnt = 0b011011; regMult.carry = 1;}; break;
        case 820:    {regMult.outMnt = 0b011101; regMult.carry = 1;}; break;
        case 821:    {regMult.outMnt = 0b011110; regMult.carry = 1;}; break;
        case 822:    {regMult.outMnt = 0b100000; regMult.carry = 1;}; break;
        case 823:    {regMult.outMnt = 0b100010; regMult.carry = 1;}; break;
        case 824:    {regMult.outMnt = 0b100100; regMult.carry = 1;}; break;
        case 825:    {regMult.outMnt = 0b100110; regMult.carry = 1;}; break;
        case 826:    {regMult.outMnt = 0b100111; regMult.carry = 1;}; break;
        case 827:    {regMult.outMnt = 0b101001; regMult.carry = 1;}; break;
        case 828:    {regMult.outMnt = 0b101011; regMult.carry = 1;}; break;
        case 829:    {regMult.outMnt = 0b101101; regMult.carry = 1;}; break;
        case 830:    {regMult.outMnt = 0b101110; regMult.carry = 1;}; break;
        case 831:    {regMult.outMnt = 0b110000; regMult.carry = 1;}; break;
        case 832:    {regMult.outMnt = 0b110100; regMult.carry = 0;}; break;
        case 833:    {regMult.outMnt = 0b111000; regMult.carry = 0;}; break;
        case 834:    {regMult.outMnt = 0b111011; regMult.carry = 0;}; break;
        case 835:    {regMult.outMnt = 0b111111; regMult.carry = 0;}; break;
        case 836:    {regMult.outMnt = 0b000001; regMult.carry = 1;}; break;
        case 837:    {regMult.outMnt = 0b000011; regMult.carry = 1;}; break;
        case 838:    {regMult.outMnt = 0b000101; regMult.carry = 1;}; break;
        case 839:    {regMult.outMnt = 0b000111; regMult.carry = 1;}; break;
        case 840:    {regMult.outMnt = 0b001000; regMult.carry = 1;}; break;
        case 841:    {regMult.outMnt = 0b001010; regMult.carry = 1;}; break;
        case 842:    {regMult.outMnt = 0b001100; regMult.carry = 1;}; break;
        case 843:    {regMult.outMnt = 0b001110; regMult.carry = 1;}; break;
        case 844:    {regMult.outMnt = 0b010000; regMult.carry = 1;}; break;
        case 845:    {regMult.outMnt = 0b010010; regMult.carry = 1;}; break;
        case 846:    {regMult.outMnt = 0b010011; regMult.carry = 1;}; break;
        case 847:    {regMult.outMnt = 0b010101; regMult.carry = 1;}; break;
        case 848:    {regMult.outMnt = 0b010111; regMult.carry = 1;}; break;
        case 849:    {regMult.outMnt = 0b011001; regMult.carry = 1;}; break;
        case 850:    {regMult.outMnt = 0b011011; regMult.carry = 1;}; break;
        case 851:    {regMult.outMnt = 0b011100; regMult.carry = 1;}; break;
        case 852:    {regMult.outMnt = 0b011110; regMult.carry = 1;}; break;
        case 853:    {regMult.outMnt = 0b100000; regMult.carry = 1;}; break;
        case 854:    {regMult.outMnt = 0b100010; regMult.carry = 1;}; break;
        case 855:    {regMult.outMnt = 0b100100; regMult.carry = 1;}; break;
        case 856:    {regMult.outMnt = 0b100110; regMult.carry = 1;}; break;
        case 857:    {regMult.outMnt = 0b100111; regMult.carry = 1;}; break;
        case 858:    {regMult.outMnt = 0b101001; regMult.carry = 1;}; break;
        case 859:    {regMult.outMnt = 0b101011; regMult.carry = 1;}; break;
        case 860:    {regMult.outMnt = 0b101101; regMult.carry = 1;}; break;
        case 861:    {regMult.outMnt = 0b101111; regMult.carry = 1;}; break;
        case 862:    {regMult.outMnt = 0b110000; regMult.carry = 1;}; break;
        case 863:    {regMult.outMnt = 0b110010; regMult.carry = 1;}; break;
        case 864:    {regMult.outMnt = 0b110110; regMult.carry = 0;}; break;
        case 865:    {regMult.outMnt = 0b111010; regMult.carry = 0;}; break;
        case 866:    {regMult.outMnt = 0b111101; regMult.carry = 0;}; break;
        case 867:    {regMult.outMnt = 0b000001; regMult.carry = 1;}; break;
        case 868:    {regMult.outMnt = 0b000010; regMult.carry = 1;}; break;
        case 869:    {regMult.outMnt = 0b000100; regMult.carry = 1;}; break;
        case 870:    {regMult.outMnt = 0b000110; regMult.carry = 1;}; break;
        case 871:    {regMult.outMnt = 0b001000; regMult.carry = 1;}; break;
        case 872:    {regMult.outMnt = 0b001010; regMult.carry = 1;}; break;
        case 873:    {regMult.outMnt = 0b001100; regMult.carry = 1;}; break;
        case 874:    {regMult.outMnt = 0b001101; regMult.carry = 1;}; break;
        case 875:    {regMult.outMnt = 0b001111; regMult.carry = 1;}; break;
        case 876:    {regMult.outMnt = 0b010001; regMult.carry = 1;}; break;
        case 877:    {regMult.outMnt = 0b010011; regMult.carry = 1;}; break;
        case 878:    {regMult.outMnt = 0b010101; regMult.carry = 1;}; break;
        case 879:    {regMult.outMnt = 0b010111; regMult.carry = 1;}; break;
        case 880:    {regMult.outMnt = 0b011000; regMult.carry = 1;}; break;
        case 881:    {regMult.outMnt = 0b011010; regMult.carry = 1;}; break;
        case 882:    {regMult.outMnt = 0b011100; regMult.carry = 1;}; break;
        case 883:    {regMult.outMnt = 0b011110; regMult.carry = 1;}; break;
        case 884:    {regMult.outMnt = 0b100000; regMult.carry = 1;}; break;
        case 885:    {regMult.outMnt = 0b100010; regMult.carry = 1;}; break;
        case 886:    {regMult.outMnt = 0b100100; regMult.carry = 1;}; break;
        case 887:    {regMult.outMnt = 0b100101; regMult.carry = 1;}; break;
        case 888:    {regMult.outMnt = 0b100111; regMult.carry = 1;}; break;
        case 889:    {regMult.outMnt = 0b101001; regMult.carry = 1;}; break;
        case 890:    {regMult.outMnt = 0b101011; regMult.carry = 1;}; break;
        case 891:    {regMult.outMnt = 0b101101; regMult.carry = 1;}; break;
        case 892:    {regMult.outMnt = 0b101111; regMult.carry = 1;}; break;
        case 893:    {regMult.outMnt = 0b110000; regMult.carry = 1;}; break;
        case 894:    {regMult.outMnt = 0b110010; regMult.carry = 1;}; break;
        case 895:    {regMult.outMnt = 0b110100; regMult.carry = 1;}; break;
        case 896:    {regMult.outMnt = 0b111000; regMult.carry = 0;}; break;
        case 897:    {regMult.outMnt = 0b111100; regMult.carry = 0;}; break;
        case 898:    {regMult.outMnt = 0b000000; regMult.carry = 1;}; break;
        case 899:    {regMult.outMnt = 0b000010; regMult.carry = 1;}; break;
        case 900:    {regMult.outMnt = 0b000100; regMult.carry = 1;}; break;
        case 901:    {regMult.outMnt = 0b000101; regMult.carry = 1;}; break;
        case 902:    {regMult.outMnt = 0b000111; regMult.carry = 1;}; break;
        case 903:    {regMult.outMnt = 0b001001; regMult.carry = 1;}; break;
        case 904:    {regMult.outMnt = 0b001011; regMult.carry = 1;}; break;
        case 905:    {regMult.outMnt = 0b001101; regMult.carry = 1;}; break;
        case 906:    {regMult.outMnt = 0b001111; regMult.carry = 1;}; break;
        case 907:    {regMult.outMnt = 0b010001; regMult.carry = 1;}; break;
        case 908:    {regMult.outMnt = 0b010010; regMult.carry = 1;}; break;
        case 909:    {regMult.outMnt = 0b010100; regMult.carry = 1;}; break;
        case 910:    {regMult.outMnt = 0b010110; regMult.carry = 1;}; break;
        case 911:    {regMult.outMnt = 0b011000; regMult.carry = 1;}; break;
        case 912:    {regMult.outMnt = 0b011010; regMult.carry = 1;}; break;
        case 913:    {regMult.outMnt = 0b011100; regMult.carry = 1;}; break;
        case 914:    {regMult.outMnt = 0b011110; regMult.carry = 1;}; break;
        case 915:    {regMult.outMnt = 0b100000; regMult.carry = 1;}; break;
        case 916:    {regMult.outMnt = 0b100010; regMult.carry = 1;}; break;
        case 917:    {regMult.outMnt = 0b100011; regMult.carry = 1;}; break;
        case 918:    {regMult.outMnt = 0b100101; regMult.carry = 1;}; break;
        case 919:    {regMult.outMnt = 0b100111; regMult.carry = 1;}; break;
        case 920:    {regMult.outMnt = 0b101001; regMult.carry = 1;}; break;
        case 921:    {regMult.outMnt = 0b101011; regMult.carry = 1;}; break;
        case 922:    {regMult.outMnt = 0b101101; regMult.carry = 1;}; break;
        case 923:    {regMult.outMnt = 0b101111; regMult.carry = 1;}; break;
        case 924:    {regMult.outMnt = 0b110000; regMult.carry = 1;}; break;
        case 925:    {regMult.outMnt = 0b110010; regMult.carry = 1;}; break;
        case 926:    {regMult.outMnt = 0b110100; regMult.carry = 1;}; break;
        case 927:    {regMult.outMnt = 0b110110; regMult.carry = 1;}; break;
        case 928:    {regMult.outMnt = 0b111010; regMult.carry = 0;}; break;
        case 929:    {regMult.outMnt = 0b111110; regMult.carry = 0;}; break;
        case 930:    {regMult.outMnt = 0b000001; regMult.carry = 1;}; break;
        case 931:    {regMult.outMnt = 0b000011; regMult.carry = 1;}; break;
        case 932:    {regMult.outMnt = 0b000101; regMult.carry = 1;}; break;
        case 933:    {regMult.outMnt = 0b000111; regMult.carry = 1;}; break;
        case 934:    {regMult.outMnt = 0b001000; regMult.carry = 1;}; break;
        case 935:    {regMult.outMnt = 0b001010; regMult.carry = 1;}; break;
        case 936:    {regMult.outMnt = 0b001100; regMult.carry = 1;}; break;
        case 937:    {regMult.outMnt = 0b001110; regMult.carry = 1;}; break;
        case 938:    {regMult.outMnt = 0b010000; regMult.carry = 1;}; break;
        case 939:    {regMult.outMnt = 0b010010; regMult.carry = 1;}; break;
        case 940:    {regMult.outMnt = 0b010100; regMult.carry = 1;}; break;
        case 941:    {regMult.outMnt = 0b010110; regMult.carry = 1;}; break;
        case 942:    {regMult.outMnt = 0b011000; regMult.carry = 1;}; break;
        case 943:    {regMult.outMnt = 0b011010; regMult.carry = 1;}; break;
        case 944:    {regMult.outMnt = 0b011100; regMult.carry = 1;}; break;
        case 945:    {regMult.outMnt = 0b011101; regMult.carry = 1;}; break;
        case 946:    {regMult.outMnt = 0b011111; regMult.carry = 1;}; break;
        case 947:    {regMult.outMnt = 0b100001; regMult.carry = 1;}; break;
        case 948:    {regMult.outMnt = 0b100011; regMult.carry = 1;}; break;
        case 949:    {regMult.outMnt = 0b100101; regMult.carry = 1;}; break;
        case 950:    {regMult.outMnt = 0b100111; regMult.carry = 1;}; break;
        case 951:    {regMult.outMnt = 0b101001; regMult.carry = 1;}; break;
        case 952:    {regMult.outMnt = 0b101011; regMult.carry = 1;}; break;
        case 953:    {regMult.outMnt = 0b101101; regMult.carry = 1;}; break;
        case 954:    {regMult.outMnt = 0b101111; regMult.carry = 1;}; break;
        case 955:    {regMult.outMnt = 0b110000; regMult.carry = 1;}; break;
        case 956:    {regMult.outMnt = 0b110010; regMult.carry = 1;}; break;
        case 957:    {regMult.outMnt = 0b110100; regMult.carry = 1;}; break;
        case 958:    {regMult.outMnt = 0b110110; regMult.carry = 1;}; break;
        case 959:    {regMult.outMnt = 0b111000; regMult.carry = 1;}; break;
        case 960:    {regMult.outMnt = 0b111100; regMult.carry = 0;}; break;
        case 961:    {regMult.outMnt = 0b000000; regMult.carry = 1;}; break;
        case 962:    {regMult.outMnt = 0b000010; regMult.carry = 1;}; break;
        case 963:    {regMult.outMnt = 0b000100; regMult.carry = 1;}; break;
        case 964:    {regMult.outMnt = 0b000110; regMult.carry = 1;}; break;
        case 965:    {regMult.outMnt = 0b001000; regMult.carry = 1;}; break;
        case 966:    {regMult.outMnt = 0b001010; regMult.carry = 1;}; break;
        case 967:    {regMult.outMnt = 0b001100; regMult.carry = 1;}; break;
        case 968:    {regMult.outMnt = 0b001110; regMult.carry = 1;}; break;
        case 969:    {regMult.outMnt = 0b001111; regMult.carry = 1;}; break;
        case 970:    {regMult.outMnt = 0b010001; regMult.carry = 1;}; break;
        case 971:    {regMult.outMnt = 0b010011; regMult.carry = 1;}; break;
        case 972:    {regMult.outMnt = 0b010101; regMult.carry = 1;}; break;
        case 973:    {regMult.outMnt = 0b010111; regMult.carry = 1;}; break;
        case 974:    {regMult.outMnt = 0b011001; regMult.carry = 1;}; break;
        case 975:    {regMult.outMnt = 0b011011; regMult.carry = 1;}; break;
        case 976:    {regMult.outMnt = 0b011101; regMult.carry = 1;}; break;
        case 977:    {regMult.outMnt = 0b011111; regMult.carry = 1;}; break;
        case 978:    {regMult.outMnt = 0b100001; regMult.carry = 1;}; break;
        case 979:    {regMult.outMnt = 0b100011; regMult.carry = 1;}; break;
        case 980:    {regMult.outMnt = 0b100101; regMult.carry = 1;}; break;
        case 981:    {regMult.outMnt = 0b100111; regMult.carry = 1;}; break;
        case 982:    {regMult.outMnt = 0b101001; regMult.carry = 1;}; break;
        case 983:    {regMult.outMnt = 0b101011; regMult.carry = 1;}; break;
        case 984:    {regMult.outMnt = 0b101100; regMult.carry = 1;}; break;
        case 985:    {regMult.outMnt = 0b101110; regMult.carry = 1;}; break;
        case 986:    {regMult.outMnt = 0b110000; regMult.carry = 1;}; break;
        case 987:    {regMult.outMnt = 0b110010; regMult.carry = 1;}; break;
        case 988:    {regMult.outMnt = 0b110100; regMult.carry = 1;}; break;
        case 989:    {regMult.outMnt = 0b110110; regMult.carry = 1;}; break;
        case 990:    {regMult.outMnt = 0b111000; regMult.carry = 1;}; break;
        case 991:    {regMult.outMnt = 0b111010; regMult.carry = 1;}; break;
        case 992:    {regMult.outMnt = 0b111110; regMult.carry = 0;}; break;
        case 993:    {regMult.outMnt = 0b000001; regMult.carry = 1;}; break;
        case 994:    {regMult.outMnt = 0b000011; regMult.carry = 1;}; break;
        case 995:    {regMult.outMnt = 0b000101; regMult.carry = 1;}; break;
        case 996:    {regMult.outMnt = 0b000111; regMult.carry = 1;}; break;
        case 997:    {regMult.outMnt = 0b001001; regMult.carry = 1;}; break;
        case 998:    {regMult.outMnt = 0b001011; regMult.carry = 1;}; break;
        case 999:    {regMult.outMnt = 0b001101; regMult.carry = 1;}; break;
        case 1000:    {regMult.outMnt = 0b001111; regMult.carry = 1;}; break;
        case 1001:    {regMult.outMnt = 0b010001; regMult.carry = 1;}; break;
        case 1002:    {regMult.outMnt = 0b010011; regMult.carry = 1;}; break;
        case 1003:    {regMult.outMnt = 0b010101; regMult.carry = 1;}; break;
        case 1004:    {regMult.outMnt = 0b010111; regMult.carry = 1;}; break;
        case 1005:    {regMult.outMnt = 0b011001; regMult.carry = 1;}; break;
        case 1006:    {regMult.outMnt = 0b011011; regMult.carry = 1;}; break;
        case 1007:    {regMult.outMnt = 0b011101; regMult.carry = 1;}; break;
        case 1008:    {regMult.outMnt = 0b011110; regMult.carry = 1;}; break;
        case 1009:    {regMult.outMnt = 0b100000; regMult.carry = 1;}; break;
        case 1010:    {regMult.outMnt = 0b100010; regMult.carry = 1;}; break;
        case 1011:    {regMult.outMnt = 0b100100; regMult.carry = 1;}; break;
        case 1012:    {regMult.outMnt = 0b100110; regMult.carry = 1;}; break;
        case 1013:    {regMult.outMnt = 0b101000; regMult.carry = 1;}; break;
        case 1014:    {regMult.outMnt = 0b101010; regMult.carry = 1;}; break;
        case 1015:    {regMult.outMnt = 0b101100; regMult.carry = 1;}; break;
        case 1016:    {regMult.outMnt = 0b101110; regMult.carry = 1;}; break;
        case 1017:    {regMult.outMnt = 0b110000; regMult.carry = 1;}; break;
        case 1018:    {regMult.outMnt = 0b110010; regMult.carry = 1;}; break;
        case 1019:    {regMult.outMnt = 0b110100; regMult.carry = 1;}; break;
        case 1020:    {regMult.outMnt = 0b110110; regMult.carry = 1;}; break;
        case 1021:    {regMult.outMnt = 0b111000; regMult.carry = 1;}; break;
        case 1022:    {regMult.outMnt = 0b111010; regMult.carry = 1;}; break;
        case 1023:    {regMult.outMnt = 0b111100; regMult.carry = 1;}; break;   
        default:    printf("error\n");
      }

      regMult.outSign = active.sign ^ weight.sign;
      if( (active.exp == 0) || (weight.exp == 0) )
      {
        regMult.outExp = 0;
      }
      else
      {
        regMult.outExp = active.exp + weight.exp + regMult.carry;
        switch (regMult.outMnt)
          {
            case 0:      out = 1.000000;  break;
            case 1:      out = 1.015625;  break;
            case 2:      out = 1.031250;  break;
            case 3:      out = 1.046875;  break;
            case 4:      out = 1.062500;  break;
            case 5:      out = 1.078125;  break;
            case 6:      out = 1.093750;  break;
            case 7:      out = 1.109375;  break;
            case 8:      out = 1.125000;  break;
            case 9:      out = 1.140625;  break;
            case 10:      out = 1.156250;  break;
            case 11:      out = 1.171875;  break;
            case 12:      out = 1.187500;  break;
            case 13:      out = 1.203125;  break;
            case 14:      out = 1.218750;  break;
            case 15:      out = 1.234375;  break;
            case 16:      out = 1.250000;  break;
            case 17:      out = 1.265625;  break;
            case 18:      out = 1.281250;  break;
            case 19:      out = 1.296875;  break;
            case 20:      out = 1.312500;  break;
            case 21:      out = 1.328125;  break;
            case 22:      out = 1.343750;  break;
            case 23:      out = 1.359375;  break;
            case 24:      out = 1.375000;  break;
            case 25:      out = 1.390625;  break;
            case 26:      out = 1.406250;  break;
            case 27:      out = 1.421875;  break;
            case 28:      out = 1.437500;  break;
            case 29:      out = 1.453125;  break;
            case 30:      out = 1.468750;  break;
            case 31:      out = 1.484375;  break;
            case 32:      out = 1.500000;  break;
            case 33:      out = 1.515625;  break;
            case 34:      out = 1.531250;  break;
            case 35:      out = 1.546875;  break;
            case 36:      out = 1.562500;  break;
            case 37:      out = 1.578125;  break;
            case 38:      out = 1.593750;  break;
            case 39:      out = 1.609375;  break;
            case 40:      out = 1.625000;  break;
            case 41:      out = 1.640625;  break;
            case 42:      out = 1.656250;  break;
            case 43:      out = 1.671875;  break;
            case 44:      out = 1.687500;  break;
            case 45:      out = 1.703125;  break;
            case 46:      out = 1.718750;  break;
            case 47:      out = 1.734375;  break;
            case 48:      out = 1.750000;  break;
            case 49:      out = 1.765625;  break;
            case 50:      out = 1.781250;  break;
            case 51:      out = 1.796875;  break;
            case 52:      out = 1.812500;  break;
            case 53:      out = 1.828125;  break;
            case 54:      out = 1.843750;  break;
            case 55:      out = 1.859375;  break;
            case 56:      out = 1.875000;  break;
            case 57:      out = 1.890625;  break;
            case 58:      out = 1.906250;  break;
            case 59:      out = 1.921875;  break;
            case 60:      out = 1.937500;  break;
            case 61:      out = 1.953125;  break;
            case 62:      out = 1.968750;  break;
            case 63:      out = 1.984375;  break;
          }
          
        //指数移位
        if(regMult.outExp == 0)
        {
          out = 0;
        }
        else
        {
          switch (regMult.outExp)
          {
            case 1:     out = out * 3.0517578125e-05;      break;
            case 2:     out = out * 6.103515625e-05;      break;
            case 3:     out = out * 0.0001220703125;      break;
            case 4:     out = out * 0.000244140625;      break;
            case 5:     out = out * 0.00048828125;      break;
            case 6:     out = out * 0.0009765625;      break;
            case 7:     out = out * 0.001953125;      break;
            case 8:     out = out * 0.00390625;      break;
            case 9:     out = out * 0.0078125;      break;
            case 10:    out = out * 0.015625;      break;
            case 11:    out = out * 0.03125;      break;
            case 12:    out = out * 0.0625;      break;
            case 13:    out = out * 0.125;      break;
            case 14:    out = out * 0.25;      break;
            case 15:    out = out * 0.5;      break;
            case 16:    out = out * 1;      break;
            case 17:    out = out * 2;      break;
            case 18:    out = out * 4;      break;
            case 19:    out = out * 8;      break;
            case 20:    out = out * 16;      break;
            case 21:    out = out * 32;      break;
            case 22:    out = out * 64;      break;
            case 23:    out = out * 128;      break;
            case 24:    out = out * 256;      break;
            case 25:    out = out * 512;      break;
            case 26:    out = out * 1024;      break;
            case 27:    out = out * 2048;      break;
            case 28:    out = out * 4096;      break;
            case 29:    out = out * 8192;      break;
            case 30:    out = out * 16384;      break;
            case 31:    out = out * 32768;      break;
          }
        }
      
        //根据符号位判断正负
        out = (regMult.outSign == 1)?(-out):out;
      }
    }
    else if(type == 3)
    {
      regMult.cbMnt = (active.mnt << 3) + weight.mnt;  
      //printf("位数为查找表输入:%d\n",regMult.cbMnt);
      //尾数位lut
      switch (regMult.cbMnt)
      {
        case 0:    {regMult.outMnt = 0b0000; regMult.carry = 0;}; break;
        case 1:    {regMult.outMnt = 0b0010; regMult.carry = 0;}; break;
        case 2:    {regMult.outMnt = 0b0100; regMult.carry = 0;}; break;
        case 3:    {regMult.outMnt = 0b0110; regMult.carry = 0;}; break;
        case 4:    {regMult.outMnt = 0b1000; regMult.carry = 0;}; break;
        case 5:    {regMult.outMnt = 0b1010; regMult.carry = 0;}; break;
        case 6:    {regMult.outMnt = 0b1100; regMult.carry = 0;}; break;
        case 7:    {regMult.outMnt = 0b1110; regMult.carry = 0;}; break;
        case 8:    {regMult.outMnt = 0b0010; regMult.carry = 0;}; break;
        case 9:    {regMult.outMnt = 0b0100; regMult.carry = 0;}; break;
        case 10:   {regMult.outMnt = 0b0110; regMult.carry = 0;}; break;
        case 11:   {regMult.outMnt = 0b1001; regMult.carry = 0;}; break;
        case 12:   {regMult.outMnt = 0b1011; regMult.carry = 0;}; break;
        case 13:   {regMult.outMnt = 0b1101; regMult.carry = 0;}; break;
        case 14:   {regMult.outMnt = 0b0000; regMult.carry = 1;}; break;
        case 15:   {regMult.outMnt = 0b0001; regMult.carry = 1;}; break;
        case 16:   {regMult.outMnt = 0b0100; regMult.carry = 0;}; break;
        case 17:   {regMult.outMnt = 0b0110; regMult.carry = 0;}; break;
        case 18:   {regMult.outMnt = 0b1001; regMult.carry = 0;}; break;
        case 19:   {regMult.outMnt = 0b1100; regMult.carry = 0;}; break;
        case 20:   {regMult.outMnt = 0b1110; regMult.carry = 0;}; break;
        case 21:   {regMult.outMnt = 0b0000; regMult.carry = 1;}; break;
        case 22:   {regMult.outMnt = 0b0010; regMult.carry = 1;}; break;
        case 23:   {regMult.outMnt = 0b0011; regMult.carry = 1;}; break;
        case 24:   {regMult.outMnt = 0b0110; regMult.carry = 0;}; break;
        case 25:   {regMult.outMnt = 0b1001; regMult.carry = 0;}; break;
        case 26:   {regMult.outMnt = 0b1100; regMult.carry = 0;}; break;
        case 27:   {regMult.outMnt = 0b1110; regMult.carry = 0;}; break;
        case 28:   {regMult.outMnt = 0b0000; regMult.carry = 1;}; break;
        case 29:   {regMult.outMnt = 0b0010; regMult.carry = 1;}; break;
        case 30:   {regMult.outMnt = 0b0011; regMult.carry = 1;}; break;
        case 31:   {regMult.outMnt = 0b0101; regMult.carry = 1;}; break;
        case 32:   {regMult.outMnt = 0b1000; regMult.carry = 0;}; break;
        case 33:   {regMult.outMnt = 0b1011; regMult.carry = 0;}; break;
        case 34:   {regMult.outMnt = 0b1110; regMult.carry = 0;}; break;
        case 35:   {regMult.outMnt = 0b0000; regMult.carry = 1;}; break;
        case 36:   {regMult.outMnt = 0b0010; regMult.carry = 1;}; break;
        case 37:   {regMult.outMnt = 0b0100; regMult.carry = 1;}; break;
        case 38:   {regMult.outMnt = 0b0101; regMult.carry = 1;}; break;
        case 39:   {regMult.outMnt = 0b0110; regMult.carry = 1;}; break;
        case 40:   {regMult.outMnt = 0b1010; regMult.carry = 0;}; break;
        case 41:   {regMult.outMnt = 0b1101; regMult.carry = 0;}; break;
        case 42:   {regMult.outMnt = 0b0000; regMult.carry = 1;}; break;
        case 43:   {regMult.outMnt = 0b0010; regMult.carry = 1;}; break;
        case 44:   {regMult.outMnt = 0b0100; regMult.carry = 1;}; break;
        case 45:   {regMult.outMnt = 0b0101; regMult.carry = 1;}; break;
        case 46:   {regMult.outMnt = 0b0111; regMult.carry = 1;}; break;
        case 47:   {regMult.outMnt = 0b1000; regMult.carry = 1;}; break;
        case 48:   {regMult.outMnt = 0b1100; regMult.carry = 0;}; break;
        case 49:   {regMult.outMnt = 0b0000; regMult.carry = 1;}; break;
        case 50:   {regMult.outMnt = 0b0010; regMult.carry = 1;}; break;
        case 51:   {regMult.outMnt = 0b0011; regMult.carry = 1;}; break;
        case 52:   {regMult.outMnt = 0b0101; regMult.carry = 1;}; break;
        case 53:   {regMult.outMnt = 0b0111; regMult.carry = 1;}; break;
        case 54:   {regMult.outMnt = 0b1000; regMult.carry = 1;}; break;
        case 55:   {regMult.outMnt = 0b1010; regMult.carry = 1;}; break;
        case 56:   {regMult.outMnt = 0b1110; regMult.carry = 0;}; break;
        case 57:   {regMult.outMnt = 0b0001; regMult.carry = 1;}; break;
        case 58:   {regMult.outMnt = 0b0011; regMult.carry = 1;}; break;
        case 59:   {regMult.outMnt = 0b0101; regMult.carry = 1;}; break;
        case 60:   {regMult.outMnt = 0b0110; regMult.carry = 1;}; break;
        case 61:   {regMult.outMnt = 0b1000; regMult.carry = 1;}; break;
        case 62:   {regMult.outMnt = 0b1010; regMult.carry = 1;}; break;
        case 63:   {regMult.outMnt = 0b1100; regMult.carry = 1;}; break;     
        default:    printf("error\n");
      }
      regMult.outSign = active.sign ^ weight.sign;
      if((active.exp == 0) || (weight.exp == 0))
      {
        regMult.outExp = 0;
      }
      else
      {
        regMult.outExp = active.exp + weight.exp + regMult.carry;
        //求尾数
        switch (regMult.outMnt)
          {
            case 0:     out = 1 +     0     +     0     +     0     +     0     ;  break;
            case 1:     out = 1 +     0     +     0     +     0     +   0.0625  ;  break;
            case 2:     out = 1 +     0     +     0     +   0.125   +     0     ;  break;
            case 3:     out = 1 +     0     +     0     +   0.125   +   0.0625  ;  break;
            case 4:     out = 1 +     0     +    0.25   +     0     +     0     ;  break;
            case 5:     out = 1 +     0     +    0.25   +     0     +   0.0625  ;  break;
            case 6:     out = 1 +     0     +    0.25   +   0.125   +     0     ;  break; 
            case 7:     out = 1 +     0     +    0.25   +   0.125   +   0.0625  ;  break;
            case 8:     out = 1 +    0.5    +     0     +     0     +     0     ;  break;
            case 9:     out = 1 +    0.5    +     0     +     0     +   0.0625  ;  break;
            case 10:    out = 1 +    0.5    +     0     +   0.125   +     0     ;  break;
            case 11:    out = 1 +    0.5    +     0     +   0.125   +   0.0625  ;  break;
            case 12:    out = 1 +    0.5    +    0.25   +     0     +     0     ;  break; 
            case 13:    out = 1 +    0.5    +    0.25   +     0     +   0.0625  ;  break;
            case 14:    out = 1 +    0.5    +    0.25   +   0.125   +     0     ;  break;
            case 15:    out = 1 +    0.5    +    0.25   +   0.125   +   0.0625  ;  break;   
          }
          
        //指数移位
        if(regMult.outExp == 0)
        {
          out = 0;
        }
        else
        {
          switch (regMult.outExp)
          {
            case 1:     out = out * 3.0517578125e-05;      break;
            case 2:     out = out * 6.103515625e-05;      break;
            case 3:     out = out * 0.0001220703125;      break;
            case 4:     out = out * 0.000244140625;      break;
            case 5:     out = out * 0.00048828125;      break;
            case 6:     out = out * 0.0009765625;      break;
            case 7:     out = out * 0.001953125;      break;
            case 8:     out = out * 0.00390625;      break;
            case 9:     out = out * 0.0078125;      break;
            case 10:    out = out * 0.015625;      break;
            case 11:    out = out * 0.03125;      break;
            case 12:    out = out * 0.0625;      break;
            case 13:    out = out * 0.125;      break;
            case 14:    out = out * 0.25;      break;
            case 15:    out = out * 0.5;      break;
            case 16:    out = out * 1;      break;
            case 17:    out = out * 2;      break;
            case 18:    out = out * 4;      break;
            case 19:    out = out * 8;      break;
            case 20:    out = out * 16;      break;
            case 21:    out = out * 32;      break;
            case 22:    out = out * 64;      break;
            case 23:    out = out * 128;      break;
            case 24:    out = out * 256;      break;
            case 25:    out = out * 512;      break;
            case 26:    out = out * 1024;      break;
            case 27:    out = out * 2048;      break;
            case 28:    out = out * 4096;      break;
            case 29:    out = out * 8192;      break;
            case 30:    out = out * 16384;      break;
            case 31:    out = out * 32768;      break;
          }
        }
      
        //根据符号位判断正负
        out = (regMult.outSign == 1)?(-out):out;
      }
    }
    return out;
  }


int main()
{
    float A = 6.5855;
    int type1 = 3;
    int type2 = 5;
    float out1, out2;
    sfp A_sfp, B_sfp;
    A_sfp = floatsfp(A ,type1);
    B_sfp = floatsfp(A, type2);
    cout << oct << A_sfp.sign <<"\n"<< oct << A_sfp.exp << "\n"<< oct << A_sfp.mnt <<"\n"<< endl;
    cout << oct << B_sfp.sign <<"\n"<< oct << B_sfp.exp << "\n"<< oct << B_sfp.mnt <<"\n"<< endl;

    out1 = sfpfixed(A_sfp, A_sfp, type1);
    out2 = sfpfixed(B_sfp, B_sfp, type2);
    cout << out1 << endl;
    cout << out2 << endl;
}